/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "cudaEvent_queue.h"
#include "../allocators.h"


decx::cudaEvent_Queue::cudaEvent_Queue()
{
    this->_cuda_event_num = 0;
    this->true_capacity = _CS_STREAM_Q_INIT_SIZE_;
    
    // allocate host memory (page-locked) for decx::hip_stream
    if (decx::alloc::_host_virtual_page_malloc(&this->_cuda_event_arr, _CS_STREAM_Q_INIT_SIZE_ * sizeof(decx::hip_stream))) {
        Print_Error_Message(4, "Failed to allocate space for cudaStream on host, cudaEvent_Queue init fail\n");
        exit(-1);
    }
}


decx::cuda_event* decx::cudaEvent_Queue::add_event_physical(const int flag)
{
    if (this->_cuda_event_num > this->true_capacity - 1) {
        // assign a temporary pointer
        decx::PtrInfo<decx::cuda_event> tmp_ptr;
        // physically alloc space for new area
        if (decx::alloc::_alloc_Hv(&(tmp_ptr.block),
            (this->true_capacity + _CS_STREAM_Q_INIT_SIZE_) * sizeof(decx::hip_stream))) {
            Print_Error_Message(4, "Failed to allocate space for cudaStream on host, cudaEvent_Queue fail to add event\n");
            exit(-1);
        }
        tmp_ptr.ptr = reinterpret_cast<decx::cuda_event*>(tmp_ptr.block->_ptr);

        // copy the old data from this to temp
        memcpy(tmp_ptr.ptr, this->_cuda_event_arr.ptr, this->true_capacity * sizeof(decx::cuda_event));
        // refresh this->true_capacity
        this->true_capacity += _CS_STREAM_Q_INIT_SIZE_;
        // deallocate the old memory space
        decx::alloc::_dealloc_Hv(this->_cuda_event_arr.block);
        // assign the new one to the class
        this->_cuda_event_arr = tmp_ptr;

        // alloc one from back (push_back())
        new(this->_cuda_event_arr.ptr + this->_cuda_event_num) decx::cuda_event(flag);
        // increament on this->_cuda_stream_num
        ++this->_cuda_event_num;
    }
    else {
        // alloc one from back (push_back())
        new(this->_cuda_event_arr.ptr + this->_cuda_event_num) decx::cuda_event(flag);
        // increament on this->_cuda_stream_num
        ++this->_cuda_event_num;
    }

    return (this->_cuda_event_arr.ptr + this->_cuda_event_num - 1);
}


bool decx::cudaEvent_Queue::_find_idle_event(uint* res_dex, const int flag)
{
    for (int i = 0; i < this->_cuda_event_num; ++i) {
        decx::cuda_event* _tmpS = this->_cuda_event_arr.ptr + i;
        if (!_tmpS->_is_occupied && _tmpS->_event_flag == flag) {
            *res_dex = i;
            //_tmpS->attach();
            return true;
        }
    }
    return false;
}


decx::cuda_event* decx::cudaEvent_Queue::event_accessor_ptr(const int flag)
{
    uint dex = 0;

    this->_mtx.lock();

    decx::cuda_event* res_ptr = NULL;
    if (this->_find_idle_event(&dex, flag)) {        // found an idle stream
        res_ptr = this->_cuda_event_arr.ptr + dex;
    }
    else {          // all the streams are occupied
        res_ptr = this->add_event_physical(flag);
    }
    res_ptr->attach();

    this->_mtx.unlock();

    return res_ptr;
}


decx::cuda_event& decx::cudaEvent_Queue::event_accessor_ref(const int flag)
{
    uint dex = 0;
    decx::cuda_event* res_ptr = NULL;
    if (this->_find_idle_event(&dex, flag)) {        // found an idle stream
        res_ptr = this->_cuda_event_arr.ptr + dex;
    }
    else {          // all the streams are occupied
        res_ptr = this->add_event_physical(flag);
    }
    res_ptr->attach();
    return *res_ptr;
}



void decx::cudaEvent_Queue::release()
{
    // call hipStreamDestroy on each stream
    for (int i = 0; i < this->_cuda_event_num; ++i) {
        (this->_cuda_event_arr.ptr + i)->release();
    }
    // deallocte the stream array
    decx::alloc::_dealloc_Hv(this->_cuda_event_arr.block);
}


decx::cudaEvent_Queue::~cudaEvent_Queue()
{
    decx::alloc::_dealloc_Hv(this->_cuda_event_arr.block);
}


_DECX_API_ decx::cuda_event* decx::cuda::get_cuda_event_ptr(const int flag)
{
    return decx::CEvent.event_accessor_ptr(flag);
}


_DECX_API_ decx::cuda_event& decx::cuda::get_cuda_event_ref(const int flag)
{
    return decx::CEvent.event_accessor_ref(flag);
}