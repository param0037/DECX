/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "cudaStream_package.h"


decx::hip_stream::hip_stream(const int flag)
{
    checkCudaErrors(hipStreamCreateWithFlags(&this->_S, flag));
    this->_stream_flag = flag;
}


void decx::hip_stream::detach()
{
    this->_is_occupied = false;
}


void decx::hip_stream::attach()
{
    this->_is_occupied = true;
}


void decx::hip_stream::this_stream_sync()
{
    checkCudaErrors(hipStreamSynchronize(this->_S));
}


hipStream_t& decx::hip_stream::get_raw_stream_ref()
{
    return this->_S;
}


hipStream_t* decx::hip_stream::get_raw_stream_ptr()
{
    return &(this->_S);
}


void decx::hip_stream::release()
{
    checkCudaErrors(hipStreamDestroy(this->_S));
}
