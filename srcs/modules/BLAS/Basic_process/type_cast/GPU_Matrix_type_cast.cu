#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "GPU_Matrix_type_cast.cuh"
#include "../../../../common/Basic_process/type_cast/type_cast_methods.h"
#include "../../../../modules/core/configs/config.h"


_DECX_API_ de::DH 
de::cuda::TypeCast(de::GPU_Matrix& src, de::GPU_Matrix& dst, const int cvt_method)
{
    using namespace decx::type_cast;

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init,
            CUDA_NOT_INIT);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Matrix* _src = dynamic_cast<decx::_GPU_Matrix*>(&src);
    decx::_GPU_Matrix* _dst = dynamic_cast<decx::_GPU_Matrix*>(&dst);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT,
            CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM,
            CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    if (cvt_method == TypeCast_Method::CVT_FP32_FP64) {
        decx::type_cast::_mm128_cvtfp32_fp64_caller2D(
            (float4*)_src->Mat.ptr, (double2*)_dst->Mat.ptr, make_ulong2(_dst->Pitch() / 4, _src->Height()), _src->Pitch(), _dst->Pitch(), S);
    }
    else if (cvt_method == TypeCast_Method::CVT_FP64_FP32) {
        decx::type_cast::_mm128_cvtfp64_fp32_caller2D(
            (double2*)_src->Mat.ptr, (float4*)_dst->Mat.ptr, make_ulong2(_src->Pitch() / 4, _src->Height()), _src->Pitch(), _dst->Pitch(), S);
    }
    else if (cvt_method == TypeCast_Method::CVT_INT32_FP32) {
        _dst->re_construct(de::_DATA_TYPES_FLAGS_::_FP32_, _src->Width(), _src->Height());

        decx::type_cast::_mm128_cvti32_fp32_caller1D((int4*)_src->Mat.ptr, (float4*)_dst->Mat.ptr, _src->Pitch() * _src->Height() / 4, S);
    }
    else if (cvt_method == TypeCast_Method::CVT_UINT8_INT32) {
        _dst->re_construct(de::_DATA_TYPES_FLAGS_::_INT32_, _src->Width(), _src->Height());

        decx::type_cast::_mm128_cvtfp32_i32_caller1D(
            (float4*)_src->Mat.ptr, (int4*)_dst->Mat.ptr, _dst->Pitch() * _dst->Height() / 4, S);
    }
    else {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_INVALID_PARAM,
            INVALID_PARAM);
        return handle;
    }

    return handle;
}