/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "GPU_Vector_fill.cuh"


_DECX_API_ de::DH de::cuda::Constant_fp32(GPU_Vector& src, const float value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != de::_DATA_TYPES_FLAGS_::_FP32_) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_TYPE_MOT_MATCH, TYPE_ERROR_NOT_MATCH);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b32_caller((float*)_src->Vec.ptr, value, _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}




_DECX_API_ de::DH de::cuda::Constant_int32(GPU_Vector& src, const int value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != de::_DATA_TYPES_FLAGS_::_INT32_) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_TYPE_MOT_MATCH, TYPE_ERROR_NOT_MATCH);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b32_caller((float*)_src->Vec.ptr, *((float*)&value), _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}




_DECX_API_ de::DH de::cuda::Constant_fp64(GPU_Vector& src, const double value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != de::_DATA_TYPES_FLAGS_::_FP64_) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_TYPE_MOT_MATCH, TYPE_ERROR_NOT_MATCH);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b64_caller((double*)_src->Vec.ptr, value, _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}