/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "GPU_Vector_fill.cuh"


_DECX_API_ de::DH de::cuda::Constant_fp32(GPU_Vector& src, const float value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        Print_Error_Message(4, CUDA_NOT_INIT);
        decx::err::CUDA_Not_init(&handle);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != decx::_DATA_TYPES_FLAGS_::_FP32_) {
        Print_Error_Message(4, TYPE_ERROR_NOT_MATCH);
        decx::err::TypeError_NotMatch(&handle);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b32_caller((float*)_src->Vec.ptr, value, _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}




_DECX_API_ de::DH de::cuda::Constant_int32(GPU_Vector& src, const int value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        Print_Error_Message(4, CUDA_NOT_INIT);
        decx::err::CUDA_Not_init(&handle);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != decx::_DATA_TYPES_FLAGS_::_INT32_) {
        Print_Error_Message(4, TYPE_ERROR_NOT_MATCH);
        decx::err::TypeError_NotMatch(&handle);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b32_caller((float*)_src->Vec.ptr, *((float*)&value), _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}




_DECX_API_ de::DH de::cuda::Constant_fp64(GPU_Vector& src, const double value)
{
    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        Print_Error_Message(4, CUDA_NOT_INIT);
        decx::err::CUDA_Not_init(&handle);
        return handle;
    }

    decx::err::Success(&handle);

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (_src->type != decx::_DATA_TYPES_FLAGS_::_FP64_) {
        Print_Error_Message(4, TYPE_ERROR_NOT_MATCH);
        decx::err::TypeError_NotMatch(&handle);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::bp::cu_fill1D_constant_v128_b64_caller((double*)_src->Vec.ptr, value, _src->length, S);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    return handle;
}