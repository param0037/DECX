#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "_mm128_fp32_int32.cuh"



__global__ void
decx::type_cast::GPUK::cu_mm128_cvtfp32_i321D(const float4* __restrict      src,
                                              int4* __restrict              dst, 
                                              const size_t                  proc_len)
{
    const uint tid = threadIdx.x + blockIdx.x * blockDim.x;

    decx::utils::_cuda_vec128 recv, store;

    if (tid < proc_len) {
        recv._vf = src[tid];
        
        store._vi.x = __float2int_rn(recv._vf.x);
        store._vi.y = __float2int_rn(recv._vf.y);
        store._vi.z = __float2int_rn(recv._vf.z);
        store._vi.w = __float2int_rn(recv._vf.w);
        
        dst[tid] = store._vi;
    }
}




__global__ void
decx::type_cast::GPUK::cu_mm128_cvti32_fp321D(const int4* __restrict        src,
                                              float4* __restrict            dst, 
                                              const size_t                  proc_len)
{
    const uint tid = threadIdx.x + blockIdx.x * blockDim.x;

    decx::utils::_cuda_vec128 recv, store;

    if (tid < proc_len) {
        recv._vi = src[tid];
        
        store._vf.x = __int2float_rn(recv._vi.x);
        store._vf.y = __int2float_rn(recv._vi.y);
        store._vf.z = __int2float_rn(recv._vi.z);
        store._vf.w = __int2float_rn(recv._vi.w);
        
        dst[tid] = store._vf;
    }
}





void 
decx::type_cast::_mm128_cvtfp32_i32_caller1D(const float4*           src, 
                                                int4*                dst, 
                                                const size_t            proc_len, 
                                                decx::hip_stream*      S)
{
    const uint block_length = decx::cuda::_get_cuda_prop().maxThreadsPerBlock;
    decx::type_cast::GPUK::cu_mm128_cvtfp32_i321D
        << <decx::utils::ceil<size_t>(proc_len, block_length), block_length, 0, S->get_raw_stream_ref() >> > (src, dst, proc_len);
}


void 
decx::type_cast::_mm128_cvti32_fp32_caller1D(const int4*            src, 
                                             float4*                dst, 
                                             const size_t           proc_len, 
                                             decx::hip_stream*     S)
{
    const uint block_length = decx::cuda::_get_cuda_prop().maxThreadsPerBlock;
    decx::type_cast::GPUK::cu_mm128_cvti32_fp321D
        << <decx::utils::ceil<size_t>(proc_len, block_length), block_length, 0, S->get_raw_stream_ref() >> > (src, dst, proc_len);
}

