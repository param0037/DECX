#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/

#include "../GEMM_kernels.cuh"
#include "MMA_FP32.cuh"



__global__ void decx::blas::GPUK::
cu_GEMM_fp32_kernel_32_128_128(const float* __restrict A,   const float* __restrict B, 
                            float* __restrict dst,       const uint2 proc_dims_v1, 
                            const uint32_t _L_v1,        const uint32_t pitchA_v1, 
                            const uint32_t pitchB_v1,    const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = 32 / 2;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = 32 / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t loc_tid_1d = threadIdx.x + threadIdx.y * blockDim.x;

    // Rearrange the 2D thread layout from 8x32 to 32x8 for LDG from A
    const uint32_t loc_tid_Ax = loc_tid_1d % _loc_LDG_Ax;
    const uint32_t loc_tid_Ay = loc_tid_1d / _loc_LDG_Ax;
    const uint32_t tid_Ay = loc_tid_Ay * 4 + blockIdx.y * 128;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t L_v2 = decx::utils::fast_uint_ceil2<uint32_t>(_L_v1);

    __shared__ float4 _frag_A[32][128 / 4 + 1];
    __shared__ float4 _frag_B[32][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = loc_tid_Ax;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = _Lloc_A * 2 + pitchA_v1 * tid_Ay;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, 32); ++i)
    {
        // Load from A
        if (_Lloc_A < L_v2){
#pragma unroll
            for (uint32_t k = 0; k < 4; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + k * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay] = _reg_aux._vf;
            }
#pragma unroll
            for (uint32_t k = 0; k < 4; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + 64 + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + (k+64) * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay + 16] = _reg_aux._vf;
            }
        }
        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < 32; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += 16;
        _Lloc_B += 32;

        dex_A += 32;
        dex_B += 32 * pitchB_v1;

        __syncthreads();
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y)  *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
        }
    }
}



__global__ void decx::blas::GPUK::
cu_GEMM_fp32_F_kernel_32_128_128(const float* __restrict A,       const float* __restrict B, 
                                 const float* __restrict C,       float* __restrict dst,       
                                 const float alpha,               const float beta, 
                                 const uint2 proc_dims_v1,        const uint32_t _L_v1,        
                                 const uint32_t pitchA_v1,        const uint32_t pitchB_v1,    
                                 const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = 32 / 2;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = 32 / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t loc_tid_1d = threadIdx.x + threadIdx.y * blockDim.x;

    // Rearrange the 2D thread layout from 8x32 to 32x8 for LDG from A
    const uint32_t loc_tid_Ax = loc_tid_1d % _loc_LDG_Ax;
    const uint32_t loc_tid_Ay = loc_tid_1d / _loc_LDG_Ax;
    const uint32_t tid_Ay = loc_tid_Ay * 4 + blockIdx.y * 128;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t L_v2 = decx::utils::fast_uint_ceil2<uint32_t>(_L_v1);

    __shared__ float4 _frag_A[32][128 / 4 + 1];
    __shared__ float4 _frag_B[32][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = loc_tid_Ax;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = _Lloc_A * 2 + pitchA_v1 * tid_Ay;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, 32); ++i)
    {
        // Load from A
        if (_Lloc_A < L_v2){
#pragma unroll
            for (uint32_t k = 0; k < 4; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + k * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay] = _reg_aux._vf;
            }
#pragma unroll
            for (uint32_t k = 0; k < 4; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + 64 + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + (k+64) * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay + 16] = _reg_aux._vf;
            }
        }
        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < 32; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += 16;
        _Lloc_B += 32;

        dex_A += 32;
        dex_B += 32 * pitchB_v1;

        __syncthreads();
    }

#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf.x = __fmul_rn(_accu[k]._vf.x, alpha);
        _accu[k]._vf.y = __fmul_rn(_accu[k]._vf.y, alpha);
        _accu[k]._vf.z = __fmul_rn(_accu[k]._vf.z, alpha);
        _accu[k]._vf.w = __fmul_rn(_accu[k]._vf.w, alpha);
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y) 
            {
                _reg_aux._vf = *((float4*)(dst + dex_dst + k * pitchdst_v1));
                _accu[k]._vf.x = __fmaf_rn(_reg_aux._vf.x, beta, _accu[k]._vf.x);
                _accu[k]._vf.y = __fmaf_rn(_reg_aux._vf.y, beta, _accu[k]._vf.y);
                _accu[k]._vf.z = __fmaf_rn(_reg_aux._vf.z, beta, _accu[k]._vf.z);
                _accu[k]._vf.w = __fmaf_rn(_reg_aux._vf.w, beta, _accu[k]._vf.w);

                *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
            }
        }
    }
}



__global__ void decx::blas::GPUK::
cu_GEMM_fp32_kernel_16_128_128(const float* __restrict A,   const float* __restrict B, 
                               float* __restrict dst,       const uint2 proc_dims_v1, 
                               const uint32_t _L_v1,        const uint32_t pitchA_v1, 
                               const uint32_t pitchB_v1,    const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = 16 / 2;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = 16 / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t loc_tid_1d = threadIdx.x + threadIdx.y * blockDim.x;

    // Rearrange the 2D thread layout from 8x32 to 32x8 for LDG from A
    const uint32_t loc_tid_Ax = loc_tid_1d % _loc_LDG_Ax;
    const uint32_t loc_tid_Ay = loc_tid_1d / _loc_LDG_Ax;
    const uint32_t tid_Ay = loc_tid_Ay * _LDG_HA_step + blockIdx.y * 128;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t L_v2 = decx::utils::fast_uint_ceil2<uint32_t>(_L_v1);

    __shared__ float4 _frag_A[16][128 / 4 + 1];
    __shared__ float4 _frag_B[16][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = loc_tid_Ax;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = _Lloc_A * 2 + pitchA_v1 * tid_Ay;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, 16); ++i)
    {
        // Load from A
        if (_Lloc_A < L_v2){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HA_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + k * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay] = _reg_aux._vf;
            }
        }
        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < 16; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += 4;
        _Lloc_B += 16;

        dex_A += 16;
        dex_B += 16 * pitchB_v1;

        __syncthreads();
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y)  *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
        }
    }
}



__global__ void decx::blas::GPUK::
cu_GEMM_fp32_F_kernel_16_128_128(const float* __restrict A,   const float* __restrict B, 
                                 const float* __restrict C,   float* __restrict dst,       
                                 const float alpha,           const float beta, 
                                 const uint2 proc_dims_v1,    const uint32_t _L_v1,        
                                 const uint32_t pitchA_v1,    const uint32_t pitchB_v1,    
                                 const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = 16 / 2;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = 16 / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t loc_tid_1d = threadIdx.x + threadIdx.y * blockDim.x;

    // Rearrange the 2D thread layout from 8x32 to 32x8 for LDG from A
    const uint32_t loc_tid_Ax = loc_tid_1d % _loc_LDG_Ax;
    const uint32_t loc_tid_Ay = loc_tid_1d / _loc_LDG_Ax;
    const uint32_t tid_Ay = loc_tid_Ay * _LDG_HA_step + blockIdx.y * 128;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t L_v2 = decx::utils::fast_uint_ceil2<uint32_t>(_L_v1);

    __shared__ float4 _frag_A[16][128 / 4 + 1];
    __shared__ float4 _frag_B[16][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = loc_tid_Ax;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = _Lloc_A * 2 + pitchA_v1 * tid_Ay;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, 16); ++i)
    {
        // Load from A
        if (_Lloc_A < L_v2){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HA_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (tid_Ay + k < proc_dims_v1.y) _regsA[k]._arrf2[0] = *((float2*)(A + dex_A + k * pitchA_v1));
            }
#pragma unroll
            for (uint32_t j = 0; j < 2; ++j){
                _reg_aux._arrf[0] = _regsA[0]._arrf[j];
                _reg_aux._arrf[1] = _regsA[1]._arrf[j];
                _reg_aux._arrf[2] = _regsA[2]._arrf[j];
                _reg_aux._arrf[3] = _regsA[3]._arrf[j];
                _frag_A[loc_tid_Ax * 2 + j][loc_tid_Ay] = _reg_aux._vf;
            }
        }
        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < 16; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += 4;
        _Lloc_B += 16;

        dex_A += 16;
        dex_B += 16 * pitchB_v1;

        __syncthreads();
    }

#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf.x = __fmul_rn(_accu[k]._vf.x, alpha);
        _accu[k]._vf.y = __fmul_rn(_accu[k]._vf.y, alpha);
        _accu[k]._vf.z = __fmul_rn(_accu[k]._vf.z, alpha);
        _accu[k]._vf.w = __fmul_rn(_accu[k]._vf.w, alpha);
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y) 
            {
                _reg_aux._vf = *((float4*)(C + dex_dst + k * pitchdst_v1));
                _accu[k]._vf.x = __fmaf_rn(_reg_aux._vf.x, beta, _accu[k]._vf.x);
                _accu[k]._vf.y = __fmaf_rn(_reg_aux._vf.y, beta, _accu[k]._vf.y);
                _accu[k]._vf.z = __fmaf_rn(_reg_aux._vf.z, beta, _accu[k]._vf.z);
                _accu[k]._vf.w = __fmaf_rn(_reg_aux._vf.w, beta, _accu[k]._vf.w);

                *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
            }  
        }
    }
}



template<uint32_t L>
__global__ void decx::blas::GPUK::
cu_GEMM_fp32_kernel_128_128_T(const float* __restrict A,   const float* __restrict B, 
                              float* __restrict dst,       const uint2 proc_dims_v1, 
                              const uint32_t _L_v1,        const uint32_t pitchA_v1, 
                              const uint32_t pitchB_v1,    const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = L / 4;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = L / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;
    const uint32_t tidx_A = threadIdx.x + blockIdx.y * blockDim.x;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t H_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.y, 4);

    __shared__ float4 _frag_A[L][128 / 4];
    __shared__ float4 _frag_B[L][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = threadIdx.y * _LDG_HA_step;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = tidx_A * 4 + pitchA_v1 * _Lloc_A;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, L); ++i)
    {
        // Load from A
        if (tidx_A < H_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HA_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_A + k < _L_v1) _regsA[k]._vf = *((float4*)(A + dex_A + k * pitchA_v1));
                _frag_A[threadIdx.y * _LDG_HA_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < L; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += L;
        _Lloc_B += L;

        dex_A += L * pitchA_v1;
        dex_B += L * pitchB_v1;

        __syncthreads();
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y)  *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
        }
    }
}

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_kernel_128_128_T<32>(const float* __restrict, const float* __restrict, float* __restrict,
    const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_kernel_128_128_T<16>(const float* __restrict, const float* __restrict, float* __restrict,
    const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_kernel_128_128_T<8>(const float* __restrict, const float* __restrict, float* __restrict,
    const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);



template<uint32_t L>
__global__ void decx::blas::GPUK::
cu_GEMM_fp32_F_kernel_128_128_T(const float* __restrict A,      const float* __restrict B, 
                                const float* __restrict C,      float* __restrict dst,       
                                const float alpha,              const float beta, 
                                const uint2 proc_dims_v1,       const uint32_t _L_v1,        
                                const uint32_t pitchA_v1,       const uint32_t pitchB_v1,    
                                const uint32_t pitchdst_v1)
{
    constexpr uint32_t _loc_LDG_Ax = L / 4;
    constexpr uint32_t _loc_LDG_Ay = 256 / _loc_LDG_Ax;
    constexpr uint32_t _LDG_HB_step = L / 8;
    constexpr uint32_t _LDG_HA_step = 128 / _loc_LDG_Ay;

    const uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;
    const uint32_t tidx_A = threadIdx.x + blockIdx.y * blockDim.x;

    const uint32_t W_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4);
    const uint32_t H_v4 = decx::utils::ceil<uint32_t>(proc_dims_v1.y, 4);

    __shared__ float4 _frag_A[L][128 / 4];
    __shared__ float4 _frag_B[L][128 / 4];

    decx::utils::_cuda_vec128 _accu[16];
    decx::utils::_cuda_vec128 _regsA[4], _reg_aux;

    uint32_t _Lloc_A = threadIdx.y * _LDG_HA_step;
    uint32_t _Lloc_B = threadIdx.y * _LDG_HB_step;
    
    uint64_t dex_A = tidx_A * 4 + pitchA_v1 * _Lloc_A;
    uint64_t dex_B = tidx * 4 + pitchB_v1 * _Lloc_B;

    // Initialize the accumulators to all zeros.
#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf = decx::utils::vec4_set1_fp32(0);
    }

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(_L_v1, L); ++i)
    {
        // Load from A
        if (tidx_A < H_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HA_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_A + k < _L_v1) _regsA[k]._vf = *((float4*)(A + dex_A + k * pitchA_v1));
                _frag_A[threadIdx.y * _LDG_HA_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        // Load from B
        if (tidx < W_v4){
#pragma unroll
            for (uint32_t k = 0; k < _LDG_HB_step; ++k) {
                _regsA[k]._vf = decx::utils::vec4_set1_fp32(0);
                if (_Lloc_B + k < _L_v1) _regsA[k]._vf = *((float4*)(B + dex_B + k * pitchB_v1));
                _frag_B[threadIdx.y * _LDG_HB_step + k][threadIdx.x] = _regsA[k]._vf;
            }
        }

        __syncthreads();

#pragma unroll
        for (uint32_t _l = 0; _l < L; ++_l)
        {
            _regsA[0]._vf = _frag_A[_l][threadIdx.y * 4];
            _regsA[1]._vf = _frag_A[_l][threadIdx.y * 4 + 1];
            _regsA[2]._vf = _frag_A[_l][threadIdx.y * 4 + 2];
            _regsA[3]._vf = _frag_A[_l][threadIdx.y * 4 + 3];

            _reg_aux._vf = _frag_B[_l][threadIdx.x];

            _MMA_FP32_1_4_16_(_regsA, _reg_aux, _accu);
        }

        _Lloc_A += L;
        _Lloc_B += L;

        dex_A += L * pitchA_v1;
        dex_B += L * pitchB_v1;

        __syncthreads();
    }

#pragma unroll 16
    for (uint32_t k = 0; k < 16; ++k){
        _accu[k]._vf.x = __fmul_rn(_accu[k]._vf.x, alpha);
        _accu[k]._vf.y = __fmul_rn(_accu[k]._vf.y, alpha);
        _accu[k]._vf.z = __fmul_rn(_accu[k]._vf.z, alpha);
        _accu[k]._vf.w = __fmul_rn(_accu[k]._vf.w, alpha);
    }

    // Store the results to dst.
    const uint64_t dex_dst = tidx * 4 + tidy * pitchdst_v1 * 16;

    if (tidx < decx::utils::ceil<uint32_t>(proc_dims_v1.x, 4))
    {
#pragma unroll 16
        for (uint32_t k = 0; k < 16; ++k) {
            if (tidy + k < proc_dims_v1.y) 
            {
                _reg_aux._vf = *((float4*)(C + dex_dst + k * pitchdst_v1));
                _accu[k]._vf.x = __fmaf_rn(_reg_aux._vf.x, beta, _accu[k]._vf.x);
                _accu[k]._vf.y = __fmaf_rn(_reg_aux._vf.y, beta, _accu[k]._vf.y);
                _accu[k]._vf.z = __fmaf_rn(_reg_aux._vf.z, beta, _accu[k]._vf.z);
                _accu[k]._vf.w = __fmaf_rn(_reg_aux._vf.w, beta, _accu[k]._vf.w);

                *((float4*)(dst + dex_dst + k * pitchdst_v1)) = _accu[k]._vf;
            }
        }
    }
}

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_F_kernel_128_128_T<32>(const float* __restrict, const float* __restrict, const float* __restrict, 
    float* __restrict, const float alpha, const float beta, const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_F_kernel_128_128_T<16>(const float* __restrict, const float* __restrict, const float* __restrict, 
    float* __restrict, const float alpha, const float beta, const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);

template __global__ void decx::blas::GPUK::cu_GEMM_fp32_F_kernel_128_128_T<8>(const float* __restrict, const float* __restrict, const float* __restrict, 
    float* __restrict, const float alpha, const float beta, const uint2, const uint32_t, const uint32_t, const uint32_t, const uint32_t);

