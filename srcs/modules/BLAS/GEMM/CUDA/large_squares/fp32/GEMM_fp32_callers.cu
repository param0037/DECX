#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/

#include "../common/cuda_GEMM_LS_planner.cuh"
#include "../fp32/GEMM_fp32_callers.cuh"
#include "../../../../../../common/Basic_process/transpose/CUDA/transpose_kernels.cuh"


decx::ResourceHandle decx::blas::g_cu_GEMM_fp32_planner;


template<>
decx::blas::CGKP decx::blas::cuda_GEMM_LS_planner<float>::_kernel_props[9] = 
{   // kernel ptr                                regs    shared  threads mio_compute     LWH                transposed_A
    {(void*)&decx::blas::GEMM_fp32_16_128_128,    114,    16640,  256,  1.5625e-2,  make_uint3(16, 128, 128),   0},
    {(void*)&decx::blas::GEMM_fp32_32_128_128,    114,    33280,  256,  1.5625e-2,  make_uint3(32, 128, 128),   0},
    {(void*)&decx::blas::GEMM_fp32_16_64_64,      64,     8448,   256,  3.1250e-2,  make_uint3(16, 64, 64),     0},
    {(void*)&decx::blas::GEMM_fp32_32_64_64,      64,     16896,  256,  3.1250e-2,  make_uint3(32, 64, 64),     0},
    // Kernels that require transposed form of matrix A
    {(void*)&decx::blas::GEMM_fp32_128_128_T<32>, 127,    32768,  256,  1.5625e-2,  make_uint3(32, 128, 128),   1},
    {(void*)&decx::blas::GEMM_fp32_128_128_T<16>, 123,    16384,  256,  1.5625e-2,  make_uint3(16, 128, 128),   1},
    {(void*)&decx::blas::GEMM_fp32_128_128_T<8>,  125,    8192,   256,  1.5625e-2,  make_uint3(8, 128, 128),    1},

    {(void*)&decx::blas::GEMM_fp32_64_64_T<32>,   63,     16384,  256,  3.1250e-2,  make_uint3(32, 64, 64),     1},
    {(void*)&decx::blas::GEMM_fp32_64_64_T<16>,   63,     8192,   256,  3.1250e-2,  make_uint3(16, 64, 64),     1}
};


template<> void decx::blas::cuda_GEMM_LS_planner<float>::
run(decx::_GPU_Matrix* A, decx::_GPU_Matrix* B, decx::_GPU_Matrix* dst, decx::hip_stream* S)
{   
    const auto* _k_prop_ptr = &decx::blas::cuda_GEMM_LS_planner<float>::_kernel_props[this->_kernel_id];

    auto* _kernel_ptr = (decx::blas::GPUK::_cu_GEMM_kernel_ptr<float>*)_k_prop_ptr->_kernel_ptr;

    if (_k_prop_ptr->_transpose_A){
        decx::blas::transpose2D_b4((float2*)A->Mat.ptr, 
                                   (float2*)this->_AT._ptr.ptr, 
                                   make_uint2(this->_A_layout.height, this->_A_layout.width), 
                                   this->_A_layout.pitch, 
                                   this->_AT._dims.x, 
                                   S);

        (*_kernel_ptr)(this->_AT._ptr.ptr,  B->Mat.ptr, 
                       dst->Mat.ptr,        make_uint2(dst->Width(), dst->Height()), 
                       A->Width(),          this->_AT._dims.x, 
                       B->Pitch(),          dst->Pitch(), 
                       S,                   NULL, 
                       1,                   1);
    }
    else{
        (*_kernel_ptr)(A->Mat.ptr,      B->Mat.ptr, 
                       dst->Mat.ptr,    make_uint2(dst->Width(), dst->Height()), 
                       A->Width(),      A->Pitch(), 
                       B->Pitch(),      dst->Pitch(), 
                       S,               NULL, 
                       1,               1);
    }
}



template<> void 
decx::blas::cuda_GEMM_LS_planner<float>::run(decx::_GPU_Matrix* A,     decx::_GPU_Matrix* B, 
                                          decx::_GPU_Matrix* C,     decx::_GPU_Matrix* dst,
                                          const float alpha,        const float beta, 
                                          decx::hip_stream* S)
{   
    const auto* _k_prop_ptr = &decx::blas::cuda_GEMM_LS_planner<float>::_kernel_props[this->_kernel_id];

    auto* _kernel_ptr = (decx::blas::GPUK::_cu_GEMM_kernel_ptr<float>*)_k_prop_ptr->_kernel_ptr;

    if (_k_prop_ptr->_transpose_A){
        decx::blas::transpose2D_b4((float2*)A->Mat.ptr, 
                                   (float2*)this->_AT._ptr.ptr, 
                                   make_uint2(this->_A_layout.height, this->_A_layout.width), 
                                   this->_A_layout.pitch, 
                                   this->_AT._dims.x, 
                                   S);

        (*_kernel_ptr)(this->_AT._ptr.ptr,  B->Mat.ptr, 
                       dst->Mat.ptr,        make_uint2(dst->Width(), dst->Height()), 
                       A->Width(),          this->_AT._dims.x, 
                       B->Pitch(),          dst->Pitch(), 
                       S,                   C->Mat.ptr, alpha, beta);
    }
    else{
        (*_kernel_ptr)(A->Mat.ptr,      B->Mat.ptr, 
                       dst->Mat.ptr,    make_uint2(dst->Width(), dst->Height()), 
                       A->Width(),      A->Pitch(), 
                       B->Pitch(),      dst->Pitch(), 
                       S,               C->Mat.ptr, 
                       alpha,           beta);
    }
}

