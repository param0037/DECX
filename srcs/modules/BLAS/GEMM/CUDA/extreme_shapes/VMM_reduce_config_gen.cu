#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "../../../Dot product/CUDA/DP2D_1way.cuh"
#include "VMM_callers.cuh"


template <bool _is_reduce_h>
void decx::blas::generate_VMM_config_fp32(decx::blas::cuda_DP2D_configs<float>* _configs, const uint2 proc_dims, decx::hip_stream* S)
{
    _configs->generate_config<_is_reduce_h>(proc_dims, S);

    if (decx::alloc::_device_malloc(&(_configs->_dev_A), _configs->_dev_mat_dims.x * _configs->_dev_mat_dims.y * sizeof(float), true, S)) {
        Print_Error_Message(4, DEV_ALLOC_FAIL);
        return;
    }
    if (decx::alloc::_device_malloc(&(_configs->_dev_B), decx::utils::align(_is_reduce_h ? proc_dims.x : proc_dims.y, _CU_REDUCE1D_MEM_ALIGN_4B_)
        * sizeof(float), true, S)) {
        Print_Error_Message(4, DEV_ALLOC_FAIL);
        return;
    }
    if (!_configs->postproc_needed()) {
        uint32_t _alloc_dst_size = 0;
        _alloc_dst_size = decx::utils::align<uint32_t>(_is_reduce_h ? proc_dims.y : proc_dims.x, _CU_REDUCE1D_MEM_ALIGN_4B_) * sizeof(float);

        if (decx::alloc::_device_malloc(&(_configs->_dev_dst), _alloc_dst_size, true, S)) {
            Print_Error_Message(4, DEV_ALLOC_FAIL);
            return;
        }
    }
}

template void decx::blas::generate_VMM_config_fp32<true>(decx::blas::cuda_DP2D_configs<float>*, const uint2, decx::hip_stream*);
template void decx::blas::generate_VMM_config_fp32<false>(decx::blas::cuda_DP2D_configs<float>*, const uint2, decx::hip_stream*);


template <bool _is_reduce_h>
void decx::blas::generate_VMM_config_fp16(decx::blas::cuda_DP2D_configs<de::Half>* _configs, const uint2 proc_dims, decx::hip_stream* S,
    const uint32_t _fp16_accu)
{
    _configs->generate_config<_is_reduce_h>(proc_dims, S, _fp16_accu);

    if (decx::alloc::_device_malloc(&(_configs->_dev_A), _configs->_dev_mat_dims.x * _configs->_dev_mat_dims.y * sizeof(de::Half), true, S)) {
        Print_Error_Message(4, DEV_ALLOC_FAIL);
        return;
    }
    if (decx::alloc::_device_malloc(&(_configs->_dev_B), decx::utils::align(_is_reduce_h ? proc_dims.x : proc_dims.y, _CU_REDUCE1D_MEM_ALIGN_4B_)
        * sizeof(de::Half), true, S)) {
        Print_Error_Message(4, DEV_ALLOC_FAIL);
        return;
    }
    if (!_configs->postproc_needed()) {
        uint32_t _alloc_dst_size = 0;
        if (_fp16_accu == decx::Fp16_Accuracy_Levels::Fp16_Accurate_L1) {
            _alloc_dst_size = decx::utils::align<uint32_t>(proc_dims.x, _CU_REDUCE1D_MEM_ALIGN_4B_) * sizeof(float);
        }
        else {
            _alloc_dst_size = decx::utils::align<uint32_t>(proc_dims.x, _CU_REDUCE1D_MEM_ALIGN_4B_) * sizeof(de::Half);
        }
        if (decx::alloc::_device_malloc(&_configs->_dev_dst, _alloc_dst_size, true, S)) {
            Print_Error_Message(4, DEV_ALLOC_FAIL);
            return;
        }
    }
}

template void decx::blas::generate_VMM_config_fp16<true>(decx::blas::cuda_DP2D_configs<de::Half>*, const uint2, decx::hip_stream*, const uint32_t);
template void decx::blas::generate_VMM_config_fp16<false>(decx::blas::cuda_DP2D_configs<de::Half>*, const uint2, decx::hip_stream*, const uint32_t);
