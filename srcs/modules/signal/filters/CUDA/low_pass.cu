#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "low_pass.cuh"


__global__ void
decx::signal::GPUK::cu_ideal_LP1D_cpl32(const float4* __restrict src, 
                                        float4* __restrict dst, 
                                        const size_t _proc_len, 
                                        const size_t real_bound,
                                        const size_t cutoff_freq)
{
    size_t dex = threadIdx.x + blockIdx.x * blockDim.x;

    bool _is_eff = false;
    de::CPf recv[2], store[2];

    if (dex < _proc_len) {
        *((float4*)recv) = src[dex];
        _is_eff = ((dex * 2) < cutoff_freq) || ((dex * 2) > (real_bound - cutoff_freq - 1));
        store[0] = _is_eff ? recv[0] : de::CPf(0, 0);
        
        _is_eff = ((dex * 2 + 1) < cutoff_freq) || ((dex * 2 + 1) > (real_bound - cutoff_freq - 1));
        store[1] = _is_eff ? recv[1] : de::CPf(0, 0);
        dst[dex] = *((float4*)store);
    }
}



__global__ void
decx::signal::GPUK::cu_ideal_LP2D_cpl32(const float4* __restrict src, 
                                        float4* __restrict dst, 
                                        const uint2 _proc_dims,         // in float4 (vec2 of datatype of de::CPf)
                                        const uint2 real_bound, 
                                        const uint2 cutoff_freq,
                                        const uint pitch)               // in float4 (vec2 of datatype of de::CPf)
{
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    uint idy = threadIdx.y + blockIdx.y * blockDim.y;
    size_t dex = 0;

    bool _is_effy_axis = false, _is_effx_axis = false;
    de::CPf recv[2], store[2];

    if (idx < _proc_dims.y && idy < _proc_dims.x) {
        dex = idx * pitch + idy;
        *((float4*)recv) = src[dex];
        _is_effy_axis = (idx < cutoff_freq.y) || (idx > (real_bound.y - cutoff_freq.y - 1));
        _is_effx_axis = ((idy * 2) < cutoff_freq.x) || ((idy * 2) > (real_bound.x - cutoff_freq.x - 1));

        store[0] = (_is_effy_axis && _is_effx_axis) ? recv[0] : de::CPf(0, 0);

        _is_effx_axis = ((idy * 2 + 1) < cutoff_freq.x) || ((idy * 2 + 1) > (real_bound.x - cutoff_freq.x - 1));
        store[1] = (_is_effy_axis && _is_effx_axis) ? recv[1] : de::CPf(0, 0);

        dst[dex] = *((float4*)store);
    }
}



_DECX_API_ de::DH 
de::signal::cuda::LowPass1D_Ideal(de::GPU_Vector& src, de::GPU_Vector& dst, const size_t cutoff_frequency)
{
    de::DH handle;
    if (!decx::cuP.is_init) {
        decx::err::CUDA_Not_init(&handle);
        Print_Error_Message(4, CUDA_NOT_INIT);
        return handle;
    }

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);
    decx::_GPU_Vector* _dst = dynamic_cast<decx::_GPU_Vector*>(&dst);

    const size_t max_freq = _src->length / 2;
    if (cutoff_frequency > max_freq) {
        Print_Error_Message(4, INVALID_PARAM);
        decx::err::InvalidParam(&handle);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }
    
    const size_t _proc_len_v2 = _src->_length / 2;
    decx::signal::GPUK::cu_ideal_LP1D_cpl32 << <decx::utils::ceil<size_t>(_proc_len_v2, decx::cuP.prop.maxThreadsPerBlock),
        decx::cuP.prop.maxThreadsPerBlock, 0, S->get_raw_stream_ref() >> > (
            (float4*)_src->Vec.ptr, (float4*)_dst->Vec.ptr, _proc_len_v2, _src->length, cutoff_frequency);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    decx::err::Success(&handle);
    return handle;
}




_DECX_API_ de::DH
de::signal::cuda::LowPass2D_Ideal(de::GPU_Matrix& src, de::GPU_Matrix& dst, const de::Point2D cutoff_frequency)
{
    de::DH handle;
    if (!decx::cuP.is_init) {
        decx::err::CUDA_Not_init(&handle);
        Print_Error_Message(4, CUDA_NOT_INIT);
        return handle;
    }

    decx::_GPU_Matrix* _src = dynamic_cast<decx::_GPU_Matrix*>(&src);
    decx::_GPU_Matrix* _dst = dynamic_cast<decx::_GPU_Matrix*>(&dst);

    const uint2 max_freq = make_uint2(_src->width / 2, _src->height / 2);
    if (cutoff_frequency.x > max_freq.x || cutoff_frequency.y > max_freq.y) {
        Print_Error_Message(4, INVALID_PARAM);
        decx::err::InvalidParam(&handle);
        return handle;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    const dim3 grid(decx::utils::ceil<uint>(_src->height, 16),
                    decx::utils::ceil<uint>(_src->pitch / 2, 16));
    const dim3 gpu_thread(16, 16);
    decx::signal::GPUK::cu_ideal_LP2D_cpl32 << <grid, gpu_thread, 0, S->get_raw_stream_ref() >> > (
            (float4*)_src->Mat.ptr, 
            (float4*)_dst->Mat.ptr, 
            make_uint2(_src->pitch / 2, _src->height), 
            make_uint2(_src->width, _src->height), 
            make_uint2(cutoff_frequency.x, cutoff_frequency.y), 
            _src->pitch / 2);

    checkCudaErrors(hipDeviceSynchronize());

    S->detach();

    decx::err::Success(&handle);
    return handle;
}