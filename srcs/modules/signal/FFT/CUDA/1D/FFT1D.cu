/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/


#include "../CUDA_FFT_configs.h"
#include "../../../../classes/Vector.h"
#include "../../../../core/configs/config.h"
#include "FFT1D_sub_functions.h"
#include "IFFT1D_sub_functions.h"
#include "../../fft_utils.h"


namespace decx {
    namespace signal {
        void FFT1D_R2C(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);


        void FFT1D_C2C(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);


        void IFFT1D_C2C(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);


        void IFFT1D_C2R(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);
    }
}


namespace de
{
    namespace signal
    {
        namespace cuda {
            _DECX_API_ de::DH FFT1D(de::Vector& src, de::Vector& dst, const int FFT_flag);


            _DECX_API_ de::DH IFFT1D(de::Vector& src, de::Vector& dst, const int FFT_flag);
        }
    }
}




void decx::signal::FFT1D_R2C(decx::_Vector* src, decx::_Vector* dst, de::DH* handle)
{
    const int src_len = src->length;

    if (!decx::signal::check_apart(src_len)) {
        decx::err::FFT_Error_length(handle);
        Print_Error_Message(4, FFT_ERROR_LENGTH);
        return;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(handle);
        return;
    }

    decx::signal::CUDA_FFT_Configs config;
    config.FFT1D_config_gen(src_len, handle);

    dst->re_construct(decx::_COMPLEX_F32_, src->length, decx::DATA_STORE_TYPE::Page_Locked);

    decx::signal::GPU_FFT1D_R2C_fp32_organizer(src, dst, &config, handle, S);
    S->detach();
}



void decx::signal::FFT1D_C2C(decx::_Vector* src, decx::_Vector* dst, de::DH *handle)
{
    const int src_len = src->length;

    if (!decx::signal::check_apart(src_len)) {
        decx::err::FFT_Error_length(handle);
        Print_Error_Message(4, FFT_ERROR_LENGTH);
        return;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(handle);
        return;
    }

    decx::signal::CUDA_FFT_Configs config;
    config.FFT1D_config_gen(src_len, handle);

    dst->re_construct(decx::_COMPLEX_F32_, src->length, decx::DATA_STORE_TYPE::Page_Locked);

    decx::signal::GPU_FFT1D_R2C_fp32_organizer(src, dst, &config, handle, S);
    S->detach();
}



void decx::signal::IFFT1D_C2C(decx::_Vector* src, decx::_Vector* dst, de::DH* handle)
{
    const int src_len = src->length;

    if (!decx::signal::check_apart(src_len)) {
        decx::err::FFT_Error_length(handle);
        Print_Error_Message(4, FFT_ERROR_LENGTH);
        return;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(handle);
        return;
    }

    decx::signal::CUDA_FFT_Configs config;
    config.FFT1D_config_gen(src_len, handle);

    dst->re_construct(decx::_COMPLEX_F32_, src->length, decx::DATA_STORE_TYPE::Page_Locked);

    decx::signal::GPU_IFFT1D_C2C_fp32_organizer(src, dst, &config, handle, S);
    S->detach();
}



void decx::signal::IFFT1D_C2R(decx::_Vector* src, decx::_Vector* dst, de::DH *handle)
{
    const int src_len = src->length;

    if (!decx::signal::check_apart(src_len)) {
        decx::err::FFT_Error_length(handle);
        Print_Error_Message(4, FFT_ERROR_LENGTH);
        return;
    }

    decx::hip_stream* S = NULL;
    S = decx::CStream.stream_accessor_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        Print_Error_Message(4, CUDA_STREAM_ACCESS_FAIL);
        decx::err::CUDA_Stream_access_fail(handle);
        return;
    }

    decx::signal::CUDA_FFT_Configs config;
    config.FFT1D_config_gen(src_len, handle);

    dst->re_construct(decx::_FP32_, src->length, decx::DATA_STORE_TYPE::Page_Locked);

    decx::signal::GPU_IFFT1D_C2R_fp32_organizer(src, dst, &config, handle, S);
    S->detach();
}



_DECX_API_ de::DH de::signal::cuda::FFT1D(de::Vector& src, de::Vector& dst, const int FFT_flag)
{
    de::DH handle;

    if (decx::cuP.is_init == false) {
        decx::Not_init(&handle);
        return handle;
    }

    decx::_Vector* _src = dynamic_cast<decx::_Vector*>(&src);
    decx::_Vector* _dst = dynamic_cast<decx::_Vector*>(&dst);

    switch (FFT_flag)
    {
    case de::signal::FFT_flags::FFT_R2C:
        decx::signal::FFT1D_R2C(_src, _dst, &handle);
        break;

    case de::signal::FFT_flags::FFT_C2C:
        decx::signal::FFT1D_R2C(_src, _dst, &handle);
        break;
    default:
        break;
    }

    decx::err::Success(&handle);
    return handle;
}



_DECX_API_ de::DH de::signal::cuda::IFFT1D(de::Vector& src, de::Vector& dst, const int FFT_flag)
{
    de::DH handle;

    if (decx::cuP.is_init == false) {
        decx::Not_init(&handle);
        return handle;
    }

    decx::_Vector* _src = dynamic_cast<decx::_Vector*>(&src);
    decx::_Vector* _dst = dynamic_cast<decx::_Vector*>(&dst);

    switch (FFT_flag)
    {
    case de::signal::FFT_flags::IFFT_C2R:
        decx::signal::IFFT1D_C2R(_src, _dst, &handle);
        break;

    case de::signal::FFT_flags::IFFT_C2C:
        decx::signal::FFT1D_C2C(_src, _dst, &handle);
        break;
    default:
        break;
    }

    decx::err::Success(&handle);
    return handle;
}
