#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "conv2_mk_im2col_fp16.h"


template <bool _print>
void decx::conv_I2R::conv2_BC_im2col_fp16(decx::_GPU_Tensor*         src,
                                      decx::_GPU_TensorArray*    kernel, 
                                      decx::_GPU_Tensor*         dst, 
                                      const int                  accu_flag, 
                                      decx::hip_stream*         S,
                                      decx::cuda_event*          E,
                                      de::DH*                    handle)
{
    const int2 kernel_shift = make_int2(8 - kernel->get_layout().height / 2, 8 - kernel->get_layout().width / 2);
    
    // the width and height of output tensor
    const uint4 dst_o_dim = make_uint4(src->get_layout().width, 
                                       src->get_layout().height,
                                       kernel->TensorNum(), 0);

    dst->re_construct(src->Type(), src->get_layout().width, src->get_layout().height, kernel->TensorNum());

    // the dimensions of kernel buffer, width : the number of active values in kernel, but dpitch included
    // height : the number of tensors 
    const int2 ker_buf_dim = make_int2(decx::utils::ceil<uint>(kernel->get_layout().plane[0] * (size_t)kernel->get_layout().dpitch, 128) * 128,
                                         decx::utils::ceil<uint>(kernel->TensorNum(), 64) * 64);

    // the dimension of the matrix after im2col operation
    const int2 eq_src_dims = make_int2(decx::utils::ceil<uint>(dst_o_dim.x, 8) * 8, dst_o_dim.y);

    // the dimensions of src buffer
    const ulong2 src_buf_dim = make_ulong2((decx::utils::ceil<size_t>(src->get_layout().width + kernel->get_layout().width - 1, 8) * 8) * (size_t)src->get_layout().dpitch,
                                           src->get_layout().height + kernel->get_layout().height - 1);
    
    const ulong2 I2C_dims = make_ulong2(eq_src_dims.x * eq_src_dims.y,
                                        kernel->get_layout().plane[0] * (size_t)kernel->get_layout().dpitch);

    const uint frag_num = decx::utils::ceil<size_t>(I2C_dims.x * I2C_dims.y, _I2C_size_fp16_);
    decx::utils::frag_manager f_mgr;
    decx::utils::frag_manager_gen(&f_mgr, dst_o_dim.y, frag_num);

    const uint I2C_alloc_width = decx::utils::ceil<uint32_t>(dst_o_dim.x, 8) * 8 * max(f_mgr.frag_len, f_mgr.frag_left_over);
    
    decx::PtrInfo<float4> src_buf, I2C_buf, ker_buf;
    if (decx::alloc::_device_malloc(&src_buf, src_buf_dim.x * src_buf_dim.y * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    if (decx::alloc::_device_malloc(&I2C_buf, I2C_dims.y * I2C_alloc_width * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    
    if (decx::alloc::_device_malloc(&ker_buf, ker_buf_dim.x * ker_buf_dim.y * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    
    // copy data from kernel(host) to kernel_buffer(device)
    for (int i = 0; i < kernel->TensorNum(); ++i) {
        hipMemcpy2DAsync(DECX_PTR_SHF_XY<float4, de::Half>(ker_buf.ptr, i, 0, ker_buf_dim.x),      kernel->get_layout().dpitch * kernel->get_layout().width * sizeof(de::Half),
                          kernel->TensptrArr.ptr[i],                                                kernel->get_layout().dp_x_wp * sizeof(de::Half),
                          kernel->get_layout().dpitch * kernel->get_layout().width * sizeof(de::Half),        kernel->get_layout().height,
                          hipMemcpyHostToDevice,                                                   S->get_raw_stream_ref());
    }

    // copy data from src(host) to src_buffer(device)
    checkCudaErrors(hipMemcpy2DAsync(
        reinterpret_cast<de::Half*>(src_buf.ptr) + (kernel->get_layout().height / 2) * src_buf_dim.x + (kernel->get_layout().width / 2) * src->get_layout().dpitch,
        src_buf_dim.x * sizeof(de::Half), 
        src->Tens.ptr,                                                          src->get_layout().dp_x_wp * sizeof(de::Half),
        src->get_layout().dp_x_wp * sizeof(de::Half),                                src->get_layout().height,
        hipMemcpyDeviceToDevice,                                               S->get_raw_stream_ref()));

    decx::conv_I2R::_conv2_I2C_params_set _params;
    _params.depth           = kernel->get_layout().dpitch;
    _params.ker_buf_dim     = ker_buf_dim;
    _params.ker_dims        = make_int2(kernel->get_layout().width, kernel->get_layout().height);
    _params.k_tensor_num    = kernel->TensorNum();
    _params.src_proc_H      = f_mgr.frag_len;
    _params.Wdst_eqMM       = decx::utils::ceil<uint>(kernel->TensorNum(), 8) * 8;
    _params.WI2C_buf        = I2C_alloc_width;
    _params.Wsrc_buf        = src_buf_dim.x;
    _params.Wdst_o          = decx::utils::ceil<uint>(dst_o_dim.x, 8) * 8;
    _params.HI2C_buf        = I2C_dims.y;

    float4* loc_src_buf = src_buf.ptr, * loc_dst = (float4*)dst->Tens.ptr;
    for (int i = 0; i < f_mgr.frag_num - 1; ++i)
    {
        decx::conv_I2R::conv2_MK_im2col_frag_fp16(loc_src_buf, ker_buf.ptr, I2C_buf.ptr, loc_dst, &_params, S, accu_flag);
        loc_src_buf += f_mgr.frag_len * _params.Wsrc_buf / 8;
        loc_dst += f_mgr.frag_len * _params.Wdst_o * dst->get_layout().dpitch / 8;
    }
    _params.src_proc_H = f_mgr.is_left ? f_mgr.frag_left_over : f_mgr.frag_len;
    decx::conv_I2R::conv2_MK_im2col_frag_fp16(loc_src_buf, ker_buf.ptr, I2C_buf.ptr, loc_dst, &_params, S, accu_flag);
    
    E->event_record(S);
    E->synchronize();

    decx::alloc::_device_dealloc(&src_buf);
    decx::alloc::_device_dealloc(&ker_buf);
    decx::alloc::_device_dealloc(&I2C_buf);
}


template void decx::conv_I2R::conv2_BC_im2col_fp16<true>(decx::_GPU_Tensor* src, decx::_GPU_TensorArray* kernel,
    decx::_GPU_Tensor* dst, const int accu_flag, decx::hip_stream* S, decx::cuda_event* E, de::DH* handle);


template void decx::conv_I2R::conv2_BC_im2col_fp16<true>(decx::_GPU_Tensor* src, decx::_GPU_TensorArray* kernel,
    decx::_GPU_Tensor* dst, const int accu_flag, decx::hip_stream* S, decx::cuda_event* E, de::DH* handle);




template <bool _print>
void decx::conv_I2R::conv2_BC_im2col_fp16_stride(decx::_GPU_Tensor*         src,
                                             decx::_GPU_TensorArray*    kernel, 
                                             decx::_GPU_Tensor*         dst, 
                                             const int                  accu_flag, 
                                             decx::hip_stream*         S,
                                             decx::cuda_event*          E,
                                             const uint2                strideXY,
                                             de::DH*                    handle)
{
    // the width and height of output tensor
    const uint4 dst_o_dim = make_uint4(src->get_layout().width / strideXY.x,
                                       src->get_layout().height / strideXY.y,
                                       kernel->TensorNum(), 0);
    
    dst->re_construct(src->Type(), dst_o_dim.x, dst_o_dim.y, kernel->TensorNum());

    // the dimensions of kernel buffer, width : the number of active values in kernel, but dpitch included
    // height : the number of tensors 
    const int2 ker_buf_dim = make_int2(decx::utils::ceil<uint>(kernel->get_layout().plane[0] * (size_t)kernel->get_layout().dpitch, 128) * 128,
                                         decx::utils::ceil<uint>(kernel->TensorNum(), 64) * 64);

    // the dimension of the matrix after im2col operation
    const int2 eq_src_dims = make_int2(decx::utils::ceil<uint>(dst_o_dim.x, 8) * 8, dst_o_dim.y);

    // the dimensions of src buffer
    const ulong2 src_buf_dim = make_ulong2((decx::utils::ceil<size_t>(src->get_layout().width + kernel->get_layout().width - 1, 8) * 8) * (size_t)src->get_layout().dpitch,
                                            src->get_layout().height + dst->get_layout().height - 1);
    
    const ulong2 I2C_dims = make_ulong2(eq_src_dims.x * eq_src_dims.y,
                                        kernel->get_layout().plane[0] * (size_t)kernel->get_layout().dpitch);

    const uint frag_num = decx::utils::ceil<size_t>(I2C_dims.x * I2C_dims.y, _I2C_size_fp16_);
    decx::utils::frag_manager f_mgr;
    decx::utils::frag_manager_gen(&f_mgr, dst_o_dim.y, frag_num);

    const uint I2C_alloc_width = decx::utils::ceil<uint32_t>(dst_o_dim.x, 8) * 8 * max(f_mgr.frag_len, f_mgr.frag_left_over);
    
    decx::PtrInfo<float4> src_buf, I2C_buf, ker_buf;
    if (decx::alloc::_device_malloc(&src_buf, src_buf_dim.x * src_buf_dim.y * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    if (decx::alloc::_device_malloc(&I2C_buf, I2C_dims.y * I2C_alloc_width * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    if (decx::alloc::_device_malloc(&ker_buf, ker_buf_dim.x * ker_buf_dim.y * sizeof(de::Half), true, S)) {
        decx::err::device_AllocateFailure<_print>(handle);
        return;
    }
    
    // copy data from kernel(host) to kernel_buffer(device)
    for (int i = 0; i < kernel->TensorNum(); ++i) {
        hipMemcpy2DAsync(DECX_PTR_SHF_XY<float4, de::Half>(ker_buf.ptr, i, 0, ker_buf_dim.x),      kernel->get_layout().dpitch * kernel->get_layout().width * sizeof(de::Half),
                          kernel->TensptrArr.ptr[i],                                                kernel->get_layout().dp_x_wp * sizeof(de::Half),
                          kernel->get_layout().dpitch * kernel->get_layout().width * sizeof(de::Half),        kernel->get_layout().height,
                          hipMemcpyHostToDevice,                                                   S->get_raw_stream_ref());
    }

    // copy data from src(host) to src_buffer(device)
    checkCudaErrors(hipMemcpy2DAsync(
        reinterpret_cast<de::Half*>(src_buf.ptr) + (kernel->get_layout().height / 2) * src_buf_dim.x + (kernel->get_layout().width / 2) * src->get_layout().dpitch,
        src_buf_dim.x * sizeof(de::Half),
        src->Tens.ptr,                                                                  src->get_layout().dp_x_wp * sizeof(de::Half),
        src->get_layout().dp_x_wp * sizeof(de::Half),                                        src->get_layout().height,
        hipMemcpyDeviceToDevice,                                                       S->get_raw_stream_ref()));

    decx::conv_I2R::_conv2_I2C_params_set _params;
    _params.depth           = kernel->get_layout().dpitch;
    _params.ker_buf_dim     = ker_buf_dim;
    _params.ker_dims        = make_int2(kernel->get_layout().width, kernel->get_layout().height);
    _params.k_tensor_num    = kernel->TensorNum();
    _params.src_proc_H      = f_mgr.frag_len;
    _params.Wdst_eqMM       = decx::utils::ceil<uint>(kernel->TensorNum(), 8) * 8;
    _params.WI2C_buf        = I2C_alloc_width;
    _params.Wsrc_buf        = src_buf_dim.x;
    _params.Wdst_o          = decx::utils::ceil<uint>(dst_o_dim.x, 8) * 8;
    _params.HI2C_buf        = I2C_dims.y;
    _params.strideXY        = strideXY;

    float4* loc_src_buf = src_buf.ptr, * loc_dst = (float4*)dst->Tens.ptr;
    for (int i = 0; i < f_mgr.frag_num - 1; ++i)
    {
        decx::conv_I2R::conv2_MK_im2col_frag_fp16_stride(loc_src_buf, ker_buf.ptr, I2C_buf.ptr, loc_dst, &_params, S, accu_flag);
        loc_src_buf += f_mgr.frag_len * _params.Wsrc_buf / 8 * strideXY.y;
        loc_dst     += f_mgr.frag_len * _params.Wdst_o * dst->get_layout().dpitch / 8;
    }
    _params.src_proc_H = f_mgr.is_left ? f_mgr.frag_left_over : f_mgr.frag_len;
    decx::conv_I2R::conv2_MK_im2col_frag_fp16_stride(loc_src_buf, ker_buf.ptr, I2C_buf.ptr, loc_dst, &_params, S, accu_flag);
    
    E->event_record(S);
    E->synchronize();

    decx::alloc::_device_dealloc(&src_buf);
    decx::alloc::_device_dealloc(&ker_buf);
    decx::alloc::_device_dealloc(&I2C_buf);
}


template void decx::conv_I2R::conv2_BC_im2col_fp16_stride<true>(decx::_GPU_Tensor* src, decx::_GPU_TensorArray* kernel,
    decx::_GPU_Tensor* dst, const int accu_flag, decx::hip_stream* S, decx::cuda_event* E, const uint2 strideXY, de::DH* handle);


template void decx::conv_I2R::conv2_BC_im2col_fp16_stride<true>(decx::_GPU_Tensor* src, decx::_GPU_TensorArray* kernel,
    decx::_GPU_Tensor* dst, const int accu_flag, decx::hip_stream* S, decx::cuda_event* E, const uint2 strideXY, de::DH* handle);