#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "im2col_fp32.cuh"
#include "../../../../core/utils/decx_cuda_vectypes_ops.cuh"


__global__ void 
decx::nn::GPUK::cu_im2col_DP4_NB_fp32(const float4* __restrict  src, 
                                      float4* __restrict        dst, 
                                      const uint2               conv2D_area, 
                                      const uint2               kernel_dims,
                                      const uint2               strides,
                                      const uint32_t            wpitch_dst_v1, 
                                      const uint32_t            wpitch_src_v1, 
                                      const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 4);
    uint64_t dex_src = 0;

    const uint32_t dex_plane_src_x = (threadIdx.x + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    const uint8_t STG_threadIdx_x = (threadIdx.x + blockDim.x * threadIdx.y) % _CUDA_WARP_SIZE_;
    const uint8_t STG_threadIdx_y = threadIdx.x / _CUDA_WARP_SIZE_;

    const uint32_t dex_plane_dst_x = STG_threadIdx_x + blockIdx.x * _CUDA_WARP_SIZE_;
    uint32_t dex_plane_dst_y = STG_threadIdx_y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 4;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 4];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        dex_src = dex_plane_src_x * strides.x + (blockIdx.z + dex_plane_src_y * strides.y + i) * wpitch_src_v1;
        for (uint32_t j = 0; j < kernel_dims.x; ++j) 
        {
            _reg._vf = decx::utils::vec4_set1_fp32(0);

            if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                _reg._vf = src[dex_src];
            }

            _shmem[threadIdx.y * 4 + 0][threadIdx.x] = _reg._arrf[0];
            _shmem[threadIdx.y * 4 + 1][threadIdx.x] = _reg._arrf[1];
            _shmem[threadIdx.y * 4 + 2][threadIdx.x] = _reg._arrf[2];
            _shmem[threadIdx.y * 4 + 3][threadIdx.x] = _reg._arrf[3];

            __syncthreads();

            _reg._vf = ((float4*)_shmem[threadIdx.y * 4 + STG_threadIdx_y])[STG_threadIdx_x];

            if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 4) * im2col_buf_pitch_v1 / 4] = _reg._vf;
            }
            dex_plane_dst_y += 4;
            dex_src += 1;

            __syncthreads();
        }
    }
}


template <bool _boundless_T, bool _boundless_B> __global__ void
decx::nn::GPUK::cu_im2col_DP4_BC_fp32(const float4* __restrict  src, 
                                      float4* __restrict        dst, 
                                      const uint2               conv2D_area, 
                                      const uint2               kernel_dims,
                                      const uint2               strides,
                                      const uint32_t            wpitch_dst_v1, 
                                      const uint32_t            wpitch_src_v1, 
                                      const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 4);
    int64_t dex_src = 0;

    const uint32_t dex_plane_src_x = (threadIdx.x + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    const uint8_t STG_threadIdx_x = (threadIdx.x + blockDim.x * threadIdx.y) % _CUDA_WARP_SIZE_;
    const uint8_t STG_threadIdx_y = threadIdx.x / _CUDA_WARP_SIZE_;

    const uint32_t dex_plane_dst_x = STG_threadIdx_x + blockIdx.x * _CUDA_WARP_SIZE_;
    uint32_t dex_plane_dst_y = STG_threadIdx_y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t& _half_KH = (kernel_dims.y / 2);

    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 4;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 4];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        const uint32_t _global_coor_y = blockIdx.z + dex_plane_src_y * strides.y + i;
        
        if ((_boundless_T || _global_coor_y > _half_KH-1) && (_boundless_B || _global_coor_y < conv2D_area.y * strides.y + _half_KH))
        {
            dex_src = dex_plane_src_x * strides.x + _global_coor_y * wpitch_src_v1;
            
            for (uint32_t j = 0; j < kernel_dims.x; ++j)
            {
                _reg._vf = decx::utils::vec4_set1_fp32(0);
                if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                    _reg._vf = src[dex_src];
                }

                _shmem[threadIdx.y * 4 + 0][threadIdx.x] = _reg._arrf[0];
                _shmem[threadIdx.y * 4 + 1][threadIdx.x] = _reg._arrf[1];
                _shmem[threadIdx.y * 4 + 2][threadIdx.x] = _reg._arrf[2];
                _shmem[threadIdx.y * 4 + 3][threadIdx.x] = _reg._arrf[3];

                __syncthreads();

                _reg._vf = ((float4*)_shmem[threadIdx.y * 4 + STG_threadIdx_y])[STG_threadIdx_x];

                if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 4) * im2col_buf_pitch_v1 / 4] = _reg._vf;
                }
                dex_plane_dst_y += 4;
                dex_src += 1;

                __syncthreads();
            }   // end for
        }   // end if
    }
}


template __global__ void decx::nn::GPUK::cu_im2col_DP4_BC_fp32<false, false>(const float4* __restrict, float4* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP4_BC_fp32<true, false>(const float4* __restrict, float4* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP4_BC_fp32<false, true>(const float4* __restrict, float4* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);



// block[32 * 4, 2] = [128, 2]
__global__ void 
decx::nn::GPUK::cu_im2col_DP8_NB_fp32(const float4* __restrict  src, 
                                      float2* __restrict        dst, 
                                      const uint2               conv2D_area, 
                                      const uint2               kernel_dims,
                                      const uint2               strides,
                                      const uint32_t            wpitch_dst_v1, 
                                      const uint32_t            wpitch_src_v1, 
                                      const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 8);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 2, threadIdx.x / 2);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 2) % _LDG_blockDim_x, threadIdx.x % 2);
    const uchar2 _logical_stgl = make_uchar2(threadIdx.x % _CUDA_WARP_SIZE_, threadIdx.x / _CUDA_WARP_SIZE_);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 2;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 2];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + (blockIdx.z + dex_plane_src_y * strides.y + i) * wpitch_src_v1) * 2;
        for (uint32_t j = 0; j < kernel_dims.x; ++j) 
        {
            _reg._vf = decx::utils::vec4_set1_fp32(0);

            if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                _reg._vf = src[dex_src];
            }

            _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[0];
            _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[1];
            _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[2];
            _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[3];

            __syncthreads();

            if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
            {
                _reg._arrf2[0] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.y])[_logical_stgl.x];
                dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 8) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[0];
                
                dex_plane_dst_y += 4;

                _reg._arrf2[1] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.y])[_logical_stgl.x + _CUDA_WARP_SIZE_];
                dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 8) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[1];
                dex_plane_dst_y += 4;
            }

            dex_src += 2;

            __syncthreads();
        }
    }
}


// block[32 * 4, 2] = [128, 2]
template <bool _boundless_T, bool _boundless_B> __global__ void
decx::nn::GPUK::cu_im2col_DP8_BC_fp32(const float4* __restrict  src, 
                                      float2* __restrict        dst, 
                                      const uint2               conv2D_area, 
                                      const uint2               kernel_dims,
                                      const uint2               strides,
                                      const uint32_t            wpitch_dst_v1, 
                                      const uint32_t            wpitch_src_v1, 
                                      const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 8);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 2, threadIdx.x / 2);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 2) % _LDG_blockDim_x, threadIdx.x % 2);
    const uchar2 _logical_stgl = make_uchar2(threadIdx.x % _CUDA_WARP_SIZE_, threadIdx.x / _CUDA_WARP_SIZE_);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t& _half_KH = (kernel_dims.y / 2);

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 2;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 2];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        const uint32_t _global_coor_y = blockIdx.z + dex_plane_src_y * strides.y + i;
        
        if ((_boundless_T || _global_coor_y > _half_KH - 1) && (_boundless_B || _global_coor_y < conv2D_area.y * strides.y + _half_KH))
        {
            dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + _global_coor_y * wpitch_src_v1) * 2;

            for (uint32_t j = 0; j < kernel_dims.x; ++j)
            {
                _reg._vf = decx::utils::vec4_set1_fp32(0);
                if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                    _reg._vf = src[dex_src];
                }

                _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[0];
                _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[1];
                _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[2];
                _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[3];

                __syncthreads();

                if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
                {
                    _reg._arrf2[0] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.y])[_logical_stgl.x];
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 8) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[0];

                    dex_plane_dst_y += 4;

                    _reg._arrf2[1] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.y])[_logical_stgl.x + _CUDA_WARP_SIZE_];
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 8) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[1];
                }

                dex_plane_dst_y += 4;
                dex_src += 2;

                __syncthreads();
            }   // end for
        }   // end if
    }
}

template __global__ void decx::nn::GPUK::cu_im2col_DP8_BC_fp32<false, false>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP8_BC_fp32<true, false>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP8_BC_fp32<false, true>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);




// block[32 * 6, 2] = [128, 2]
__global__ void 
decx::nn::GPUK::cu_im2col_DP12_NB_fp32(const float4* __restrict  src, 
                                      float2* __restrict        dst, 
                                      const uint2               conv2D_area, 
                                      const uint2               kernel_dims,
                                      const uint2               strides,
                                      const uint32_t            wpitch_dst_v1, 
                                      const uint32_t            wpitch_src_v1, 
                                      const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D12_FP32_BLOCK_X_, 12);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 3, threadIdx.x / 3);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 3) % _LDG_blockDim_x, threadIdx.x % 3);
    const uchar3 _logical_stgl = make_uchar3(threadIdx.x % _CUDA_WARP_SIZE_, (threadIdx.x / _CUDA_WARP_SIZE_) % 3, (threadIdx.x / _CUDA_WARP_SIZE_) / 3);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y + _logical_stgl.z * 3;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 2;       // /2 because of float2
    
    __shared__ float _shmem[_IM2COL_D12_FP32_BLOCK_Y_ * 4][_IM2COL_D12_FP32_BLOCK_X_ + 2];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + (blockIdx.z + dex_plane_src_y * strides.y + i) * wpitch_src_v1) * 3;
        for (uint32_t j = 0; j < kernel_dims.x; ++j) 
        {
            _reg._vf = decx::utils::vec4_set1_fp32(0);

            if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                _reg._vf = src[dex_src];
            }

            _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[0];
            _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[1];
            _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[2];
            _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[3];

            __syncthreads();

            if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
            {
                _reg._arrf2[0] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.z])[_logical_stgl.x + _CUDA_WARP_SIZE_ * _logical_stgl.y];
                dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 12) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[0];
                
                dex_plane_dst_y += 6;

                _reg._arrf2[1] = ((float2*)_shmem[threadIdx.y * 4 + 2 + _logical_stgl.z])[_logical_stgl.x + _CUDA_WARP_SIZE_ * _logical_stgl.y];
                dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 12) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[1];
                dex_plane_dst_y += 6;
            }

            dex_src += 3;

            __syncthreads();
        }
    }
}



// block[32 * 4, 2] = [128, 2]
template <bool _boundless_T, bool _boundless_B> __global__ void
decx::nn::GPUK::cu_im2col_DP12_BC_fp32(const float4* __restrict  src, 
                                       float2* __restrict        dst, 
                                       const uint2               conv2D_area, 
                                       const uint2               kernel_dims,
                                       const uint2               strides,
                                       const uint32_t            wpitch_dst_v1, 
                                       const uint32_t            wpitch_src_v1, 
                                       const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D12_FP32_BLOCK_X_, 12);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 3, threadIdx.x / 3);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 3) % _LDG_blockDim_x, threadIdx.x % 3);
    const uchar3 _logical_stgl = make_uchar3(threadIdx.x % _CUDA_WARP_SIZE_, (threadIdx.x / _CUDA_WARP_SIZE_) % 3, (threadIdx.x / _CUDA_WARP_SIZE_) / 3);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y + _logical_stgl.z * 3;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t& _half_KH = (kernel_dims.y / 2);

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1 / 2;
    
    __shared__ float _shmem[_IM2COL_D12_FP32_BLOCK_Y_ * 4][_IM2COL_D12_FP32_BLOCK_X_ + 2];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        const uint32_t _global_coor_y = blockIdx.z + dex_plane_src_y * strides.y + i;
        
        if ((_boundless_T || _global_coor_y > _half_KH - 1) && (_boundless_B || _global_coor_y < conv2D_area.y * strides.y + _half_KH))
        {
            dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + _global_coor_y * wpitch_src_v1) * 3;

            for (uint32_t j = 0; j < kernel_dims.x; ++j)
            {
                _reg._vf = decx::utils::vec4_set1_fp32(0);
                if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                    _reg._vf = src[dex_src];
                }

                _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[0];
                _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[1];
                _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[2];
                _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 64 * _logical_gl2shmem.y] = _reg._arrf[3];

                __syncthreads();

                if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
                {
                    _reg._arrf2[0] = ((float2*)_shmem[threadIdx.y * 4 + _logical_stgl.z])[_logical_stgl.x + _CUDA_WARP_SIZE_ * _logical_stgl.y];
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 12) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[0];

                    dex_plane_dst_y += 6;

                    _reg._arrf2[1] = ((float2*)_shmem[threadIdx.y * 4 + 2 + _logical_stgl.z])[_logical_stgl.x + _CUDA_WARP_SIZE_ * _logical_stgl.y];
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 12) * im2col_buf_pitch_v1 / 2] = _reg._arrf2[1];
                }

                dex_plane_dst_y += 6;
                dex_src += 3;

                __syncthreads();
            }   // end for
        }   // end if
    }
}

template __global__ void decx::nn::GPUK::cu_im2col_DP12_BC_fp32<false, false>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP12_BC_fp32<true, false>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP12_BC_fp32<false, true>(const float4* __restrict, float2* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);



// block[32 * 4, 2] = [128, 2]
__global__ void 
decx::nn::GPUK::cu_im2col_DP16_NB_fp32(const float4* __restrict  src, 
                                       float* __restrict         dst, 
                                       const uint2               conv2D_area, 
                                       const uint2               kernel_dims,
                                       const uint2               strides,
                                       const uint32_t            wpitch_dst_v1, 
                                       const uint32_t            wpitch_src_v1, 
                                       const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 16);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 4, threadIdx.x / 4);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 4) % _LDG_blockDim_x, threadIdx.x % 4);
    const uchar2 _logical_stgl = make_uchar2(threadIdx.x % _CUDA_WARP_SIZE_, threadIdx.x / _CUDA_WARP_SIZE_);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 1];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + (blockIdx.z + dex_plane_src_y * strides.y + i) * wpitch_src_v1) * 4;
        for (uint32_t j = 0; j < kernel_dims.x; ++j) 
        {
            _reg._vf = decx::utils::vec4_set1_fp32(0);

            if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                _reg._vf = src[dex_src];
            }

            _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[0];
            _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[1];
            _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[2];
            _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[3];

            __syncthreads();

            if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
            {
#pragma unroll 4
                for (int k = 0; k < 4; ++k) {
                    _reg._arrf[k] = _shmem[threadIdx.y * 4 + _logical_stgl.y][_logical_stgl.x + _CUDA_WARP_SIZE_ * k];
                    
                    dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 16) * im2col_buf_pitch_v1] = _reg._arrf[k];

                    dex_plane_dst_y += 4;
                }
            }
            
            dex_src += 4;

            __syncthreads();
        }
    }
}




// block[32 * 4, 2] = [128, 2]
template <bool _boundless_T, bool _boundless_B> __global__ void
decx::nn::GPUK::cu_im2col_DP16_BC_fp32(const float4* __restrict  src, 
                                       float* __restrict         dst, 
                                       const uint2               conv2D_area, 
                                       const uint2               kernel_dims,
                                       const uint2               strides,
                                       const uint32_t            wpitch_dst_v1, 
                                       const uint32_t            wpitch_src_v1, 
                                       const uint64_t            im2col_buf_pitch_v1)
{
    constexpr uint32_t _LDG_blockDim_x = _IM2COL_GET_STG_BLOCKDIM_X_(_IM2COL_D4N_FP32_BLOCK_X_, 16);

    uint64_t dex_src = 0;

    const uchar2 _logical_ldgl = make_uchar2(threadIdx.x % 4, threadIdx.x / 4);
    const uchar2 _logical_gl2shmem = make_uchar2((threadIdx.x / 4) % _LDG_blockDim_x, threadIdx.x % 4);
    const uchar2 _logical_stgl = make_uchar2(threadIdx.x % _CUDA_WARP_SIZE_, threadIdx.x / _CUDA_WARP_SIZE_);

    const uint32_t dex_plane_src_x = (_logical_ldgl.y + blockIdx.x * _LDG_blockDim_x);
    const uint32_t dex_plane_src_y = (threadIdx.y + blockIdx.y * blockDim.y);

    uint32_t dex_plane_dst_y = _logical_stgl.y;

    decx::utils::_cuda_vec128 _reg;

    const uint32_t& _half_KH = (kernel_dims.y / 2);

    const uint32_t dex_plane_dst_x = _logical_stgl.x + _CUDA_WARP_SIZE_ * blockIdx.x;
    const uint32_t STG_dex_x = dex_plane_dst_x + dex_plane_src_y * wpitch_dst_v1;
    
    __shared__ float _shmem[_IM2COL_D4N_FP32_BLOCK_Y_ * 4][_IM2COL_D4N_FP32_BLOCK_X_ + 1];

    for (uint32_t i = 0; i < decx::utils::ceil<uint32_t>(kernel_dims.y, gridDim.z); ++i) 
    {
        const uint32_t _global_coor_y = blockIdx.z + dex_plane_src_y * strides.y + i;
        
        if ((_boundless_T || _global_coor_y > _half_KH - 1) && (_boundless_B || _global_coor_y < conv2D_area.y * strides.y + _half_KH))
        {
            dex_src = _logical_ldgl.x + (dex_plane_src_x * strides.x + _global_coor_y * wpitch_src_v1) * 4;

            for (uint32_t j = 0; j < kernel_dims.x; ++j)
            {
                _reg._vf = decx::utils::vec4_set1_fp32(0);

                if (dex_plane_src_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y) {
                    _reg._vf = src[dex_src];
                }

                _shmem[threadIdx.y * 4 + 0][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[0];
                _shmem[threadIdx.y * 4 + 1][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[1];
                _shmem[threadIdx.y * 4 + 2][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[2];
                _shmem[threadIdx.y * 4 + 3][_logical_gl2shmem.x + 32 * _logical_gl2shmem.y] = _reg._arrf[3];

                __syncthreads();

                if (dex_plane_dst_x < conv2D_area.x && dex_plane_src_y < conv2D_area.y)
                {
#pragma unroll 4
                    for (int k = 0; k < 4; ++k) {
                        _reg._arrf[k] = _shmem[threadIdx.y * 4 + _logical_stgl.y][_logical_stgl.x + _CUDA_WARP_SIZE_ * k];

                        dst[STG_dex_x + (dex_plane_dst_y + blockIdx.z * kernel_dims.x * 16) * im2col_buf_pitch_v1] = _reg._arrf[k];

                        dex_plane_dst_y += 4;
                    }
                }

                dex_src += 4;

                __syncthreads();
            }   // end for
        }   // end if
    }
}

template __global__ void decx::nn::GPUK::cu_im2col_DP16_BC_fp32<false, false>(const float4* __restrict, float* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP16_BC_fp32<true, false>(const float4* __restrict, float* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::nn::GPUK::cu_im2col_DP16_BC_fp32<false, true>(const float4* __restrict, float* __restrict, const uint2, const uint2,
    const uint2, const uint32_t, const uint32_t, const uint64_t);
