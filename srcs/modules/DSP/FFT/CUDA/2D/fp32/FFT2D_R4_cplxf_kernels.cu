#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../FFT2D_kernels.cuh"


// [32 * 2, 8] = [64, 8]
__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_1st_R2C_cplxf(const float2* __restrict src,
                                               float4* __restrict dst,
                                               const uint32_t _signal_len,
                                               const uint32_t _pitchsrc_v2,
                                               const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 4;

    decx::utils::_cuda_vec64 recv[4];
    decx::utils::_cuda_vec128 res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            recv[i]._vf2 = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 4;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        // 1st
        res._vf = decx::utils::vec4_set1_fp32(0.f);
        res._arrcplxf2[0].real = __fadd_rn(__fadd_rn(recv[0]._vf2.x, recv[1]._vf2.x),
                                              __fadd_rn(recv[2]._vf2.x, recv[3]._vf2.x));
        res._arrcplxf2[1].real = __fadd_rn(__fadd_rn(recv[0]._vf2.y, recv[1]._vf2.y),
                                              __fadd_rn(recv[2]._vf2.y, recv[3]._vf2.y));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 2nd
        res._arrcplxf2[0].real = __fsub_rn(recv[0]._vf2.x, recv[2]._vf2.x);
        res._arrcplxf2[1].real = __fsub_rn(recv[0]._vf2.y, recv[2]._vf2.y);

        res._arrcplxf2[0].image = __fsub_rn(recv[1]._vf2.x, recv[3]._vf2.x);
        res._arrcplxf2[1].image = __fsub_rn(recv[1]._vf2.y, recv[3]._vf2.y);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 3rd
        res._vf = decx::utils::vec4_set1_fp32(0.f);
        res._arrcplxf2[0].real = __fadd_rn(__fsub_rn(recv[0]._vf2.x, recv[1]._vf2.x), __fsub_rn(recv[2]._vf2.x, recv[3]._vf2.x));
        res._arrcplxf2[1].real = __fadd_rn(__fsub_rn(recv[0]._vf2.y, recv[1]._vf2.y), __fsub_rn(recv[2]._vf2.y, recv[3]._vf2.y));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 4th 
        res._arrcplxf2[0].real = __fsub_rn(recv[0]._vf2.x, recv[2]._vf2.x);
        res._arrcplxf2[1].real = __fsub_rn(recv[0]._vf2.y, recv[2]._vf2.y);

        res._arrcplxf2[0].image = __fsub_rn(recv[3]._vf2.x, recv[1]._vf2.x);
        res._arrcplxf2[1].image = __fsub_rn(recv[3]._vf2.y, recv[1]._vf2.y);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}



// [32 * 2, 8] = [64, 8]
__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_1st_R2C_uc8_cplxf(const ushort* __restrict src,
                                               float4* __restrict dst,
                                               const uint32_t _signal_len,
                                               const uint32_t _pitchsrc_v2,
                                               const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 4;

    uchar2 recv[4];
    decx::utils::_cuda_vec128 res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            *((ushort*)&recv[i]) = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 4;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        // 1st
        res._vf = decx::utils::vec4_set1_fp32(0.f);
        res._arrcplxf2[0].real = __fadd_rn(__fadd_rn(recv[0].x, recv[1].x),
                                              __fadd_rn(recv[2].x, recv[3].x));
        res._arrcplxf2[1].real = __fadd_rn(__fadd_rn(recv[0].y, recv[1].y),
                                              __fadd_rn(recv[2].y, recv[3].y));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 2nd
        res._arrcplxf2[0].real = __fsub_rn(recv[0].x, recv[2].x);
        res._arrcplxf2[1].real = __fsub_rn(recv[0].y, recv[2].y);

        res._arrcplxf2[0].image = __fsub_rn(recv[1].x, recv[3].x);
        res._arrcplxf2[1].image = __fsub_rn(recv[1].y, recv[3].y);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 3rd
        res._vf = decx::utils::vec4_set1_fp32(0.f);
        res._arrcplxf2[0].real = __fadd_rn(__fsub_rn(recv[0].x, recv[1].x), __fsub_rn(recv[2].x, recv[3].x));
        res._arrcplxf2[1].real = __fadd_rn(__fsub_rn(recv[0].y, recv[1].y), __fsub_rn(recv[2].y, recv[3].y));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        // 4th 
        res._arrcplxf2[0].real = __fsub_rn(recv[0].x, recv[2].x);
        res._arrcplxf2[1].real = __fsub_rn(recv[0].y, recv[2].y);

        res._arrcplxf2[0].image = __fsub_rn(recv[3].x, recv[1].x);
        res._arrcplxf2[1].image = __fsub_rn(recv[3].y, recv[1].y);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}



template<bool _div> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_1st_C2C_cplxf(const float4* __restrict src,
                                               float4* __restrict dst,
                                               const uint32_t _signal_len,
                                               const uint32_t _pitchsrc_v2,
                                               const uint32_t _pitchdst_v2,
                                               const uint64_t _div_length)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 4;

    decx::utils::_cuda_vec128 recv[4];
    decx::utils::_cuda_vec128 res, tmp1, tmp2;

    const float _numer = __ull2float_rn(_div_length ? _div_length : _signal_len);
    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            recv[i]._vf = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            if (_div) { recv[i]._vf = decx::utils::cuda::__float_div4_1(recv[i]._vf, _numer); }
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 4;

    if (tidy < _Bops_num && tidx < _pitchdst_v2) 
    {
        // Calculate the first and third output
        tmp1._vf = decx::utils::cuda::__float_add4(recv[0]._vf, recv[2]._vf);
        tmp2._vf = decx::utils::cuda::__float_add4(recv[1]._vf, recv[3]._vf);

        // Store the first output
        res._vf = decx::utils::cuda::__float_add4(tmp1._vf, tmp2._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += 2;
        
        // Store the third output
        res._vf = decx::utils::cuda::__float_sub4(tmp1._vf, tmp2._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        --_FFT_domain_dex;

        // Calculate and store the second output
        res._arrcplxf2[0].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image), 
                                           __fadd_rn(recv[1]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].real));
        res._arrcplxf2[0].image = __fadd_rn(__fsub_rn(recv[0]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].image), 
                                            __fsub_rn(recv[1]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].real));
        res._arrcplxf2[1].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image), 
                                           __fadd_rn(recv[1]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].real));
        res._arrcplxf2[1].image = __fadd_rn(__fsub_rn(recv[0]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].image), 
                                            __fsub_rn(recv[1]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].real));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += 2;

        res._arrcplxf2[0].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[1]._arrcplxf2[0].image), 
                                           __fadd_rn(recv[2]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image));
        res._arrcplxf2[0].image = __fsub_rn(__fsub_rn(recv[0]._arrcplxf2[0].image, recv[1]._arrcplxf2[0].real), 
                                            __fsub_rn(recv[2]._arrcplxf2[0].image, recv[3]._arrcplxf2[0].real));
        res._arrcplxf2[1].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[1]._arrcplxf2[1].image), 
                                           __fadd_rn(recv[2]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image));
        res._arrcplxf2[1].image = __fsub_rn(__fsub_rn(recv[0]._arrcplxf2[1].image, recv[1]._arrcplxf2[1].real), 
                                            __fsub_rn(recv[2]._arrcplxf2[1].image, recv[3]._arrcplxf2[1].real));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}

template __global__ void
decx::dsp::fft::GPUK::cu_FFT2_R4_1st_C2C_cplxf<true>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t _div_length);

template __global__ void
decx::dsp::fft::GPUK::cu_FFT2_R4_1st_C2C_cplxf<false>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t _div_length);



template <bool _conj> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_C2C_cplxf(const float4* __restrict src,
                                           float4* __restrict dst,
                                           const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                           const uint32_t _pitchsrc_v2,
                                           const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 4;

    decx::utils::_cuda_vec128 recv[4];
    decx::utils::_cuda_vec128 res, tmp1, tmp2;

    uint32_t _FFT_domain_dex, warp_loc_id;

    decx::utils::_cuda_vec64 W;

    warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        _FFT_domain_dex = tidy;
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            recv[i]._vf = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W._cplxf32.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[1]._vf, W._cplxf32);

    W._cplxf32.construct_with_phase(__fmul_rn(Four_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[2]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[2]._vf, W._cplxf32);

    W._cplxf32.construct_with_phase(__fmul_rn(Six_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[3]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[3]._vf, W._cplxf32);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        // Calculate the first and third output
        tmp1._vf = decx::utils::cuda::__float_add4(recv[0]._vf, recv[2]._vf);
        tmp2._vf = decx::utils::cuda::__float_add4(recv[1]._vf, recv[3]._vf);

        // Store the first output
        res._vf = decx::utils::cuda::__float_add4(tmp1._vf, tmp2._vf);
        if (_conj) { res = decx::dsp::fft::GPUK::_complex4_conjugate_fp32(res); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += (_kernel_info._store_pitch << 1);
        
        // Store the third output
        res._vf = decx::utils::cuda::__float_sub4(tmp1._vf, tmp2._vf);
        if (_conj) { res = decx::dsp::fft::GPUK::_complex4_conjugate_fp32(res); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;

        // Calculate and store the second output
        res._arrcplxf2[0].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image), 
                                           __fadd_rn(recv[1]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].real));
        res._arrcplxf2[0].image = __fadd_rn(__fsub_rn(recv[0]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].image), 
                                            __fsub_rn(recv[1]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].real));
        res._arrcplxf2[1].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image), 
                                           __fadd_rn(recv[1]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].real));
        res._arrcplxf2[1].image = __fadd_rn(__fsub_rn(recv[0]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].image), 
                                            __fsub_rn(recv[1]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].real));
        _FFT_domain_dex -= (_kernel_info._store_pitch);
        if (_conj) { res = decx::dsp::fft::GPUK::_complex4_conjugate_fp32(res); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;

        res._arrcplxf2[0].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[1]._arrcplxf2[0].image), 
                                           __fadd_rn(recv[2]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image));
        res._arrcplxf2[0].image = __fsub_rn(__fsub_rn(recv[0]._arrcplxf2[0].image, recv[1]._arrcplxf2[0].real), 
                                            __fsub_rn(recv[2]._arrcplxf2[0].image, recv[3]._arrcplxf2[0].real));
        res._arrcplxf2[1].real = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[1]._arrcplxf2[1].image), 
                                           __fadd_rn(recv[2]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image));
        res._arrcplxf2[1].image = __fsub_rn(__fsub_rn(recv[0]._arrcplxf2[1].image, recv[1]._arrcplxf2[1].real), 
                                            __fsub_rn(recv[2]._arrcplxf2[1].image, recv[3]._arrcplxf2[1].real));

        _FFT_domain_dex += (_kernel_info._store_pitch << 1);
        if (_conj) { res = decx::dsp::fft::GPUK::_complex4_conjugate_fp32(res); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}

template __global__ void decx::dsp::fft::GPUK::cu_FFT2_R4_C2C_cplxf<true>(const float4* __restrict, float4* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT2_R4_C2C_cplxf<false>(const float4* __restrict, float4* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);



__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_C2R_cplxf_u8(const float4* __restrict src,
                                              uchar2* __restrict dst,
                                              const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                              const uint32_t _pitchsrc_v2,
                                              const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 4;

    decx::utils::_cuda_vec128 recv[4], tmp;
    decx::utils::_cuda_vec128 tmp1, tmp2;
    decx::utils::_cuda_vec64 res;
    de::CPf W;

    uint32_t _FFT_domain_dex = tidy;
    uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            recv[i]._vf = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[1]._vf, W);

    W.construct_with_phase(__fmul_rn(Four_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[2]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[2]._vf, W);

    W.construct_with_phase(__fmul_rn(Six_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[3]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[3]._vf, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        // Calculate the first and third output
        tmp1._vf = decx::utils::cuda::__float_add4(recv[0]._vf, recv[2]._vf);
        tmp2._vf = decx::utils::cuda::__float_add4(recv[1]._vf, recv[3]._vf);

        // Store the first output
        res._vf2.x = __fadd_rn(tmp1._vf.x, tmp2._vf.x);
        res._vf2.y = __fadd_rn(tmp1._vf.z, tmp2._vf.z);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = make_uchar2(res._vf2.x, res._vf2.y);
        _FFT_domain_dex += (_kernel_info._store_pitch << 1);
        
        // Store the third output
        res._vf2.x = __fsub_rn(tmp1._vf.x, tmp2._vf.x);
        res._vf2.y = __fsub_rn(tmp1._vf.z, tmp2._vf.z);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = make_uchar2(res._vf2.x, res._vf2.y);

        // Calculate and store the second output
        res._vf2.x = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image), 
                               __fadd_rn(recv[1]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].real));
        res._vf2.y = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image), 
                               __fadd_rn(recv[1]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].real));
        _FFT_domain_dex -= (_kernel_info._store_pitch);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = make_uchar2(res._vf2.x, res._vf2.y);

        res._vf2.x = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[1]._arrcplxf2[0].image), 
                               __fadd_rn(recv[2]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image));
        res._vf2.y = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[1]._arrcplxf2[1].image), 
                               __fadd_rn(recv[2]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image));

        _FFT_domain_dex += (_kernel_info._store_pitch << 1);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = make_uchar2(res._vf2.x, res._vf2.y);
    }
}




__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R4_C2R_cplxf_fp32(const float4* __restrict src,
                                              float2* __restrict dst,
                                              const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                              const uint32_t _pitchsrc_v2,
                                              const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 4;

    decx::utils::_cuda_vec128 recv[4], tmp;
    decx::utils::_cuda_vec128 tmp1, tmp2;
    decx::utils::_cuda_vec64 res;
    de::CPf W;

    uint32_t _FFT_domain_dex = tidy;
    uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 4
        for (uint8_t i = 0; i < 4; ++i) {
            recv[i]._vf = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[1]._vf, W);

    W.construct_with_phase(__fmul_rn(Four_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[2]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[2]._vf, W);

    W.construct_with_phase(__fmul_rn(Six_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[3]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[3]._vf, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        // Calculate the first and third output
        tmp1._vf = decx::utils::cuda::__float_add4(recv[0]._vf, recv[2]._vf);
        tmp2._vf = decx::utils::cuda::__float_add4(recv[1]._vf, recv[3]._vf);

        // Store the first output
        res._vf2.x = __fadd_rn(tmp1._vf.x, tmp2._vf.x);
        res._vf2.y = __fadd_rn(tmp1._vf.z, tmp2._vf.z);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
        _FFT_domain_dex += (_kernel_info._store_pitch << 1);
        
        // Store the third output
        res._vf2.x = __fsub_rn(tmp1._vf.x, tmp2._vf.x);
        res._vf2.y = __fsub_rn(tmp1._vf.z, tmp2._vf.z);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;

        // Calculate and store the second output
        res._vf2.x = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image), 
                               __fadd_rn(recv[1]._arrcplxf2[0].image, recv[2]._arrcplxf2[0].real));
        res._vf2.y = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image), 
                               __fadd_rn(recv[1]._arrcplxf2[1].image, recv[2]._arrcplxf2[1].real));
        _FFT_domain_dex -= (_kernel_info._store_pitch);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;

        res._vf2.x = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[0].real, recv[1]._arrcplxf2[0].image), 
                               __fadd_rn(recv[2]._arrcplxf2[0].real, recv[3]._arrcplxf2[0].image));
        res._vf2.y = __fsub_rn(__fadd_rn(recv[0]._arrcplxf2[1].real, recv[1]._arrcplxf2[1].image), 
                               __fadd_rn(recv[2]._arrcplxf2[1].real, recv[3]._arrcplxf2[1].image));

        _FFT_domain_dex += (_kernel_info._store_pitch << 1);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
    }
}