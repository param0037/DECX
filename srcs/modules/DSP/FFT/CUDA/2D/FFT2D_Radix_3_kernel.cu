#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "FFT2D_Radix_3_kernel.cuh"



__global__
void decx::signal::GPUK::cu_FFT2D_R3_R2C_first(const float* src, 
                            float2* dst, 
                            const uint B_ops_num,
                            const uint pitchsrc,        // in float
                            const uint pitchdst,
                            const uint procH)        // in float2
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_src = dex_h * pitchsrc + dex_w;
    size_t dex_base_dst = dex_h * pitchdst;

    float recv[3];
    de::CPf res[3] = { de::CPf(0, 0), de::CPf(0, 0), de::CPf(0, 0) };

    if (dex_h < procH && dex_w < B_ops_num) {
        recv[0] = src[dex_src];
        recv[1] = src[dex_src + B_ops_num];
        recv[2] = src[dex_src + B_ops_num * 2];

        res[0].real = __fadd_rn(__fadd_rn(recv[0], recv[1]), recv[2]);

        res[1].real = fmaf(-0.5f, __fadd_rn(recv[1], recv[2]), recv[0]);
        res[1].image = __fmul_rn(__fsub_rn(recv[1], recv[2]), 0.8660254f);

        res[2].real = res[1].real;
        res[2].image = __fmul_rn(__fsub_rn(recv[2], recv[1]), 0.8660254f);

        dst[dex_base_dst + dex_w * 3] = *((float2*)&res[0]);
        dst[dex_base_dst + dex_w * 3 + 1] = *((float2*)&res[1]);
        dst[dex_base_dst + dex_w * 3 + 2] = *((float2*)&res[2]);
    }
}




__global__
void decx::signal::GPUK::cu_FFT2D_R3_C2C_first(const float2* src, 
                            float2* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float2
                            const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    de::CPf recv[3];
    de::CPf res[3] = { de::CPf(0, 0), de::CPf(0, 0), de::CPf(0, 0) };

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base_dst + dex_w];
        *((float2*)&recv[1]) = src[dex_base_dst + dex_w + B_ops_num];
        *((float2*)&recv[2]) = src[dex_base_dst + dex_w + B_ops_num * 2];
        
        res[0].real = __fadd_rn(__fadd_rn(recv[0].real, recv[1].real), recv[2].real);
        res[0].image = __fadd_rn(__fadd_rn(recv[0].image, recv[1].image), recv[2].image);

        res[1] = _complex_fma(recv[1], de::CPf(-0.5, 0.8660254f), recv[0]);
        res[1] = _complex_fma(recv[2], de::CPf(-0.5, -0.8660254f), res[1]);

        res[2] = _complex_fma(recv[1], de::CPf(-0.5, -0.8660254f), recv[0]);
        res[2] = _complex_fma(recv[2], de::CPf(-0.5, 0.8660254f), res[2]);

        dst[dex_base_dst + dex_w * 3] = *((float2*)&res[0]);
        dst[dex_base_dst + dex_w * 3 + 1] = *((float2*)&res[1]);
        dst[dex_base_dst + dex_w * 3 + 2] = *((float2*)&res[2]);
    }
}


__global__
void decx::signal::GPUK::cu_IFFT2D_R3_C2R_once(const float2* src, 
                            float* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float2
                            const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    de::CPf recv[3], tmp;
    float res[3];
    const float signal_len = (float)B_ops_num * 3;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base_dst + dex_w];
        *((float2*)&recv[1]) = src[dex_base_dst + dex_w + B_ops_num];
        *((float2*)&recv[2]) = src[dex_base_dst + dex_w + B_ops_num * 2];

        res[0] = __fadd_rn(__fadd_rn(recv[0].real, recv[1].real), recv[2].real);

        tmp = _complex_fma(recv[1], de::CPf(-0.5, 0.8660254f), recv[0]);
        res[1] = _complex_fma_preserve_R(recv[2], de::CPf(-0.5, -0.8660254f), tmp);

        tmp = _complex_fma(recv[1], de::CPf(-0.5, -0.8660254f), recv[0]);
        res[2] = _complex_fma_preserve_R(recv[2], de::CPf(-0.5, 0.8660254f), tmp);

#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            res[i] = __fdividef(res[i], signal_len);
        }

        dst[dex_base_dst * 2 + dex_w * 3] = res[0];
        dst[dex_base_dst * 2 + dex_w * 3 + 1] = res[1];
        dst[dex_base_dst * 2 + dex_w * 3 + 2] = res[2];
    }
}


__global__
void decx::signal::GPUK::cu_IFFT2D_R3_C2C_first(const float2* src, 
                            float2* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float2
                            const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    de::CPf recv[3];
    de::CPf res[3] = { de::CPf(0, 0), de::CPf(0, 0), de::CPf(0, 0) };

    const float signal_len = (float)B_ops_num * 3;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base_dst + dex_w];
        *((float2*)&recv[1]) = src[dex_base_dst + dex_w + B_ops_num];
        *((float2*)&recv[2]) = src[dex_base_dst + dex_w + B_ops_num * 2];
        
        // conj and * /= signal_len
        recv[0].real = __fdividef(recv[0].real, signal_len);        recv[0].image = __fdividef(recv[0].image, -signal_len);
        recv[1].real = __fdividef(recv[1].real, signal_len);        recv[1].image = __fdividef(recv[1].image, -signal_len);
        recv[2].real = __fdividef(recv[2].real, signal_len);        recv[2].image = __fdividef(recv[2].image, -signal_len);

        res[0].real = __fadd_rn(__fadd_rn(recv[0].real, recv[1].real), recv[2].real);
        res[0].image = __fadd_rn(__fadd_rn(recv[0].image, recv[1].image), recv[2].image);

        res[1] = _complex_fma(recv[1], de::CPf(-0.5, 0.8660254f), recv[0]);
        res[1] = _complex_fma(recv[2], de::CPf(-0.5, -0.8660254f), res[1]);

        res[2] = _complex_fma(recv[1], de::CPf(-0.5, -0.8660254f), recv[0]);
        res[2] = _complex_fma(recv[2], de::CPf(-0.5, 0.8660254f), res[2]);

        dst[dex_base_dst + dex_w * 3] = *((float2*)&res[0]);
        dst[dex_base_dst + dex_w * 3 + 1] = *((float2*)&res[1]);
        dst[dex_base_dst + dex_w * 3 + 2] = *((float2*)&res[2]);
    }
}



__global__
void decx::signal::GPUK::cu_FFT2D_R3_C2C(const float2* src, 
                     float2* dst, 
                     const uint B_ops_num, 
                     const uint warp_proc_len,
                     const uint pitch,              // in de::CPf
                     const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;

    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 3;

    de::CPf recv[3], tmp;
    de::CPf W, res[3] = { de::CPf(0, 0), de::CPf(0, 0), de::CPf(0, 0) };

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base + dex_w];
        *((float2*)&recv[1]) = src[dex_base + dex_w + B_ops_num];
        *((float2*)&recv[2]) = src[dex_base + dex_w + B_ops_num * 2];

        warp_loc_id = dex_w % num_of_Bcalc_in_warp;
        W.dev_construct_with_phase(Two_Pi * (float)warp_loc_id / (float)warp_proc_len);
        recv[1] = _complex_mul(recv[1], W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 2) / (float)warp_proc_len);
        recv[2] = _complex_mul(recv[2], W);

        res[0].real = recv[0].real + recv[1].real + recv[2].real;
        res[0].image = recv[0].image + recv[1].image + recv[2].image;

        tmp = recv[0];
        tmp = _complex_fma(recv[1], de::CPf(-0.5, 0.8660254f), tmp);
        res[1] = _complex_fma(recv[2], de::CPf(-0.5, -0.8660254f), tmp);

        tmp = recv[0];
        tmp = _complex_fma(recv[1], de::CPf(-0.5, -0.8660254f), tmp);
        res[2] = _complex_fma(recv[2], de::CPf(-0.5, 0.8660254f), tmp);

        dex_store_w = (dex_w / num_of_Bcalc_in_warp) * warp_proc_len + warp_loc_id;

        dst[dex_base + dex_store_w] = *((float2*)&res[0]);
        dst[dex_base + dex_store_w + num_of_Bcalc_in_warp] = *((float2*)&res[1]);
        dst[dex_base + dex_store_w + num_of_Bcalc_in_warp * 2] = *((float2*)&res[2]);
    }
}


__global__
void decx::signal::GPUK::cu_IFFT2D_R3_C2R_last(const float2* src, 
                           float* dst, 
                           const uint B_ops_num, 
                           const uint warp_proc_len,
                           const uint pitch,              // in de::CPf
                           const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;

    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 3;

    de::CPf recv[3], tmp;
    de::CPf W;
    float res[3];
    const float signal_len = (float)B_ops_num * 3;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base + dex_w];
        *((float2*)&recv[1]) = src[dex_base + dex_w + B_ops_num];
        *((float2*)&recv[2]) = src[dex_base + dex_w + B_ops_num * 2];

        warp_loc_id = dex_w % num_of_Bcalc_in_warp;
        W.dev_construct_with_phase(Two_Pi * (float)warp_loc_id / (float)warp_proc_len);
        recv[1] = _complex_mul(recv[1], W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 2) / (float)warp_proc_len);
        recv[2] = _complex_mul(recv[2], W);

        res[0] = recv[0].real + recv[1].real + recv[2].real;

        tmp = recv[0];
        tmp = _complex_fma(recv[1], de::CPf(-0.5, 0.8660254f), tmp);
        res[1] = _complex_fma_preserve_R(recv[2], de::CPf(-0.5, -0.8660254f), tmp);

        tmp = recv[0];
        tmp = _complex_fma(recv[1], de::CPf(-0.5, -0.8660254f), tmp);
        res[2] = _complex_fma_preserve_R(recv[2], de::CPf(-0.5, 0.8660254f), tmp);

        dex_store_w = (dex_w / num_of_Bcalc_in_warp) * warp_proc_len + warp_loc_id;

#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            res[i] = __fdividef(res[i], signal_len);
        }

        dst[dex_base * 2 + dex_store_w] = res[0];
        dst[dex_base * 2 + dex_store_w + num_of_Bcalc_in_warp] = res[1];
        dst[dex_base * 2 + dex_store_w + num_of_Bcalc_in_warp * 2] = res[2];
    }
}


__global__
/*
* @param B_ops_num : in Vec4
* @param warp_proc_len : element
* @param pitch : in float4
*/
void decx::signal::GPUK::cu_FFT2D_R3_C2C_vec4(const float4* src, 
                          float4* dst, 
                          const uint B_ops_num, 
                          const uint warp_proc_len,
                          const uint pitch,
                          const uint procH)
{
    using namespace decx::signal::cuda::dev;
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;

    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 3 / 4;

    de::CPf W/*, tmp[4]*/;
    float4 recv[6], res;

    if (dex_h < procH && dex_w < B_ops_num) {
        recv[0] = src[dex_base + dex_w * 2];
        recv[1] = src[dex_base + dex_w * 2 + 1];
        recv[2] = src[dex_base + (dex_w + B_ops_num) * 2];
        recv[3] = src[dex_base + (dex_w + B_ops_num) * 2 + 1];
        recv[4] = src[dex_base + (dex_w + B_ops_num * 2) * 2];
        recv[5] = src[dex_base + (dex_w + B_ops_num * 2) * 2 + 1];

        warp_loc_id = dex_w % (size_t)num_of_Bcalc_in_warp;
        dex_store_w = (dex_w / (size_t)num_of_Bcalc_in_warp) * (size_t)warp_proc_len / 4 + warp_loc_id;

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].x) = _complex_mul(*((de::CPf*)&recv[2].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].z) = _complex_mul(*((de::CPf*)&recv[2].z), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].x) = _complex_mul(*((de::CPf*)&recv[3].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].z) = _complex_mul(*((de::CPf*)&recv[3].z), W);

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[4].x) = _complex_mul(*((de::CPf*)&recv[4].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[4].z) = _complex_mul(*((de::CPf*)&recv[4].z), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[5].x) = _complex_mul(*((de::CPf*)&recv[5].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[5].z) = _complex_mul(*((de::CPf*)&recv[5].z), W);

        // output 1
        res.x = recv[0].x + recv[2].x + recv[4].x;
        res.y = recv[0].y + recv[2].y + recv[4].y;
        res.z = recv[0].z + recv[2].z + recv[4].z;
        res.w = recv[0].w + recv[2].w + recv[4].w;
        dst[dex_base + dex_store_w * 2] = res;

        res.x = recv[1].x + recv[3].x + recv[5].x;
        res.y = recv[1].y + recv[3].y + recv[5].y;
        res.z = recv[1].z + recv[3].z + recv[5].z;
        res.w = recv[1].w + recv[3].w + recv[5].w;
        dst[dex_base + dex_store_w * 2 + 1] = res;

        // output 2
        dex_store_w += num_of_Bcalc_in_warp;
        res.x = recv[0].x;          res.y = recv[0].y;
        res.z = recv[0].z;          res.w = recv[0].w;
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[2].x), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[4].x), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[2].z), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.z));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[4].z), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.z));
        dst[dex_base + dex_store_w * 2] = res;

        res.x = recv[1].x;          res.y = recv[1].y;
        res.z = recv[1].z;          res.w = recv[1].w;
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[3].x), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[5].x), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[3].z), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.z));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[5].z), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.z));
        dst[dex_base + dex_store_w * 2 + 1] = res;

        // output 3
        dex_store_w += num_of_Bcalc_in_warp;
        res.x = recv[0].x;          res.y = recv[0].y;
        res.z = recv[0].z;          res.w = recv[0].w;
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[2].x), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[4].x), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[2].z), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.z));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[4].z), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.z));
        dst[dex_base + dex_store_w * 2] = res;

        res.x = recv[1].x;          res.y = recv[1].y;
        res.z = recv[1].z;          res.w = recv[1].w;
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[3].x), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.x) = _complex_fma(*((de::CPf*)&recv[5].x), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.x));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[3].z), de::CPf(-0.5, -0.8660254f), *((de::CPf*)&res.z));
        *((de::CPf*)&res.z) = _complex_fma(*((de::CPf*)&recv[5].z), de::CPf(-0.5, 0.8660254f), *((de::CPf*)&res.z));
        dst[dex_base + dex_store_w * 2 + 1] = res;
    }
}



__global__
/*
* @param B_ops_num : in Vec4
* @param warp_proc_len : element
* @param pitch : in float4
*/
void decx::signal::GPUK::cu_IFFT2D_R3_C2R_vec4_last(const float4* src, 
                                float4* dst, 
                                const uint B_ops_num, 
                                const uint warp_proc_len,
                                const uint pitch,
                                const uint procH)
{
    using namespace decx::signal::cuda::dev;
#define _fdividef_float4_ \
    res.x = __fdividef(res.x, signal_len);  \
    res.y = __fdividef(res.y, signal_len);  \
    res.z = __fdividef(res.z, signal_len);  \
    res.w = __fdividef(res.w, signal_len);  \

    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;
    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 3 / 4;

    const float signal_len = (float)B_ops_num * 3 * 4;

    de::CPf W, tmp;
    float4 recv[6], res;

    if (dex_h < procH && dex_w < B_ops_num) {
        recv[0] = src[dex_base + dex_w * 2];
        recv[1] = src[dex_base + dex_w * 2 + 1];
        recv[2] = src[dex_base + (dex_w + B_ops_num) * 2];
        recv[3] = src[dex_base + (dex_w + B_ops_num) * 2 + 1];
        recv[4] = src[dex_base + (dex_w + B_ops_num * 2) * 2];
        recv[5] = src[dex_base + (dex_w + B_ops_num * 2) * 2 + 1];

        warp_loc_id = dex_w % (size_t)num_of_Bcalc_in_warp;
        dex_store_w = (dex_w / (size_t)num_of_Bcalc_in_warp) * (size_t)warp_proc_len / 4 + warp_loc_id;

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].x) = _complex_mul(*((de::CPf*)&recv[2].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].z) = _complex_mul(*((de::CPf*)&recv[2].z), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].x) = _complex_mul(*((de::CPf*)&recv[3].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].z) = _complex_mul(*((de::CPf*)&recv[3].z), W);

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[4].x) = _complex_mul(*((de::CPf*)&recv[4].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[4].z) = _complex_mul(*((de::CPf*)&recv[4].z), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[5].x) = _complex_mul(*((de::CPf*)&recv[5].x), W);
        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) * 2 / (float)warp_proc_len);
        *((de::CPf*)&recv[5].z) = _complex_mul(*((de::CPf*)&recv[5].z), W);

        // output 1
        res.x = recv[0].x + recv[2].x + recv[4].x;
        res.y = recv[0].z + recv[2].z + recv[4].z;
        res.z = recv[1].x + recv[3].x + recv[5].x;
        res.w = recv[1].z + recv[3].z + recv[5].z;

        _fdividef_float4_;
        dst[dex_base + dex_store_w] = res;      // store

        // output 2
        dex_store_w += num_of_Bcalc_in_warp;
        tmp.real = recv[0].x;          tmp.image = recv[0].y;
        tmp = _complex_fma(*((de::CPf*)&recv[2].x), de::CPf(-0.5, 0.8660254f), tmp);
        res.x = _complex_fma_preserve_R(*((de::CPf*)&recv[4].x), de::CPf(-0.5, -0.8660254f), tmp);
        tmp.real = recv[0].z;          tmp.image = recv[0].w;
        tmp = _complex_fma(*((de::CPf*)&recv[2].z), de::CPf(-0.5, 0.8660254f), tmp);
        res.y = _complex_fma_preserve_R(*((de::CPf*)&recv[4].z), de::CPf(-0.5, -0.8660254f), tmp);
        tmp.real = recv[1].x;          tmp.image = recv[1].y;
        tmp = _complex_fma(*((de::CPf*)&recv[3].x), de::CPf(-0.5, 0.8660254f), tmp);
        res.z = _complex_fma_preserve_R(*((de::CPf*)&recv[5].x), de::CPf(-0.5, -0.8660254f), tmp);
        tmp.real = recv[1].z;          tmp.image = recv[1].w;
        tmp = _complex_fma(*((de::CPf*)&recv[3].z), de::CPf(-0.5, 0.8660254f), tmp);
        res.w = _complex_fma_preserve_R(*((de::CPf*)&recv[5].z), de::CPf(-0.5, -0.8660254f), tmp);

        _fdividef_float4_;
        dst[dex_base + dex_store_w] = res;          // store

        // output 3
        dex_store_w += num_of_Bcalc_in_warp;
        tmp.real = recv[0].x;          tmp.image = recv[0].y;
        tmp = _complex_fma(*((de::CPf*)&recv[2].x), de::CPf(-0.5, -0.8660254f), tmp);
        res.x = _complex_fma_preserve_R(*((de::CPf*)&recv[4].x), de::CPf(-0.5, 0.8660254f), tmp);
        tmp.real = recv[0].z;          tmp.image = recv[0].w;
        tmp = _complex_fma(*((de::CPf*)&recv[2].z), de::CPf(-0.5, -0.8660254f), tmp);
        res.y = _complex_fma_preserve_R(*((de::CPf*)&recv[4].z), de::CPf(-0.5, 0.8660254f), tmp);
        tmp.real = recv[1].x;          tmp.image = recv[1].y;
        tmp = _complex_fma(*((de::CPf*)&recv[3].x), de::CPf(-0.5, -0.8660254f), tmp);
        res.z = _complex_fma_preserve_R(*((de::CPf*)&recv[5].x), de::CPf(-0.5, 0.8660254f), tmp);
        tmp.real = recv[1].z;          tmp.image = recv[1].w;
        tmp = _complex_fma(*((de::CPf*)&recv[3].z), de::CPf(-0.5, -0.8660254f), tmp);
        res.w = _complex_fma_preserve_R(*((de::CPf*)&recv[5].z), de::CPf(-0.5, 0.8660254f), tmp);

        _fdividef_float4_;
        dst[dex_base + dex_store_w] = res;      // store
    }
#undef _fdividef_float4_
}