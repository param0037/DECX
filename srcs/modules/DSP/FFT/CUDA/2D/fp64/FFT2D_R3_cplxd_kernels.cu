#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "../FFT2D_kernels.cuh"


// [32 * 2, 8] = [64, 8]
__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R3_1st_R2C_uc8_cplxd(const uint8_t* __restrict src,
                                               double2* __restrict dst,
                                               const uint32_t _signal_len,
                                               const uint32_t _pitchsrc_v1,
                                               const uint32_t _pitchdst_v1)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 3;

    uint8_t recv[3];
    decx::utils::_cuda_vec128 tmp, res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc_v1)
    {
#pragma unroll 3
        for (uint8_t i = 0; i < 3; ++i) {
            recv[i] = src[_FFT_domain_dex * _pitchsrc_v1 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 3;

    if (tidy < _Bops_num && tidx < _pitchdst_v1)
    {
        res._cplxd.real = __dadd_rn(__dadd_rn(__ull2double_rn(recv[0]), 
                                              __ull2double_rn(recv[1])), 
                                              __ull2double_rn(recv[2]));
        res._cplxd.image = 0.0;
        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = res._vd;
        ++_FFT_domain_dex;

        tmp._arrd[0] = __fma_rn(-0.5, __dadd_rn(__ull2double_rn(recv[1]), 
                                                __ull2double_rn(recv[2])), 
                                                __ull2double_rn(recv[0]));

        res._cplxd.real = tmp._arrd[0];
        res._cplxd.image = __dmul_rn(__dsub_rn(__ull2double_rn(recv[1]), 
                                               __ull2double_rn(recv[2])), 0.8660254037844386);
        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._cplxd.real = tmp._arrd[0];
        res._cplxd.image = __dmul_rn(__dsub_rn(__ull2double_rn(recv[2]), 
                                               __ull2double_rn(recv[1])), 0.8660254037844386);
        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = res._vd;
    }
}



__global__ void 
decx::dsp::fft::GPUK::cu_FFT2_R3_C2R_cplxd_u8(const double2* __restrict src,
                                              uint8_t* __restrict dst,
                                              const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                              const uint32_t _pitchsrc_v1,
                                              const uint32_t _pitchdst_v1)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 3;

    decx::utils::_cuda_vec128 recv[3], tmp;
    decx::utils::_cuda_vec64 res;
    de::CPd W;

    uint32_t _FFT_domain_dex = tidy;
    uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc_v1)
    {
#pragma unroll 3
        for (uint8_t i = 0; i < 3; ++i) {
            recv[i]._vd = src[_FFT_domain_dex * _pitchsrc_v1 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W.construct_with_phase(__dmul_rn(Two_Pi, __ddiv_rn(__ull2double_rn(_warp_loc_id), 
                                                       __ull2double_rn(_kernel_info._warp_proc_len))));
    recv[1]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[1]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Four_Pi, __ddiv_rn(__ull2double_rn(_warp_loc_id), 
                                                        __ull2double_rn(_kernel_info._warp_proc_len))));
    recv[2]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[2]._cplxd, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v1)
    {
        res._fp64 = __dadd_rn(__dadd_rn(recv[0]._vd.x, recv[1]._vd.x), recv[2]._vd.x);

        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = __double2ull_rn(res._fp64);
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.5, 0.8660254037844386), recv[0]._cplxd);
        res._fp64 = __dsub_rn(__fma_rn(recv[2]._vd.x, -0.5, tmp._vd.x), __dmul_rn(recv[2]._vd.y, -0.8660254037844386));

        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = __double2ull_rn(res._fp64);
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.5, -0.8660254037844386), recv[0]._cplxd);
        res._fp64 = __dsub_rn(__fma_rn(recv[2]._vd.x, -0.5, tmp._vd.x), __dmul_rn(recv[2]._vd.y, 0.8660254037844386));

        dst[_FFT_domain_dex * _pitchdst_v1 + tidx] = __double2ull_rn(res._fp64);
    }
}
