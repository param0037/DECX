#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/


#include "FFT2D_Radix_2_kernel.cuh"


__global__
void decx::signal::GPUK::cu_FFT2D_R2_R2C_first(const float* src, 
                            float4* dst, 
                            const uint B_ops_num,
                            const uint pitchsrc,        // in float
                            const uint pitchdst,
                            const uint procH)        // in float4
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_src = dex_h * pitchsrc + dex_w;
    size_t dex_base_dst = dex_h * pitchdst;

    float2 recv;
    de::CPf res[2] = { de::CPf(0, 0), de::CPf(0, 0) };

    if (dex_h < procH && dex_w < B_ops_num) {
        recv.x = src[dex_src];
        recv.y = src[dex_src + B_ops_num];

        res[0].real = __fadd_rn(recv.x, recv.y);
        res[1].real = __fsub_rn(recv.x, recv.y);

        dst[dex_base_dst + dex_w] = *((float4*)&res[0]);
    }
}



__global__
void decx::signal::GPUK::cu_IFFT2D_R2_C2C_first(const float2* src,
                            float4* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float4
                            const uint procH)
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    float4 recv;
    de::CPf res[2];
    const float signal_length = (float)B_ops_num * 2;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv.x) = src[dex_base_dst * 2 + dex_w];
        *((float2*)&recv.z) = src[dex_base_dst * 2 + dex_w + B_ops_num];
        
        // conj and * /= signal_len
        recv.x /= signal_length;
        recv.y /= -signal_length;
        recv.z /= signal_length;
        recv.w /= -signal_length;

        res[0].real = __fadd_rn(recv.x, recv.z);
        res[0].image = __fadd_rn(recv.y, recv.w);
        res[1].real = __fsub_rn(recv.x, recv.z);
        res[1].image = __fsub_rn(recv.y, recv.w);

        dst[dex_base_dst + dex_w] = *((float4*)&res[0]);
    }
}


__global__
void decx::signal::GPUK::cu_IFFT2D_R2_C2R_once(const float2* src,
                            float2* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float2
                            const uint procH)
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    float4 recv;
    float2 res;
    const float signal_len = (float)B_ops_num * 2;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv.x) = src[dex_base_dst + dex_w];
        *((float2*)&recv.z) = src[dex_base_dst + dex_w + B_ops_num];

        res.x = __fdividef(__fadd_rn(recv.x, recv.z), signal_len);
        res.y = __fdividef(__fsub_rn(recv.x, recv.z), signal_len);

        dst[dex_base_dst + dex_w] = res;
    }
}


__global__
void decx::signal::GPUK::cu_FFT2D_R2_C2C_first(const float2* src,
                            float4* dst, 
                            const uint B_ops_num,
                            const uint pitch,       // in float4
                            const uint procH)
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;

    size_t dex_base_dst = dex_h * pitch;

    float4 recv;
    de::CPf res[2];
    const float signal_length = (float)B_ops_num * 2;

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv.x) = src[dex_base_dst * 2 + dex_w];
        *((float2*)&recv.z) = src[dex_base_dst * 2 + dex_w + B_ops_num];

        res[0].real = __fadd_rn(recv.x, recv.z);
        res[0].image = __fadd_rn(recv.y, recv.w);
        res[1].real = __fsub_rn(recv.x, recv.z);
        res[1].image = __fsub_rn(recv.y, recv.w);

        dst[dex_base_dst + dex_w] = *((float4*)&res[0]);
    }
}



__global__
void decx::signal::GPUK::cu_FFT2D_R2_C2C(const float2* src,
                     float2* dst, 
                     const uint B_ops_num, 
                     const uint warp_proc_len,
                     const uint pitch,              // in de::CPf
                     const uint procH)
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;

    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 2;

    de::CPf recv[2], tmp;
    de::CPf W, res[2] = { de::CPf(0, 0), de::CPf(0, 0) };

    if (dex_h < procH && dex_w < B_ops_num) {
        *((float2*)&recv[0]) = src[dex_base + dex_w];
        *((float2*)&recv[1]) = src[dex_base + dex_w + B_ops_num];

        warp_loc_id = dex_w % num_of_Bcalc_in_warp;
        W.dev_construct_with_phase(2 * Pi * (float)warp_loc_id / (float)warp_proc_len);

        tmp.real = __fsub_rn(__fmul_rn(recv[1].real, W.real), __fmul_rn(recv[1].image, W.image));
        tmp.image = __fadd_rn(__fmul_rn(recv[1].real, W.image), __fmul_rn(recv[1].image, W.real));

        res[0].real = __fadd_rn(recv[0].real, tmp.real);
        res[0].image = __fadd_rn(recv[0].image, tmp.image);
        res[1].real = __fsub_rn(recv[0].real, tmp.real);
        res[1].image = __fsub_rn(recv[0].image, tmp.image);

        dex_store_w = (dex_w / num_of_Bcalc_in_warp) * warp_proc_len + warp_loc_id;

        dst[dex_base + dex_store_w] = *((float2*)&res[0]);
        dst[dex_base + dex_store_w + num_of_Bcalc_in_warp] = *((float2*)&res[1]);
    }
}


__global__
/*
* @param B_ops_num : in Vec4
* @param warp_proc_len : element
* @param pitch : in float4
*/
void decx::signal::GPUK::cu_FFT2D_R2_C2C_vec4(const float4* src,
                          float4* dst, 
                          const uint B_ops_num, 
                          const uint warp_proc_len,
                          const uint pitch,
                          const uint procH)
{
    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;

    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 2 / 4;

    de::CPf W, tmp[4];
    float4 recv[4], res;

    if (dex_h < procH && dex_w < B_ops_num) {
        recv[0] = src[dex_base + dex_w * 2];
        recv[1] = src[dex_base + dex_w * 2 + 1];
        recv[2] = src[dex_base + (dex_w + B_ops_num) * 2];
        recv[3] = src[dex_base + (dex_w + B_ops_num) * 2 + 1];

        warp_loc_id = dex_w % (size_t)num_of_Bcalc_in_warp;
        dex_store_w = (dex_w / (size_t)num_of_Bcalc_in_warp) * (size_t)warp_proc_len / 4 + warp_loc_id;

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) / (float)warp_proc_len);
        tmp[0].real = __fsub_rn(__fmul_rn(recv[2].x, W.real), __fmul_rn(recv[2].y, W.image));
        tmp[0].image = __fadd_rn(__fmul_rn(recv[2].x, W.image), __fmul_rn(recv[2].y, W.real));

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) / (float)warp_proc_len);
        tmp[1].real = __fsub_rn(__fmul_rn(recv[2].z, W.real), __fmul_rn(recv[2].w, W.image));
        tmp[1].image = __fadd_rn(__fmul_rn(recv[2].z, W.image), __fmul_rn(recv[2].w, W.real));

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) / (float)warp_proc_len);
        tmp[2].real = __fsub_rn(__fmul_rn(recv[3].x, W.real), __fmul_rn(recv[3].y, W.image));
        tmp[2].image = __fadd_rn(__fmul_rn(recv[3].x, W.image), __fmul_rn(recv[3].y, W.real));

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) / (float)warp_proc_len);
        tmp[3].real = __fsub_rn(__fmul_rn(recv[3].z, W.real), __fmul_rn(recv[3].w, W.image));
        tmp[3].image = __fadd_rn(__fmul_rn(recv[3].z, W.image), __fmul_rn(recv[3].w, W.real));

        res.x = __fadd_rn(recv[0].x, tmp[0].real);
        res.y = __fadd_rn(recv[0].y, tmp[0].image);
        res.z = __fadd_rn(recv[0].z, tmp[1].real);
        res.w = __fadd_rn(recv[0].w, tmp[1].image);
        dst[dex_base + dex_store_w * 2] = res;

        res.x = __fadd_rn(recv[1].x, tmp[2].real);
        res.y = __fadd_rn(recv[1].y, tmp[2].image);
        res.z = __fadd_rn(recv[1].z, tmp[3].real);
        res.w = __fadd_rn(recv[1].w, tmp[3].image);
        dst[dex_base + dex_store_w * 2 + 1] = res;

        // output 2
        dex_store_w += num_of_Bcalc_in_warp;
        res.x = __fsub_rn(recv[0].x, tmp[0].real);
        res.y = __fsub_rn(recv[0].y, tmp[0].image);
        res.z = __fsub_rn(recv[0].z, tmp[1].real);
        res.w = __fsub_rn(recv[0].w, tmp[1].image);
        dst[dex_base + dex_store_w * 2] = res;

        res.x = __fsub_rn(recv[1].x, tmp[2].real);
        res.y = __fsub_rn(recv[1].y, tmp[2].image);
        res.z = __fsub_rn(recv[1].z, tmp[3].real);
        res.w = __fsub_rn(recv[1].w, tmp[3].image);
        dst[dex_base + dex_store_w * 2 + 1] = res;
    }
}


__global__
/*
* @param B_ops_num : in Vec4
* @param warp_proc_len : element
* @param pitch : in float4
*/
void decx::signal::GPUK::cu_IFFT2D_R2_C2R_vec4_last(const float4* src,
                                float4* dst, 
                                const uint B_ops_num, 
                                const uint warp_proc_len,
                                const uint pitch,
                                const uint procH)
{
    using namespace decx::signal::cuda::dev;
#define _fdividef_float4_ \
    res.x = __fdividef(res.x, signal_len);  \
    res.y = __fdividef(res.y, signal_len);  \
    res.z = __fdividef(res.z, signal_len);  \
    res.w = __fdividef(res.w, signal_len);  \

    uint dex_w = threadIdx.y + blockDim.y * blockIdx.y;
    uint dex_h = threadIdx.x + blockDim.x * blockIdx.x;
    uint dex_store_w;

    size_t dex_base = dex_h * pitch;
    uint warp_loc_id, num_of_Bcalc_in_warp = warp_proc_len / 2 / 4;

    const float signal_len = (float)B_ops_num * 2 * 4;

    de::CPf W, tmp[4];
    float4 recv[4], res;

    if (dex_h < procH && dex_w < B_ops_num) {
        recv[0] = src[dex_base + dex_w * 2];
        recv[1] = src[dex_base + dex_w * 2 + 1];
        recv[2] = src[dex_base + (dex_w + B_ops_num) * 2];
        recv[3] = src[dex_base + (dex_w + B_ops_num) * 2 + 1];

        warp_loc_id = dex_w % (size_t)num_of_Bcalc_in_warp;
        dex_store_w = (dex_w / (size_t)num_of_Bcalc_in_warp) * (size_t)warp_proc_len / 4 + warp_loc_id;

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].x) = _complex_mul(*((de::CPf*)&recv[2].x), W);

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 1) / (float)warp_proc_len);
        *((de::CPf*)&recv[2].z) = _complex_mul(*((de::CPf*)&recv[2].z), W);

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 2) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].x) = _complex_mul(*((de::CPf*)&recv[3].x), W);

        W.dev_construct_with_phase(Two_Pi * (float)(warp_loc_id * 4 + 3) / (float)warp_proc_len);
        *((de::CPf*)&recv[3].z) = _complex_mul(*((de::CPf*)&recv[3].z), W);

        res.x = __fadd_rn(recv[0].x, recv[2].x);
        res.y = __fadd_rn(recv[0].z, recv[2].z);
        res.z = __fadd_rn(recv[1].x, recv[3].x);
        res.w = __fadd_rn(recv[1].z, recv[3].z);

        _fdividef_float4_;
        dst[dex_base + dex_store_w] = res;

        // output 2
        dex_store_w += num_of_Bcalc_in_warp;
        res.x = __fsub_rn(recv[0].x, recv[2].x);
        res.y = __fsub_rn(recv[0].z, recv[2].z);
        res.z = __fsub_rn(recv[1].x, recv[3].x);
        res.w = __fsub_rn(recv[1].z, recv[3].z);

        _fdividef_float4_;
        dst[dex_base + dex_store_w] = res;
    }
#undef _fdividef_float4_
}