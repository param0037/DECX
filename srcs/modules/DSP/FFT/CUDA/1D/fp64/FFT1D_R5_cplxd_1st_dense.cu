#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../FFT1D_1st_kernels_dense.cuh"



__global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R5_1st_cplxd_R2C(const double* __restrict src, 
                                                double2* __restrict dst,
                                                const uint32_t _signal_len, 
                                                const uint32_t _pitchsrc, 
                                                const uint32_t _pitchdst)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 5;

    double recv[5];
    decx::utils::_cuda_vec128 res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i] = src[_FFT_domain_dex * _pitchsrc + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 5;

    if (tidy < _Bops_num && tidx < _pitchdst)
    {
        res._vd = decx::utils::vec2_set1_fp64(0.0);

        res._cplxd.real = __dadd_rn(__dadd_rn(recv[0], recv[1]),
                                    __dadd_rn(recv[2], recv[3]));
        res._cplxd.real = __dadd_rn(res._cplxd.real, recv[4]);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._cplxd.real = __fma_rn(0.309017, recv[1], recv[0]);
        res._cplxd.image = __dmul_rn(recv[1], 0.9510565);
        res._cplxd.real = __fma_rn(-0.809017, recv[2], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.5877853, recv[2], res._cplxd.image);
        res._cplxd.real = __fma_rn(-0.809017, recv[3], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.5877853, recv[3], res._cplxd.image);
        res._cplxd.real = __fma_rn(0.309017, recv[4], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.9510565, recv[4], res._cplxd.image);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._cplxd.real = __fma_rn(-0.809017, recv[1], recv[0]);
        res._cplxd.image = __dmul_rn(recv[1], 0.5877853);
        res._cplxd.real = __fma_rn(0.309017, recv[2], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.9510565, recv[2], res._cplxd.image);
        res._cplxd.real = __fma_rn(0.309017, recv[3], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.9510565, recv[3], res._cplxd.image);
        res._cplxd.real = __fma_rn(-0.809017, recv[4], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.5877853, recv[4], res._cplxd.image);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._cplxd.real = __fma_rn(-0.809017, recv[1], recv[0]);
        res._cplxd.image = __dmul_rn(recv[1], -0.5877853);
        res._cplxd.real = __fma_rn(0.309017, recv[2], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.9510565, recv[2], res._cplxd.image);
        res._cplxd.real = __fma_rn(0.309017, recv[3], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.9510565, recv[3], res._cplxd.image);
        res._cplxd.real = __fma_rn(-0.809017, recv[4], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.5877853, recv[4], res._cplxd.image);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._cplxd.real = __fma_rn(0.309017, recv[1], recv[0]);
        res._cplxd.image = __dmul_rn(recv[1], -0.9510565);
        res._cplxd.real = __fma_rn(-0.809017, recv[2], res._cplxd.real);
        res._cplxd.image = __fma_rn(-0.5877853, recv[2], res._cplxd.image);
        res._cplxd.real = __fma_rn(-0.809017, recv[3], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.5877853, recv[3], res._cplxd.image);
        res._cplxd.real = __fma_rn(0.309017, recv[4], res._cplxd.real);
        res._cplxd.image = __fma_rn(0.9510565, recv[4], res._cplxd.image);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
    }
}



template <bool _div> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R5_1st_cplxd_C2C(const double2* __restrict src,
                                                      double2* __restrict dst,
                                                      const uint32_t _signal_len,
                                                      const uint32_t _pitchsrc,
                                                      const uint32_t _pitchdst,
                                                      const uint64_t _div_length)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const float _numer = __ull2float_rn(_div_length ? _div_length : _signal_len);
    const uint32_t _Bops_num = _signal_len / 5;

    decx::utils::_cuda_vec128 recv[5];
    decx::utils::_cuda_vec128 res;

    
    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i]._vd = src[_FFT_domain_dex * _pitchsrc + tidx];
            if (_div) { recv[i]._vd = decx::utils::cuda::__double_div2_1(recv[i]._vd, _numer); }
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 5;

    if (tidy < _Bops_num && tidx < _pitchdst)
    {
        res._cplxd.real = __dadd_rn(__dadd_rn(recv[0]._cplxd.real, recv[1]._cplxd.real), 
                                      __dadd_rn(recv[2]._cplxd.real, recv[3]._cplxd.real));
        res._cplxd.real = __dadd_rn(res._cplxd.real, recv[4]._cplxd.real);
        res._cplxd.image = __dadd_rn(__dadd_rn(recv[0]._cplxd.image, recv[1]._cplxd.image), 
                                       __dadd_rn(recv[2]._cplxd.image, recv[3]._cplxd.image));
        res._cplxd.image = __dadd_rn(res._cplxd.image, recv[4]._cplxd.image);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        ++_FFT_domain_dex;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
    }
}

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R5_1st_cplxd_C2C<true>(const double2* __restrict, double2* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R5_1st_cplxd_C2C<false>(const double2* __restrict, double2* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t);



template <bool _conj> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R5_end_cplxd_C2C(const double2* __restrict src,
                                           double2* __restrict dst,
                                           const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                           const uint32_t _pitchsrc,
                                           const uint32_t _pitchdst)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 5;

    decx::utils::_cuda_vec128 recv[5];
    decx::utils::_cuda_vec128 res;
    de::CPd W;

    uint32_t _FFT_domain_dex = tidy;
    const uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i]._vd = src[_FFT_domain_dex * _pitchsrc + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    const double _frac = __ddiv_rn(_warp_loc_id, _kernel_info._warp_proc_len);
    W.construct_with_phase(__dmul_rn(Two_Pi, _frac));
    recv[1]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[1]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Four_Pi, _frac));
    recv[2]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[2]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Six_Pi, _frac));
    recv[3]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[3]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Eight_Pi, _frac));
    recv[4]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[4]._cplxd, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst) 
    {
        res._cplxd.real = __dadd_rn(__dadd_rn(recv[0]._cplxd.real, recv[1]._cplxd.real), 
                                      __dadd_rn(recv[2]._cplxd.real, recv[3]._cplxd.real));
        res._cplxd.real = __dadd_rn(res._cplxd.real, recv[4]._cplxd.real);
        res._cplxd.image = __dadd_rn(__dadd_rn(recv[0]._cplxd.image, recv[1]._cplxd.image), 
                                       __dadd_rn(recv[2]._cplxd.image, recv[3]._cplxd.image));
        res._cplxd.image = __dadd_rn(res._cplxd.image, recv[4]._cplxd.image);
        if (_conj) { res._cplxd = decx::dsp::fft::GPUK::_complex_conjugate_fp64(res._cplxd); }
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        if (_conj) { res._cplxd = decx::dsp::fft::GPUK::_complex_conjugate_fp64(res._cplxd); }
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        if (_conj) { res._cplxd = decx::dsp::fft::GPUK::_complex_conjugate_fp64(res._cplxd); }
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        if (_conj) { res._cplxd = decx::dsp::fft::GPUK::_complex_conjugate_fp64(res._cplxd); }
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vd = recv[0]._vd;
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, -0.9510565), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, -0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, 0.5877853), res._cplxd);
        res._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[4]._cplxd, de::CPd(0.309017, 0.9510565), res._cplxd);
        if (_conj) { res._cplxd = decx::dsp::fft::GPUK::_complex_conjugate_fp64(res._cplxd); }
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vd;
    }
}

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R5_end_cplxd_C2C<true>(const double2* __restrict, double2* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R5_end_cplxd_C2C<false>(const double2* __restrict, double2* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);



__global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R5_end_cplxd_C2R(const double2* __restrict src,
                                              double* __restrict dst,
                                              const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                              const uint32_t _pitchsrc,
                                              const uint32_t _pitchdst)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 5;

    decx::utils::_cuda_vec128 recv[5], tmp;
    double res;
    de::CPd W;

    uint32_t _FFT_domain_dex = tidy;
    uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i]._vd = src[_FFT_domain_dex * _pitchsrc + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    const double _frac = __ddiv_rn(_warp_loc_id, _kernel_info._warp_proc_len);
    W.construct_with_phase(__dmul_rn(Two_Pi, _frac));
    recv[1]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[1]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Four_Pi, _frac));
    recv[2]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[2]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Six_Pi, _frac));
    recv[3]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[3]._cplxd, W);

    W.construct_with_phase(__dmul_rn(Eight_Pi, _frac));
    recv[4]._cplxd = decx::dsp::fft::GPUK::_complex_mul_fp64(recv[4]._cplxd, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst)
    {
        res = __dadd_rn(__dadd_rn(recv[0]._cplxd.real, recv[1]._cplxd.real), 
                        __dadd_rn(recv[2]._cplxd.real, recv[3]._cplxd.real));
        res = __dadd_rn(res, recv[4]._cplxd.real);

        dst[_FFT_domain_dex * _pitchdst + tidx] = res;
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._vd = recv[0]._vd;
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, 0.9510565), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, 0.5877853), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, -0.5877853), tmp._cplxd);
        res = __dsub_rn(__fma_rn(recv[4]._vd.x, 0.309017, tmp._vd.x), __dmul_rn(recv[4]._vd.y, -0.9510565));

        dst[_FFT_domain_dex * _pitchdst + tidx] = res;
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._vd = recv[0]._vd;
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, 0.5877853), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, -0.9510565), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, 0.9510565), tmp._cplxd);
        res = __dsub_rn(__fma_rn(recv[4]._vd.x, -0.809017, tmp._vd.x), __dmul_rn(recv[4]._vd.y, -0.5877853));

        dst[_FFT_domain_dex * _pitchdst + tidx] = res;
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._vd = recv[0]._vd;
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(-0.809017, -0.5877853), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(0.309017, 0.9510565), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(0.309017, -0.9510565), tmp._cplxd);
        res = __dsub_rn(__fma_rn(recv[4]._vd.x, -0.809017, tmp._vd.x), __dmul_rn(recv[4]._vd.y, 0.5877853));

        dst[_FFT_domain_dex * _pitchdst + tidx] = res;
        _FFT_domain_dex += _kernel_info._store_pitch;

        tmp._vd = recv[0]._vd;
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[1]._cplxd, de::CPd(0.309017, -0.9510565), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[2]._cplxd, de::CPd(-0.809017, -0.5877853), tmp._cplxd);
        tmp._cplxd = decx::dsp::fft::GPUK::_complex_fma_fp64(recv[3]._cplxd, de::CPd(-0.809017, 0.5877853), tmp._cplxd);
        res = __dsub_rn(__fma_rn(recv[4]._vd.x, 0.309017, tmp._vd.x), __dmul_rn(recv[4]._vd.y, 0.9510565));

        dst[_FFT_domain_dex * _pitchdst + tidx] = res;
    }
}
