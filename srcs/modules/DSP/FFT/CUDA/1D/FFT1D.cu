#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "../../../../core/basic.h"
#include "../../../../classes/Vector.h"
#include "../../../../classes/GPU_Vector.h"
#include "CUDA_FFT1D_planner.cuh"
#include "FFT1D_1st_kernels_dense.cuh"
#include "../2D/FFT2D_kernels.cuh"
#include "../../../../BLAS/basic_process/transpose/CUDA/transpose_kernels.cuh"
#include "FFT1D_kernel_callers.cuh"
#include "../2D/FFT2D_1way_kernel_callers.cuh"


namespace decx
{
namespace dsp {
    namespace fft 
    {
        template <typename _type_in> _CRSR_ 
        static void _FFT1D_cplxf32_on_GPU(decx::_GPU_Vector* src, decx::_GPU_Vector* dst, de::DH* handle);


        template <typename _type_out> _CRSR_ 
        static void _IFFT1D_cplxf32_on_GPU(decx::_GPU_Vector* src, decx::_GPU_Vector* dst, de::DH* handle);


        template <typename _type_in> _CRSR_
            static void _FFT1D_cplxf32(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);


        template <typename _type_out> _CRSR_
            static void _IFFT1D_cplxf32(decx::_Vector* src, decx::_Vector* dst, de::DH* handle);
    }
}
}


template <typename _type_in> _CRSR_ 
static void decx::dsp::fft::_FFT1D_cplxf32_on_GPU(decx::_GPU_Vector* src, decx::_GPU_Vector* dst, de::DH* handle)
{
    decx::hip_stream* S = NULL;
    decx::cuda_event* E = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (S == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return;
    }
    if (E == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return;
    }

    decx::dsp::fft::_cuda_FFT1D_planner<float> _planner;
    _planner.plan(src->Len(), handle, S);

    const decx::dsp::fft::_cuda_FFT2D_planner<float>* _formal_FFT2D_ptr = _planner.get_FFT2D_planner();
    decx::utils::double_buffer_manager _double_buffer(_formal_FFT2D_ptr->get_tmp1_ptr<void>(),
                                                      _formal_FFT2D_ptr->get_tmp2_ptr<void>());

    decx::dsp::fft::FFT1D_partition_cplxf_1st_caller<_type_in, false>(src->Vec.ptr, &_double_buffer,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Vertical),
        S);

    decx::bp::transpose2D_b8_for_FFT(_double_buffer.get_leading_ptr<double2>(),
                                     _double_buffer.get_lagging_ptr<double2>(),
                                     make_uint2(_planner.get_larger_FFT_lengths(0), _planner.get_larger_FFT_lengths(1)),
                                     _formal_FFT2D_ptr->get_buffer_dims().x,
                                     _formal_FFT2D_ptr->get_buffer_dims().y, S);

    _double_buffer.update_states();

    decx::dsp::fft::FFT1D_partition_cplxf_end_caller<_FFT1D_END_>(&_double_buffer, dst->Vec.ptr,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Horizontal),
        S);

    E->event_record(S);
    S->synchronize();
}



template <typename _type_in> _CRSR_ 
static void decx::dsp::fft::_FFT1D_cplxf32(decx::_Vector* src, decx::_Vector* dst, de::DH* handle)
{
    decx::hip_stream* S = NULL;
    decx::cuda_event* E = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (S == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return;
    }
    if (E == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return;
    }

    decx::dsp::fft::_cuda_FFT1D_planner<float> _planner;
    _planner.plan(src->Len(), handle, S);

    const decx::dsp::fft::_cuda_FFT2D_planner<float>* _formal_FFT2D_ptr = _planner.get_FFT2D_planner();
    decx::utils::double_buffer_manager _double_buffer(_formal_FFT2D_ptr->get_tmp1_ptr<void>(),
                                                      _formal_FFT2D_ptr->get_tmp2_ptr<void>());

    checkCudaErrors(hipMemcpyAsync(_double_buffer.get_buffer2<void>(), src->Vec.ptr, src->Len() * sizeof(_type_in),
        hipMemcpyHostToDevice, S->get_raw_stream_ref()));

    decx::dsp::fft::FFT1D_partition_cplxf_1st_caller<_type_in, false>(NULL, &_double_buffer,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Vertical),
        S);

    decx::bp::transpose2D_b8_for_FFT(_double_buffer.get_leading_ptr<double2>(),
                                     _double_buffer.get_lagging_ptr<double2>(),
                                     make_uint2(_planner.get_larger_FFT_lengths(0), _planner.get_larger_FFT_lengths(1)),
                                     _formal_FFT2D_ptr->get_buffer_dims().x,
                                     _formal_FFT2D_ptr->get_buffer_dims().y, S);

    _double_buffer.update_states();

    decx::dsp::fft::FFT1D_partition_cplxf_end_caller<_FFT1D_END_>(&_double_buffer, NULL,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Horizontal),
        S);

    checkCudaErrors(hipMemcpyAsync(dst->Vec.ptr, _double_buffer.get_leading_ptr<void>(), src->Len() * sizeof(de::CPf),
        hipMemcpyDeviceToHost, S->get_raw_stream_ref()));

    E->event_record(S);
    S->synchronize();
}



template <typename _type_out> _CRSR_ 
static void decx::dsp::fft::_IFFT1D_cplxf32_on_GPU(decx::_GPU_Vector* src, decx::_GPU_Vector* dst, de::DH* handle)
{
    decx::hip_stream* S = NULL;
    decx::cuda_event* E = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (S == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return;
    }
    if (E == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return;
    }

    decx::dsp::fft::_cuda_FFT1D_planner<float> _planner;
    _planner.plan(src->Len(), handle, S);

    const decx::dsp::fft::_cuda_FFT2D_planner<float>* _formal_FFT2D_ptr = _planner.get_FFT2D_planner();
    decx::utils::double_buffer_manager _double_buffer(_formal_FFT2D_ptr->get_tmp1_ptr<void>(),
                                                      _formal_FFT2D_ptr->get_tmp2_ptr<void>());

    decx::dsp::fft::FFT1D_partition_cplxf_1st_caller<de::CPf, true>(src->Vec.ptr, &_double_buffer,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Vertical),
        S, _planner.get_signal_length());

    decx::bp::transpose2D_b8_for_FFT(_double_buffer.get_leading_ptr<double2>(),
                                     _double_buffer.get_lagging_ptr<double2>(),
                                     make_uint2(_planner.get_larger_FFT_lengths(0), _planner.get_larger_FFT_lengths(1)),
                                     _formal_FFT2D_ptr->get_buffer_dims().x,
                                     _formal_FFT2D_ptr->get_buffer_dims().y, S);

    _double_buffer.update_states();

    decx::dsp::fft::FFT1D_partition_cplxf_end_caller<_IFFT1D_END_(_type_out)>(&_double_buffer, dst->Vec.ptr,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Horizontal),
        S);

    E->event_record(S);
    S->synchronize();
}



template <typename _type_out> _CRSR_ 
static void decx::dsp::fft::_IFFT1D_cplxf32(decx::_Vector* src, decx::_Vector* dst, de::DH* handle)
{
    decx::hip_stream* S = NULL;
    decx::cuda_event* E = NULL;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (S == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return;
    }
    if (E == NULL) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return;
    }

    decx::dsp::fft::_cuda_FFT1D_planner<float> _planner;
    _planner.plan(src->Len(), handle, S);

    const decx::dsp::fft::_cuda_FFT2D_planner<float>* _formal_FFT2D_ptr = _planner.get_FFT2D_planner();
    decx::utils::double_buffer_manager _double_buffer(_formal_FFT2D_ptr->get_tmp1_ptr<void>(),
                                                      _formal_FFT2D_ptr->get_tmp2_ptr<void>());

    checkCudaErrors(hipMemcpyAsync(_double_buffer.get_buffer2<void>(), src->Vec.ptr, src->Len() * sizeof(de::CPf),
        hipMemcpyHostToDevice, S->get_raw_stream_ref()));

    decx::dsp::fft::FFT1D_partition_cplxf_1st_caller<de::CPf, true>(NULL, &_double_buffer,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Vertical),
        S, _planner.get_signal_length());

    decx::bp::transpose2D_b8_for_FFT(_double_buffer.get_leading_ptr<double2>(),
                                     _double_buffer.get_lagging_ptr<double2>(),
                                     make_uint2(_planner.get_larger_FFT_lengths(0), _planner.get_larger_FFT_lengths(1)),
                                     _formal_FFT2D_ptr->get_buffer_dims().x,
                                     _formal_FFT2D_ptr->get_buffer_dims().y, S);

    _double_buffer.update_states();

    decx::dsp::fft::FFT1D_partition_cplxf_end_caller<_IFFT1D_END_(_type_out)>(&_double_buffer, NULL,
        _formal_FFT2D_ptr->get_FFT_info(decx::dsp::fft::_cuda_FFT2D_planner<float>::_FFT_Horizontal),
        S);

    checkCudaErrors(hipMemcpyAsync(dst->Vec.ptr, _double_buffer.get_leading_ptr<void>(), src->Len() * sizeof(_type_out),
        hipMemcpyDeviceToHost, S->get_raw_stream_ref()));

    E->event_record(S);
    S->synchronize();
}



namespace de
{
namespace dsp {
    namespace cuda {
        _DECX_API_ de::DH FFT(de::Vector& src, de::Vector& dst);


        _DECX_API_ de::DH FFT(de::GPU_Vector& src, de::GPU_Vector& dst);


        _DECX_API_ de::DH IFFT(de::Vector& src, de::Vector& dst, const de::_DATA_TYPES_FLAGS_ _type_out);


        _DECX_API_ de::DH IFFT(de::GPU_Vector& src, de::GPU_Vector& dst, const de::_DATA_TYPES_FLAGS_ _type_out);
    }
}
}



_DECX_API_ de::DH de::dsp::cuda::FFT(de::GPU_Vector& src, de::GPU_Vector& dst)
{
    de::DH handle;

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);
    decx::_GPU_Vector* _dst = dynamic_cast<decx::_GPU_Vector*>(&dst);

    switch (_src->Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_FFT1D_cplxf32_on_GPU<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_FFT1D_cplxf32_on_GPU<de::CPf>(_src, _dst, &handle);
        break;
    default:
        break;
    }

    return handle;
}



_DECX_API_ de::DH de::dsp::cuda::FFT(de::Vector& src, de::Vector& dst)
{
    de::DH handle;

    decx::_Vector* _src = dynamic_cast<decx::_Vector*>(&src);
    decx::_Vector* _dst = dynamic_cast<decx::_Vector*>(&dst);

    switch (_src->Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_FFT1D_cplxf32<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_FFT1D_cplxf32<de::CPf>(_src, _dst, &handle);
        break;
    default:
        break;
    }

    return handle;
}




_DECX_API_ de::DH de::dsp::cuda::IFFT(de::GPU_Vector& src, de::GPU_Vector& dst, const de::_DATA_TYPES_FLAGS_ _type_out)
{
    de::DH handle;

    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);
    decx::_GPU_Vector* _dst = dynamic_cast<decx::_GPU_Vector*>(&dst);

    switch (_type_out)
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_IFFT1D_cplxf32_on_GPU<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_IFFT1D_cplxf32_on_GPU<de::CPf>(_src, _dst, &handle);
        break;
    default:
        break;
    }

    return handle;
}




_DECX_API_ de::DH de::dsp::cuda::IFFT(de::Vector& src, de::Vector& dst, const de::_DATA_TYPES_FLAGS_ _type_out)
{
    de::DH handle;

    decx::_Vector* _src = dynamic_cast<decx::_Vector*>(&src);
    decx::_Vector* _dst = dynamic_cast<decx::_Vector*>(&dst);

    switch (_type_out)
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_IFFT1D_cplxf32<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_IFFT1D_cplxf32<de::CPf>(_src, _dst, &handle);
        break;
    default:
        break;
    }

    return handle;
}