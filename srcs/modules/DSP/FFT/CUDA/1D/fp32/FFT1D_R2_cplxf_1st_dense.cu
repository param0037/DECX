#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../FFT1D_1st_kernels_dense.cuh"



// [32 * 2, 8] = [64, 8]
__global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R2_1st_cplxf_R2C_dense(const float* __restrict src,
                                                      float2* __restrict dst,
                                                      const uint32_t _signal_len,
                                                      const uint32_t _pitchsrc_v2,
                                                      const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 2;

    float recv[2];
    decx::utils::_cuda_vec64 res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 2
        for (uint8_t i = 0; i < 2; ++i) {
            recv[i] = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 2;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        res._cplxf32.image = 0.f;
        res._cplxf32.real = __fadd_rn(recv[0], recv[1]);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
        ++_FFT_domain_dex;

        res._cplxf32.real = __fsub_rn(recv[0], recv[1]);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
    }
}


template<bool _div> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R2_1st_cplxf_C2C_dense(const float2* __restrict src,
                                               float2* __restrict dst,
                                               const uint32_t _signal_len,
                                               const uint32_t _pitchsrc,
                                               const uint32_t _pitchdst,
                                               const uint64_t _div_length)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const float _numer = __ull2float_rn(_div_length ? _div_length : _signal_len);
    const uint32_t _Bops_num = _signal_len / 2;

    decx::utils::_cuda_vec64 recv[2];
    decx::utils::_cuda_vec64 res;

    uint32_t _FFT_domain_dex = tidy;

    if (tidy < _Bops_num && tidx < _pitchsrc)
    {
#pragma unroll 2
        for (uint8_t i = 0; i < 2; ++i) {
            recv[i]._vf2 = src[_FFT_domain_dex * _pitchsrc + tidx];
            if (_div) { recv[i]._vf2 = decx::utils::cuda::__float_div2_1(recv[i]._vf2, _numer); }
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = tidy * 2;

    if (tidy < _Bops_num && tidx < _pitchdst)
    {
        res._vf2 = decx::utils::cuda::__float_add2(recv[0]._vf2, recv[1]._vf2);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vf2;
        ++_FFT_domain_dex;

        res._vf2 = decx::utils::cuda::__float_sub2(recv[0]._vf2, recv[1]._vf2);
        dst[_FFT_domain_dex * _pitchdst + tidx] = res._vf2;
    }
}


template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R2_1st_cplxf_C2C_dense<true>(const float2* __restrict, float2* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R2_1st_cplxf_C2C_dense<false>(const float2* __restrict, float2* __restrict,
    const uint32_t, const uint32_t, const uint32_t, const uint64_t);



template <bool _conj> __global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R2_end_cplxf_C2C_dense(const float2* __restrict src,
                                                      float2* __restrict dst,
                                                      const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                                      const uint32_t _pitchsrc_v2,
                                                      const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 2;

    decx::utils::_cuda_vec64 recv[2];
    decx::utils::_cuda_vec64 res;

    uint32_t _FFT_domain_dex, warp_loc_id;

    decx::utils::_cuda_vec64 W;

    warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        _FFT_domain_dex = tidy;
#pragma unroll 2
        for (uint8_t i = 0; i < 2; ++i) {
            recv[i]._vf2 = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W._cplxf32.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._cplxf32 = decx::dsp::fft::GPUK::_complex_mul_fp32(recv[1]._cplxf32, W._cplxf32);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        res._vf2 = decx::utils::cuda::__float_add2(recv[0]._vf2, recv[1]._vf2);
        if (_conj) { res._cplxf32 = decx::dsp::fft::GPUK::_complex_conjugate_fp32(res._cplxf32); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf2 = decx::utils::cuda::__float_sub2(recv[0]._vf2, recv[1]._vf2);
        if (_conj) { res._cplxf32 = decx::dsp::fft::GPUK::_complex_conjugate_fp32(res._cplxf32); }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf2;
    }
}

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R2_end_cplxf_C2C_dense<true>(const float2* __restrict, float2* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT2D_R2_end_cplxf_C2C_dense<false>(const float2* __restrict, float2* __restrict,
    const decx::dsp::fft::FKI_4_2DK, const uint32_t, const uint32_t);



__global__ void 
decx::dsp::fft::GPUK::cu_FFT2D_R2_end_cplxf_C2R_dense(const float2* __restrict src,
                                              float* __restrict dst,
                                              const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                              const uint32_t _pitchsrc_v2,
                                              const uint32_t _pitchdst_v2)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 2;

    decx::utils::_cuda_vec64 recv[2], tmp;
    float res;
    de::CPf W;

    uint32_t _FFT_domain_dex = tidy;
    uint32_t _warp_loc_id = tidy % _kernel_info._store_pitch;

    if (tidy < _Bops_num && tidx < _pitchsrc_v2)
    {
#pragma unroll 2
        for (uint8_t i = 0; i < 2; ++i) {
            recv[i]._vf2 = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    W.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)_warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._cplxf32 = decx::dsp::fft::GPUK::_complex_mul_fp32(recv[1]._cplxf32, W);

    _FFT_domain_dex = (tidy / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id;

    if (tidy < _Bops_num && tidx < _pitchdst_v2)
    {
        res = __fadd_rn(recv[0]._vf2.x, recv[1]._vf2.x);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res = __fsub_rn(recv[0]._vf2.x, recv[1]._vf2.x);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res;
    }
}
