#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "FFT3D_planner.cuh"


//
//template <typename _data_type> _CRSR_
//decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::_cuda_FFT3D_planner(const uint3 signal_dims)
//{
//    this->_signal_dims = signal_dims;
//}
//
//template _CRSR_ decx::dsp::fft::_cuda_FFT3D_planner<float>::_cuda_FFT3D_planner(const uint3);



template <typename _data_type> _CRSR_
void decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::plan(const decx::_tensor_layout* _src_layout, 
                                                         const decx::_tensor_layout* _dst_layout,
                                                         de::DH* handle, decx::hip_stream* S)
{
    this->_signal_dims.x = _src_layout->depth;
    this->_signal_dims.y = _src_layout->width;
    this->_signal_dims.z = _src_layout->height;

    this->_input_typesize = _src_layout->_single_element_size;
    this->_output_typesize = _dst_layout->_single_element_size;

    constexpr uint8_t _alignment = 8 / sizeof(_data_type);

    this->_FFT_H.plan(this->_signal_dims.z);
    this->_FFT_D.plan(this->_signal_dims.x);
    this->_FFT_W._1way_FFT_conf.plan(this->_signal_dims.y);
    
    this->_FFT_H._pitchsrc = _src_layout->dp_x_wp;
    this->_FFT_H._pitchtmp = _src_layout->dp_x_wp;
    this->_FFT_H._pitchdst = this->_FFT_H._pitchtmp;

#if _CUDA_FFT3D_restrict_coalesce_
    this->_FFT_W._1way_FFT_conf._pitchsrc = decx::utils::align<uint32_t>(_src_layout->dpitch, 16);
    this->_FFT_W._1way_FFT_conf._pitchdst = decx::utils::align<uint32_t>(_src_layout->dpitch, 16);
    this->_FFT_W._1way_FFT_conf._pitchtmp = decx::utils::align<uint32_t>(_src_layout->dpitch, 16);

    this->_sync_dpitchdst_needed = (this->_FFT_W._1way_FFT_conf._pitchdst != _src_layout->dpitch);
#else
    this->_FFT_W._1way_FFT_conf._pitchsrc = _src_layout->dpitch;
    this->_FFT_W._1way_FFT_conf._pitchdst = _src_layout->dpitch;
    this->_FFT_W._1way_FFT_conf._pitchtmp = _src_layout->dpitch;
#endif

    this->_FFT_W._signal_pitch_src = _src_layout->wpitch;
    this->_FFT_W._signal_pitch_dst = _dst_layout->wpitch;
    this->_FFT_W._parallel = this->_signal_dims.z;

    this->_FFT_D._pitchsrc = _dst_layout->wpitch * _dst_layout->height;
    this->_FFT_D._pitchdst = _dst_layout->wpitch * _dst_layout->height;
    this->_FFT_D._pitchtmp = _dst_layout->wpitch * _dst_layout->height;

    const ulonglong3 _alloc_sizes = make_ulonglong3(this->_FFT_H._pitchtmp * this->_FFT_H.get_signal_len(),
                    this->_FFT_W._1way_FFT_conf._pitchtmp * this->_FFT_W._1way_FFT_conf.get_signal_len() * this->_FFT_W._parallel,
                    this->_FFT_D._pitchtmp * this->_FFT_D.get_signal_len());

    const uint64_t alloc_size = max(max(_alloc_sizes.x, _alloc_sizes.y), _alloc_sizes.z);
    if (decx::alloc::_device_malloc(&this->_tmp1, alloc_size * sizeof(_data_type) * 2, true, S) ||
        decx::alloc::_device_malloc(&this->_tmp2, alloc_size * sizeof(_data_type) * 2, true, S)) {
        decx::err::handle_error_info_modify(handle, decx::DECX_error_types::DECX_FAIL_ALLOCATION, ALLOC_FAIL);
        return;
    }
}

template _CRSR_ void decx::dsp::fft::_cuda_FFT3D_planner<float>::plan(const decx::_tensor_layout*,
    const decx::_tensor_layout*, de::DH*, decx::hip_stream*);

template _CRSR_ void decx::dsp::fft::_cuda_FFT3D_planner<double>::plan(const decx::_tensor_layout*,
    const decx::_tensor_layout*, de::DH*, decx::hip_stream*);



template <typename _data_type> const decx::dsp::fft::_FFT2D_1way_config* 
decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::get_FFT_info(const decx::dsp::fft::FFT_directions _dir) const
{
    switch (_dir)
    {
    case decx::dsp::fft::FFT_directions::_FFT_AlongH:
        return &this->_FFT_H;
        break;

    case decx::dsp::fft::FFT_directions::_FFT_AlongD:
        return &this->_FFT_D;
        break;

    default:
        return NULL;
        break;
    }
}

template const decx::dsp::fft::_FFT2D_1way_config*
decx::dsp::fft::_cuda_FFT3D_planner<float>::get_FFT_info(const FFT_directions) const;

template const decx::dsp::fft::_FFT2D_1way_config*
decx::dsp::fft::_cuda_FFT3D_planner<double>::get_FFT_info(const FFT_directions) const;


template <typename _data_type> const decx::dsp::fft::_cuda_FFT3D_mid_config* 
decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::get_midFFT_info() const
{
    return &this->_FFT_W;
}

template const decx::dsp::fft::_cuda_FFT3D_mid_config*
decx::dsp::fft::_cuda_FFT3D_planner<float>::get_midFFT_info() const;

template const decx::dsp::fft::_cuda_FFT3D_mid_config*
decx::dsp::fft::_cuda_FFT3D_planner<double>::get_midFFT_info() const;



template <typename _data_type>
bool decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::changed(const decx::_tensor_layout* src_layout, 
                                                            const decx::_tensor_layout* dst_layout) const
{
    return (this->_signal_dims.x ^ src_layout->depth) |
        (this->_signal_dims.y ^ src_layout->width) |
        (this->_signal_dims.z ^ src_layout->height) |
        (this->_input_typesize ^ src_layout->_single_element_size) |
        (this->_output_typesize ^ dst_layout->_single_element_size);
}

template bool decx::dsp::fft::_cuda_FFT3D_planner<float>::changed(const decx::_tensor_layout*, const decx::_tensor_layout*) const;
template bool decx::dsp::fft::_cuda_FFT3D_planner<double>::changed(const decx::_tensor_layout*, const decx::_tensor_layout*) const;


template <typename _data_type>
void decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::release(decx::dsp::fft::_cuda_FFT3D_planner<_data_type>* _fake_this)
{
    decx::alloc::_device_dealloc(&_fake_this->_tmp1);
    decx::alloc::_device_dealloc(&_fake_this->_tmp2);
}

template void decx::dsp::fft::_cuda_FFT3D_planner<float>::release(decx::dsp::fft::_cuda_FFT3D_planner<float>*);
template void decx::dsp::fft::_cuda_FFT3D_planner<double>::release(decx::dsp::fft::_cuda_FFT3D_planner<double>*);


template <typename _data_type>
decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::~_cuda_FFT3D_planner()
{
    decx::dsp::fft::_cuda_FFT3D_planner<_data_type>::release(this);
}

template decx::dsp::fft::_cuda_FFT3D_planner<float>::~_cuda_FFT3D_planner();
template decx::dsp::fft::_cuda_FFT3D_planner<double>::~_cuda_FFT3D_planner();
