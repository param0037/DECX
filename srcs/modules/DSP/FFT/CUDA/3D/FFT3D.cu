/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "../../../../core/basic.h"
#include "../2D/FFT2D_kernels.cuh"
#include "../../../../core/utils/double_buffer.h"
#include "../../../../BLAS/basic_process/transpose/CUDA/transpose_kernels.cuh"
#include "FFT3D_planner.cuh"
#include "../2D/FFT2D_1way_kernel_callers.cuh"
#include "FFT3D_MidProc_caller.cuh"
#include "../CUDA_FFTs.cuh"


namespace decx
{
namespace dsp {
    namespace fft {
        template <typename _type_in> _CRSR_
        static void _FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, de::DH* handle);


        template <typename _type_out> _CRSR_
        static void _IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, de::DH* handle);
    }
}
}



template <typename _type_in> _CRSR_
static void decx::dsp::fft::_FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, de::DH* handle)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_FFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), handle, S);
        Check_Runtime_Error(handle);
    }

    _planner->Forward<_type_in>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.unlock();
}



template <typename _type_out> _CRSR_
static void decx::dsp::fft::_IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, de::DH* handle)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_IFFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), handle, S);
        Check_Runtime_Error(handle);
    }

    _planner->Inverse<_type_out>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.unlock();
}




_DECX_API_ de::DH de::dsp::cuda::FFT(de::GPU_Tensor& src, de::GPU_Tensor& dst)
{
    de::DH handle;

    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify<true>(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return handle;
    }

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);

    switch (_src->Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        decx::dsp::fft::_FFT3D_caller_cplxf<uint8_t>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<de::CPf>(_src, _dst, &handle);
        break;

    default:
        decx::err::handle_error_info_modify<true>(&handle, decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }

    return handle;
}



_DECX_API_ de::DH de::dsp::cuda::IFFT(de::GPU_Tensor& src, de::GPU_Tensor& dst, const de::_DATA_TYPES_FLAGS_ type_out)
{
    de::DH handle;

    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify<true>(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return handle;
    }

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);

    switch (type_out)
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<float>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<de::CPf>(_src, _dst, &handle);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<uint8_t>(_src, _dst, &handle);
        break;
    default:
        decx::err::handle_error_info_modify<true>(&handle, decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }

    return handle;
}
