/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "../../../../core/basic.h"
#include "../2D/FFT2D_kernels.cuh"
#include "../../../../core/utils/double_buffer.h"
#include "../../../../BLAS/basic_process/transpose/CUDA/transpose_kernels.cuh"
#include "FFT3D_planner.cuh"
#include "../2D/FFT2D_1way_kernel_callers.cuh"
#include "FFT3D_MidProc_caller.cuh"
#include "../CUDA_FFTs.cuh"


namespace decx
{
namespace dsp {
    namespace fft {
        template <typename _type_in> _CRSR_
        static void _FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst);


        template <typename _type_in> _CRSR_
        static void _FFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst);


        template <typename _type_out> _CRSR_
        static void _IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst);


        template <typename _type_out> _CRSR_
        static void _IFFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst);
    }
}
}



template <typename _type_in> _CRSR_
static void decx::dsp::fft::_FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_FFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Forward<_type_in>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.unlock();
}



template <typename _type_in> _CRSR_
static void decx::dsp::fft::_FFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_FFT3D_cplxd64_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_FFT3D_cplxd64_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<double>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<double>::release);
    }

    decx::dsp::fft::cuda_FFT3D_cplxd64_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<double>* _planner =
        decx::dsp::fft::cuda_FFT3D_cplxd64_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<double>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Forward<_type_in>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_FFT3D_cplxd64_planner.unlock();
}


template <typename _type_out> _CRSR_
static void decx::dsp::fft::_IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_IFFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Inverse<_type_out>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.unlock();
}



template <typename _type_out> _CRSR_
static void decx::dsp::fft::_IFFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst)
{
    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (decx::dsp::fft::cuda_IFFT3D_cplxd64_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<double>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<double>::release);
    }

    decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<double>* _planner =
        decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<double>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Inverse<_type_out>(src, dst, S);

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.unlock();
}



_DECX_API_ void de::dsp::cuda::FFT(de::GPU_Tensor& src, de::GPU_Tensor& dst)
{
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify<true>(de::GetLastError(), 
            decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return;
    }

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);

    switch (_src->Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<float>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        decx::dsp::fft::_FFT3D_caller_cplxf<uint8_t>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<de::CPf>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::dsp::fft::_FFT3D_caller_cplxd<double>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F64_:
        decx::dsp::fft::_FFT3D_caller_cplxd<de::CPd>(_src, _dst);
        break;

    default:
        decx::err::handle_error_info_modify<true>(de::GetLastError(),
            decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }
}



_DECX_API_ void de::dsp::cuda::IFFT(de::GPU_Tensor& src, de::GPU_Tensor& dst, const de::_DATA_TYPES_FLAGS_ type_out)
{
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify<true>(de::GetLastError(),
            decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return;
    }

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);

    switch (type_out)
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<float>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<de::CPf>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<uint8_t>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::dsp::fft::_IFFT3D_caller_cplxd<double>(_src, _dst);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F64_:
        decx::dsp::fft::_IFFT3D_caller_cplxd<de::CPd>(_src, _dst);
        break;

    default:
        decx::err::handle_error_info_modify<true>(de::GetLastError(),
            decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }
}
