#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../FFT3D_kernels.cuh"



template <bool _div> __global__ void 
decx::dsp::fft::GPUK::cu_FFT3_R5_1st_C2C_cplxf(const float4* __restrict src, 
                                               float4* __restrict dst, 
                                               const uint32_t _signal_len,
                                               const uint2 _signal_pitch, 
                                               const uint32_t _pitchsrc_v2, 
                                               const uint32_t _pitchdst_v2,
                                               const uint32_t _paral)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 5;

    decx::utils::_cuda_vec128 recv[5];
    decx::utils::_cuda_vec128 res;

    uint32_t _FFT_domain_dex = (tidy % _Bops_num);
    const uint32_t _lane_id = tidy / _Bops_num;

    if (tidy < _Bops_num * _paral && tidx < _pitchsrc_v2)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i]._vf = src[(_FFT_domain_dex + _lane_id * _signal_pitch.x) * _pitchsrc_v2 + tidx];
            if (_div) { recv[i]._vf = decx::utils::cuda::__float_div4_1(recv[i]._vf, _signal_len); }
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = (tidy % _Bops_num) * 5 + _lane_id * _signal_pitch.y;

    if (tidy < _Bops_num * _paral && tidx < _pitchdst_v2)
    {
        for (uint8_t i = 0; i < 2; ++i) {
            res._arrcplxf2[i].real = __fadd_rn(__fadd_rn(recv[0]._arrcplxf2[i].real, recv[1]._arrcplxf2[i].real) + 
                recv[2]._arrcplxf2[i].real, __fadd_rn(recv[3]._arrcplxf2[i].real, recv[4]._arrcplxf2[i].real));
            res._arrcplxf2[i].image = __fadd_rn(__fadd_rn(recv[0]._arrcplxf2[i].image, recv[1]._arrcplxf2[i].image) +
                recv[2]._arrcplxf2[i].image, __fadd_rn(recv[3]._arrcplxf2[i].image, recv[4]._arrcplxf2[i].image));
        }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(0.309017, 0.9510565),  res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.809017, 0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(-0.809017, -0.5877853),res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(0.309017, -0.9510565), res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.809017, 0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(0.309017, -0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(0.309017, 0.9510565),  res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(-0.809017, -0.5877853),res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.809017, -0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(0.309017, 0.9510565),   res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(0.309017, -0.9510565),  res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(-0.809017, 0.5877853),  res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(0.309017, -0.9510565),  res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.809017, -0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(-0.809017, 0.5877853),  res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(0.309017, 0.9510565),   res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}

template __global__ void
decx::dsp::fft::GPUK::cu_FFT3_R5_1st_C2C_cplxf<true>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint2, const uint32_t, const uint32_t, const uint32_t);

template __global__ void
decx::dsp::fft::GPUK::cu_FFT3_R5_1st_C2C_cplxf<false>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint2, const uint32_t, const uint32_t, const uint32_t);


__global__ void
decx::dsp::fft::GPUK::cu_FFT3_R5_C2C_cplxf(const float4* __restrict src, 
                                           float4* __restrict dst, 
                                           const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                           const uint32_t signal_pitch, 
                                           const uint32_t _pitchsrc_v2, 
                                           const uint32_t _pitchdst_v2, 
                                           const uint32_t _paral)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 5;

    decx::utils::_cuda_vec128 recv[5];
    decx::utils::_cuda_vec128 res;
    de::CPf W;

    uint32_t _FFT_domain_dex = (tidy % _Bops_num);
    const uint32_t _lane_id = tidy / _Bops_num;

    if (tidy < _Bops_num * _paral && tidx < _pitchsrc_v2)
    {
#pragma unroll 5
        for (uint8_t i = 0; i < 5; ++i) {
            recv[i]._vf = src[(_FFT_domain_dex + _lane_id * signal_pitch) * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }
    const uint32_t _warp_loc_id = (tidy % _Bops_num) % _kernel_info._store_pitch;

    const float _frac = __fdividef(_warp_loc_id, _kernel_info._warp_proc_len);
    W.construct_with_phase(__fmul_rn(Two_Pi, _frac));
    recv[1]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[1]._vf, W);

    W.construct_with_phase(__fmul_rn(Four_Pi, _frac));
    recv[2]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[2]._vf, W);

    W.construct_with_phase(__fmul_rn(Six_Pi, _frac));
    recv[3]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[3]._vf, W);

    W.construct_with_phase(__fmul_rn(Eight_Pi, _frac));
    recv[4]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[4]._vf, W);

    _FFT_domain_dex = ((tidy % _Bops_num) / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + _warp_loc_id
        + _lane_id * signal_pitch;

    if (tidy < _Bops_num * _paral && tidx < _pitchdst_v2) 
    {
        for (uint8_t i = 0; i < 2; ++i) {
            res._arrcplxf2[i].real = __fadd_rn(__fadd_rn(recv[0]._arrcplxf2[i].real, recv[1]._arrcplxf2[i].real) +
                recv[2]._arrcplxf2[i].real, __fadd_rn(recv[3]._arrcplxf2[i].real, recv[4]._arrcplxf2[i].real));
            res._arrcplxf2[i].image = __fadd_rn(__fadd_rn(recv[0]._arrcplxf2[i].image, recv[1]._arrcplxf2[i].image) +
                recv[2]._arrcplxf2[i].image, __fadd_rn(recv[3]._arrcplxf2[i].image, recv[4]._arrcplxf2[i].image));
        }
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(0.309017, 0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.809017, 0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(-0.809017, -0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(0.309017, -0.9510565), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.809017, 0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(0.309017, -0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(0.309017, 0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(-0.809017, -0.5877853), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.809017, -0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(0.309017, 0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(0.309017, -0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(-0.809017, 0.5877853), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = recv[0]._vf;
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(0.309017, -0.9510565), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.809017, -0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[3]._vf, de::CPf(-0.809017, 0.5877853), res._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[4]._vf, de::CPf(0.309017, 0.9510565), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}
