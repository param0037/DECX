#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../FFT3D_kernels.cuh"



template<bool _div> __global__ void 
decx::dsp::fft::GPUK::cu_FFT3_R3_1st_C2C_cplxf(const float4* __restrict src, 
                                               float4* __restrict dst, 
                                               const uint32_t _signal_len,
                                               const uint2 _signal_pitch, 
                                               const uint32_t _pitchsrc_v2, 
                                               const uint32_t _pitchdst_v2, 
                                               const uint32_t _paral)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _signal_len / 3;

    decx::utils::_cuda_vec128 recv[3];
    decx::utils::_cuda_vec128 res, tmp1, tmp2;

    uint32_t _FFT_domain_dex = (tidy % _Bops_num);
    const uint32_t _lane_id = tidy / _Bops_num;

    if (tidy < _Bops_num * _paral && tidx < _pitchsrc_v2)
    {
#pragma unroll 3
        for (uint8_t i = 0; i < 3; ++i) {
            recv[i]._vf = src[(_FFT_domain_dex + _lane_id * _signal_pitch.x) * _pitchsrc_v2 + tidx];
            if (_div) { recv[i]._vf = decx::utils::cuda::__float_div4_1(recv[i]._vf, _signal_len); }
            _FFT_domain_dex += _Bops_num;
        }
    }

    _FFT_domain_dex = (tidy % _Bops_num) * 3 + _lane_id * _signal_pitch.y;

    if (tidy < _Bops_num * _paral && tidx < _pitchdst_v2) 
    {
        res._vf = decx::utils::cuda::__float_add4(recv[2]._vf,
            decx::utils::cuda::__float_add4(recv[0]._vf, recv[1]._vf));
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.5, 0.8660254f), recv[0]._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.5, -0.8660254f), res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        ++_FFT_domain_dex;

        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.5, -0.8660254f), recv[0]._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.5, 0.8660254f), res._vf);
        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}

template __global__ void decx::dsp::fft::GPUK::cu_FFT3_R3_1st_C2C_cplxf<true>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint2, const uint32_t, const uint32_t, const uint32_t);

template __global__ void decx::dsp::fft::GPUK::cu_FFT3_R3_1st_C2C_cplxf<false>(const float4* __restrict, float4* __restrict,
    const uint32_t, const uint2, const uint32_t, const uint32_t, const uint32_t);



__global__ void 
decx::dsp::fft::GPUK::cu_FFT3_R3_C2C_cplxf(const float4* __restrict src, 
                                           float4* __restrict dst, 
                                           const decx::dsp::fft::FKI_4_2DK _kernel_info,
                                           const uint32_t signal_pitch, 
                                           const uint32_t _pitchsrc_v2, 
                                           const uint32_t _pitchdst_v2, 
                                           const uint32_t _paral)
{
    uint32_t tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tidy = threadIdx.y + blockIdx.y * blockDim.y;

    const uint32_t _Bops_num = _kernel_info._signal_len / 3;

    decx::utils::_cuda_vec128 recv[3];
    decx::utils::_cuda_vec128 res, tmp1, tmp2;

    uint32_t _FFT_domain_dex;
    const uint32_t _lane_id = tidy / _Bops_num;

    decx::utils::_cuda_vec64 W;

    if (tidy < _Bops_num * _paral && tidx < _pitchdst_v2)
    {
        _FFT_domain_dex = (tidy % _Bops_num) + _lane_id * signal_pitch;
#pragma unroll 3
        for (uint8_t i = 0; i < 3; ++i) {
            recv[i]._vf = src[_FFT_domain_dex * _pitchsrc_v2 + tidx];
            _FFT_domain_dex += _Bops_num;
        }
    }

    const uint32_t warp_loc_id = (tidy % _Bops_num) % _kernel_info._store_pitch;

    W._cplxf32.construct_with_phase(__fmul_rn(Two_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[1]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[1]._vf, W._cplxf32);

    W._cplxf32.construct_with_phase(__fmul_rn(Four_Pi, __fdividef((float)warp_loc_id, (float)_kernel_info._warp_proc_len)));
    recv[2]._vf = decx::dsp::fft::GPUK::_complex_2mul1_fp32(recv[2]._vf, W._cplxf32);

    _FFT_domain_dex = ((tidy % _Bops_num) / _kernel_info._store_pitch) * _kernel_info._warp_proc_len + warp_loc_id
        + _lane_id * signal_pitch;

    if (tidy < _Bops_num * _paral && tidx < _pitchdst_v2)
    {
        res._vf = decx::utils::cuda::__float_add4(recv[2]._vf,
            decx::utils::cuda::__float_add4(recv[0]._vf, recv[1]._vf));

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.5, 0.8660254f), recv[0]._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.5, -0.8660254f), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
        _FFT_domain_dex += _kernel_info._store_pitch;

        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[1]._vf, de::CPf(-0.5, -0.8660254f), recv[0]._vf);
        res._vf = decx::dsp::fft::GPUK::_complex_2fma1_fp32(recv[2]._vf, de::CPf(-0.5, 0.8660254f), res._vf);

        dst[_FFT_domain_dex * _pitchdst_v2 + tidx] = res._vf;
    }
}
