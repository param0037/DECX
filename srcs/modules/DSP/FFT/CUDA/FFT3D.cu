/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "2D/FFT2D_kernels.cuh"
#include "3D/FFT3D_planner.cuh"
#include "2D/FFT2D_1way_kernel_callers.cuh"
#include "3D/FFT3D_MidProc_caller.cuh"
#include "CUDA_FFTs.cuh"
#include "../../../core/configs/config.h"


namespace decx
{
namespace dsp {
namespace fft {
    template <typename _type_in> _CRSR_
    static void _FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, 
        decx::hip_stream* S, decx::cuda_event* E);


    template <typename _type_in> _CRSR_
    static void _FFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, 
        decx::hip_stream* S, decx::cuda_event* E);


    template <typename _type_out> _CRSR_
    static void _IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, 
        decx::hip_stream* S, decx::cuda_event* E);


    template <typename _type_out> _CRSR_
    static void _IFFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst, 
        decx::hip_stream* S, decx::cuda_event* E);
}
}
}



template <typename _type_in> _CRSR_
static void decx::dsp::fft::_FFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst,
    decx::hip_stream* S, decx::cuda_event* E)
{
    if (decx::dsp::fft::cuda_FFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_FFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Forward<_type_in>(src, dst, S);

    decx::dsp::fft::cuda_FFT3D_cplxf32_planner.unlock();
}



template <typename _type_in> _CRSR_
static void decx::dsp::fft::_FFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst,
    decx::hip_stream* S, decx::cuda_event* E)
{
    if (decx::dsp::fft::cuda_FFT3D_cplxd64_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_FFT3D_cplxd64_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<double>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<double>::release);
    }

    decx::dsp::fft::cuda_FFT3D_cplxd64_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<double>* _planner =
        decx::dsp::fft::cuda_FFT3D_cplxd64_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<double>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Forward<_type_in>(src, dst, S);

    decx::dsp::fft::cuda_FFT3D_cplxd64_planner.unlock();
}


template <typename _type_out> _CRSR_
static void decx::dsp::fft::_IFFT3D_caller_cplxf(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst,
    decx::hip_stream* S, decx::cuda_event* E)
{
    if (decx::dsp::fft::cuda_IFFT3D_cplxf32_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<float>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<float>::release);
    }

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<float>* _planner =
        decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<float>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Inverse<_type_out>(src, dst, S);

    decx::dsp::fft::cuda_IFFT3D_cplxf32_planner.unlock();
}



template <typename _type_out> _CRSR_
static void decx::dsp::fft::_IFFT3D_caller_cplxd(decx::_GPU_Tensor* src, decx::_GPU_Tensor* dst,
    decx::hip_stream* S, decx::cuda_event* E)
{
    if (decx::dsp::fft::cuda_IFFT3D_cplxd64_planner._res_ptr == NULL) {
        decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.RegisterResource(new decx::dsp::fft::_cuda_FFT3D_planner<double>,
            5, &decx::dsp::fft::_cuda_FFT3D_planner<double>::release);
    }

    decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.lock();
    decx::dsp::fft::_cuda_FFT3D_planner<double>* _planner =
        decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.get_resource_raw_ptr<decx::dsp::fft::_cuda_FFT3D_planner<double>>();

    if (_planner->changed(&src->get_layout(), &dst->get_layout())) {
        _planner->plan(&src->get_layout(), &dst->get_layout(), de::GetLastError(), S);
        Check_Runtime_Error(de::GetLastError());
    }

    _planner->Inverse<_type_out>(src, dst, S);

    decx::dsp::fft::cuda_IFFT3D_cplxd64_planner.unlock();
}



_DECX_API_ void de::dsp::cuda::FFT(de::GPU_Tensor& src, de::GPU_Tensor& dst, const de::_DATA_TYPES_FLAGS_ _output_type)
{
    de::ResetLastError();

    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify(de::GetLastError(), 
            decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return;
    }

    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);
    
    if (!(decx::dsp::fft::validate_type_FFT2D(_src->Type()) && decx::dsp::fft::validate_type_FFT2D(_output_type)))
    {
        decx::err::handle_error_info_modify(de::GetLastError(), decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            "FFT2D CUDA only supports float, double, uint8_t, de::CPf and de::CPd input");
        return;
    }

    if ((_src->Type() & 3) == 1) {        // (complex)_Fp32
        _dst->re_construct(de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_, _src->Width(), _src->Height(), _src->Depth(), S);
    }
    else if ((_src->Type() & 3) == 2) {       // (complex)_Fp64
        _dst->re_construct(de::_DATA_TYPES_FLAGS_::_COMPLEX_F64_, _src->Width(), _src->Height(), _src->Depth(), S);
    }
    else {  // If is _UINT8_
        _dst->re_construct(_output_type, _src->Width(), _src->Height(), _src->Depth(), S);
    }

    switch (_src->Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<float>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        if (_output_type == de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_) {
            decx::dsp::fft::_FFT3D_caller_cplxf<uint8_t>(_src, _dst, S, E);
        }
        else {
            decx::dsp::fft::_FFT3D_caller_cplxd<uint8_t>(_src, _dst, S, E);
        }
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_FFT3D_caller_cplxf<de::CPf>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::dsp::fft::_FFT3D_caller_cplxd<double>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F64_:
        decx::dsp::fft::_FFT3D_caller_cplxd<de::CPd>(_src, _dst, S, E);
        break;

    default:
        decx::err::handle_error_info_modify(de::GetLastError(), decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();
}



_DECX_API_ void de::dsp::cuda::IFFT(de::GPU_Tensor& src, de::GPU_Tensor& dst, const de::_DATA_TYPES_FLAGS_ _output_type)
{
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify(de::GetLastError(),
            decx::DECX_error_types::DECX_FAIL_CUDA_not_init, CUDA_NOT_INIT);
        return;
    }

    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);
    decx::_GPU_Tensor* _dst = dynamic_cast<decx::_GPU_Tensor*>(&dst);

    decx::hip_stream* S;
    S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    decx::cuda_event* E;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);

    if (!(decx::dsp::fft::validate_type_FFT2D(_src->Type()) && decx::dsp::fft::validate_type_FFT2D(_output_type))) {
        decx::err::handle_error_info_modify(de::GetLastError(), decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            "FFT2D CUDA only supports float, double, uint8_t, de::CPf and de::CPd input");
        return;
    }

    if (_output_type != de::_DATA_TYPES_FLAGS_::_UINT8_) // Ensures it's either fp32(cplxf) or fp64(cplxd)
    {
        if (!decx::dsp::fft::check_type_matched_FFT(_src->Type(), _output_type)) {
            decx::err::handle_error_info_modify(de::GetLastError(), decx::DECX_error_types::DECX_FAIL_TYPE_MOT_MATCH,
                "Conversion between fp32 and fp64 in FFT is not supported");
            return;
        }
    }
    else {
        _dst->re_construct(_output_type, _src->Width(), _src->Height(), _src->Depth(), S);
    }

    switch (_output_type)
    {
    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<float>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F32_:
        decx::dsp::fft::_IFFT3D_caller_cplxf<de::CPf>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_UINT8_:
        decx::dsp::fft::_IFFT3D_caller_cplxd<uint8_t>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::dsp::fft::_IFFT3D_caller_cplxd<double>(_src, _dst, S, E);
        break;

    case de::_DATA_TYPES_FLAGS_::_COMPLEX_F64_:
        decx::dsp::fft::_IFFT3D_caller_cplxd<de::CPd>(_src, _dst, S, E);
        break;

    default:
        decx::err::handle_error_info_modify(de::GetLastError(), decx::DECX_error_types::DECX_FAIL_UNSUPPORTED_TYPE,
            UNSUPPORTED_TYPE);
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();
}
