#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "Conv2_fp32_kernels.cuh"


__global__
void decx::conv::GPUK::cu_sConv2_r8_within(const float4* __restrict   src, 
                         const float* __restrict    kernel,
                         float4* __restrict         dst,
                         const uint                 pitch_src, 
                         const uint                 pitch_dst,
                         const uint                 total_ker_len, 
                         const uint                 Wker,
                         const uint2                 kernel_shift,
                         const uint2                dst_dims)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ float src_frag[32][80 + sharedmem_offset];

    float4 reg_0, reg_1;

    uint glo_dex = idx * pitch_src + idy;
    reg_0 = src[glo_dex];
    glo_dex += 16 * pitch_src;
    reg_1 = src[glo_dex];

    store_to_shmem_L
    
    if (threadIdx.y < 4) {
        glo_dex = idx * pitch_src + idy + 16;
        reg_0 = src[glo_dex];
        glo_dex += 16 * pitch_src;
        reg_1 = src[glo_dex];

        store_to_shmem_R
    }

    __syncthreads();

    int dx, dy;
    float tmp_ker;
    reg_1 = make_float4(init_valuef, init_valuef, init_valuef, init_valuef);
    for (int i = 0; i < total_ker_len; ++i)
    {
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            reg_0.x = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy];
            reg_0.y = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 1];
            reg_0.z = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 2];
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        else {
            reg_shift_f(&reg_0);
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        tmp_ker = kernel[i];
        
        reg_1.x = fmaf(reg_0.x, tmp_ker, reg_1.x);    
        reg_1.y = fmaf(reg_0.y, tmp_ker, reg_1.y);    
        reg_1.z = fmaf(reg_0.z, tmp_ker, reg_1.z);    
        reg_1.w = fmaf(reg_0.w, tmp_ker, reg_1.w);    
    }

    glo_dex = idx * pitch_dst + idy;
    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = reg_1;
    }
}





__global__
void decx::conv::GPUK::cu_sConv2_r16_within(const float4* __restrict           src, 
                          const float* __restrict    kernel,
                          float4* __restrict           dst,
                          const uint        pitch_src, 
                          const uint        pitch_dst,
                          const uint        total_ker_len, 
                          const uint        Wker,
                          const uint2        kernel_shift,
                          const uint2       dst_dims)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    __shared__ float src_frag[48][96 + sharedmem_offset];

    float4 reg_0, reg_1;

    uint glo_dex = idx * pitch_src + idy;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(0)

    glo_dex += 16 * pitch_src;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(16)

    glo_dex += 16 * pitch_src;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(32)

    if (threadIdx.y < 8) {
        glo_dex = idx * pitch_src + idy + 16;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(0)

            glo_dex += 16 * pitch_src;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(16)

            glo_dex += 16 * pitch_src;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(32)
    }

    __syncthreads();

    int dx, dy;
    float tmp_ker;
    reg_1 = make_float4(init_valuef, init_valuef, init_valuef, init_valuef);
    for (int i = 0; i < total_ker_len; ++i)
    {
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            reg_0.x = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy];
            reg_0.y = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 1];
            reg_0.z = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 2];
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        else {
            reg_shift_f(&reg_0);
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        tmp_ker = kernel[i];
        
        reg_1.x = fmaf(reg_0.x, tmp_ker, reg_1.x);    
        reg_1.y = fmaf(reg_0.y, tmp_ker, reg_1.y);    
        reg_1.z = fmaf(reg_0.z, tmp_ker, reg_1.z);    
        reg_1.w = fmaf(reg_0.w, tmp_ker, reg_1.w);    
    }

    glo_dex = idx * pitch_dst + idy;
    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = reg_1;
    }
}




__global__
void decx::conv::GPUK::cu_sConv2_r816_within(const float4* __restrict    src, 
    const float* __restrict    kernel,
                           float4* __restrict    dst,
                           const uint            pitch_src, 
                           const uint            pitch_dst,
                           const uint            total_ker_len, 
                           const uint            Wker,
                           const uint2            kernel_shift,
                           const uint2           dst_dims)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ float src_frag[32][96 + sharedmem_offset];

    float4 reg_0, reg_1;

    uint glo_dex = idx * pitch_src + idy;
    reg_0 = src[glo_dex];
    glo_dex += 16 * pitch_src;
    reg_1 = src[glo_dex];

    store_to_shmem_L
    
    if (threadIdx.y < 8) {
        glo_dex = idx * pitch_src + idy + 16;
        reg_0 = src[glo_dex];
        glo_dex += 16 * pitch_src;
        reg_1 = src[glo_dex];

        store_to_shmem_R
    }

    __syncthreads();

    int dx, dy;
    float tmp_ker;
    reg_1 = make_float4(init_valuef, init_valuef, init_valuef, init_valuef);
    for (int i = 0; i < total_ker_len; ++i)
    {
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + i % Wker;
        if (dy == kernel_shift.y) {
            reg_0.x = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy];
            reg_0.y = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 1];
            reg_0.z = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 2];
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        else {
            reg_shift_f(&reg_0);
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        tmp_ker = kernel[i];

        reg_1.x = fmaf(reg_0.x, tmp_ker, reg_1.x);    
        reg_1.y = fmaf(reg_0.y, tmp_ker, reg_1.y);    
        reg_1.z = fmaf(reg_0.z, tmp_ker, reg_1.z);    
        reg_1.w = fmaf(reg_0.w, tmp_ker, reg_1.w);    
    }

    glo_dex = idx * pitch_dst + idy;
    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = reg_1;
    }
}




__global__
void decx::conv::GPUK::cu_sConv2_r168_within(const float4* __restrict         src, 
    const float* __restrict    kernel,
                           float4* __restrict               dst,
                           const uint            pitch_src, 
                           const uint            pitch_dst,
                           const uint            total_ker_len, 
                           const uint            Wker,
                           const uint2            kernel_shift,
                           const uint2           dst_dims)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ float src_frag[48][80 + sharedmem_offset];

    float4 reg_0, reg_1;

    uint glo_dex = idx * pitch_src + idy;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(0)

    glo_dex += 16 * pitch_src;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(16)

    glo_dex += 16 * pitch_src;
    reg_0 = src[glo_dex];
    store_to_shmem_L3(32)

    if (threadIdx.y < 4) {
        glo_dex = idx * pitch_src + idy + 16;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(0)

        glo_dex += 16 * pitch_src;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(16)

        glo_dex += 16 * pitch_src;
        reg_0 = src[glo_dex];
        store_to_shmem_R3(32)
    }

    __syncthreads();

    int dx, dy;
    float tmp_ker;
    reg_1 = make_float4(init_valuef, init_valuef, init_valuef, init_valuef);
    for (int i = 0; i < total_ker_len; ++i)
    {
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + i % Wker;
        if (dy == kernel_shift.y) {
            reg_0.x = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy];
            reg_0.y = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 1];
            reg_0.z = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 2];
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        else {
            reg_shift_f(&reg_0);
            reg_0.w = src_frag[threadIdx.x + dx][4 * (threadIdx.y) + dy + 3];
        }
        tmp_ker = kernel[i];

        reg_1.x = fmaf(reg_0.x, tmp_ker, reg_1.x);    
        reg_1.y = fmaf(reg_0.y, tmp_ker, reg_1.y);    
        reg_1.z = fmaf(reg_0.z, tmp_ker, reg_1.z);    
        reg_1.w = fmaf(reg_0.w, tmp_ker, reg_1.w);    
    }

    glo_dex = idx * pitch_dst + idy;
    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = reg_1;
    }
}