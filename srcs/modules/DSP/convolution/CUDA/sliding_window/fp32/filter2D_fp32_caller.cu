#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "../cuda_Filter2D_planner.cuh"
#include "../filter2D_kernel.cuh"


decx::ResourceHandle decx::dsp::_cuda_filter2D_fp32;


template <> template <uint32_t _ext_w> void decx::dsp::cuda_Filter2D_planner<float>::
_cu_Filter2D_fp32_caller(const decx::dsp::cuda_Filter2D_planner<float>* _fake_this, 
                           const float4* src,
                           const float* kernel, 
                           float4* dst, 
                           const uint32_t pitchdst_v1, 
                           decx::hip_stream* S)
{
    if (_fake_this->_conv_border_method != de::extend_label::_EXTEND_NONE_) 
    {
        checkCudaErrors(hipMemcpy2DAsync((float*)_fake_this->_ext_src._ptr.ptr + (_fake_this->_kernel_layout->width >> 1),
            _fake_this->_ext_src._dims.x * sizeof(float),
            src,
            _fake_this->_src_layout->pitch * sizeof(float),
            _fake_this->_src_layout->width * sizeof(float),
            _fake_this->_src_layout->height,
            hipMemcpyDeviceToDevice,
            S->get_raw_stream_ref()));
        
        decx::dsp::GPUK::cu_filter2D_BC_fp32<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (float4*)_fake_this->_ext_src._ptr.ptr,
            (float*)kernel,
            (float4*)dst,
            _fake_this->_ext_src._dims.x / 4,
            pitchdst_v1 / 4,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
    else
    {
        decx::dsp::GPUK::cu_filter2D_NB_fp32<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (float4*)src,
            (float*)kernel,
            (float4*)dst,
            _fake_this->_src_layout->pitch / 4,
            pitchdst_v1 / 4,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
}



namespace decx
{
namespace dsp {
    decx::dsp::_cu_F2_FP32_Kcaller _cu_F2_FP32_Kcallers[32] = 
    {
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<4>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<8>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<12>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<16>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<20>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<24>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<28>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<32>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<36>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<40>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<44>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<48>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<52>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<56>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<60>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<64>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<68>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<72>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<76>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<80>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<84>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<88>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<92>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<96>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<100>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<104>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<108>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<112>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<116>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<120>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<124>,
        &decx::dsp::cuda_Filter2D_planner<float>::_cu_Filter2D_fp32_caller<128>,
    };
}
}


template <> void
decx::dsp::cuda_Filter2D_planner<float>::run(decx::_GPU_Matrix* src, decx::_GPU_Matrix* kernel,
    decx::_GPU_Matrix* dst, decx::hip_stream* S, de::DH* handle)
{
    _cu_F2_FP32_Kcaller _kernel_ptr = decx::dsp::_cu_F2_FP32_Kcallers[(this->_kernel_layout->width - 2) / 4];

    _kernel_ptr(this, (float4*)src->Mat.ptr, (float*)kernel->Mat.ptr, (float4*)dst->Mat.ptr, dst->get_layout().pitch, S);
}
