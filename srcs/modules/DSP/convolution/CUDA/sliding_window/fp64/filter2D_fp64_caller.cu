#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "../cuda_Filter2D_planner.cuh"
#include "../filter2D_kernel.cuh"


decx::ResourceHandle decx::dsp::_cuda_filter2D_fp64;


template <> template <uint32_t _ext_w> void decx::dsp::cuda_Filter2D_planner<double>::
_cu_Filter2D_fp64_caller(const decx::dsp::cuda_Filter2D_planner<double>* _fake_this, 
                           const double2* src,
                           const double* kernel, 
                           double2* dst, 
                           const uint32_t pitchdst_v1, 
                           decx::hip_stream* S)
{
    if (_fake_this->_conv_border_method != de::extend_label::_EXTEND_NONE_) 
    {
        checkCudaErrors(hipMemcpy2DAsync((double*)_fake_this->_ext_src._ptr.ptr + (_fake_this->_kernel_layout->width >> 1),
            _fake_this->_ext_src._dims.x * sizeof(double),
            src,
            _fake_this->_src_layout->pitch * sizeof(double),
            _fake_this->_src_layout->width * sizeof(double),
            _fake_this->_src_layout->height,
            hipMemcpyDeviceToDevice,
            S->get_raw_stream_ref()));
        
        decx::dsp::GPUK::cu_filter2D_BC_fp64<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (double2*)_fake_this->_ext_src._ptr.ptr,
            (double*)kernel,
            (double2*)dst,
            _fake_this->_ext_src._dims.x / 2,
            pitchdst_v1 / 2,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
    else {
        decx::dsp::GPUK::cu_filter2D_NB_fp64<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (double2*)src,
            (double*)kernel,
            (double2*)dst,
            _fake_this->_src_layout->pitch / 2,
            pitchdst_v1 / 2,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
}



namespace decx
{
namespace dsp {
    decx::dsp::_cu_F2_FP64_Kcaller _cu_F2_FP64_Kcallers[32] = 
    {
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<2>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<4>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<6>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<8>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<10>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<12>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<14>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<16>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<18>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<20>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<22>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<24>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<26>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<28>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<30>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<32>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<34>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<36>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<38>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<40>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<42>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<44>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<46>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<48>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<50>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<52>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<54>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<56>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<58>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<60>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<62>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<64>,
    };
}
}



template <> void
decx::dsp::cuda_Filter2D_planner<double>::run(decx::_GPU_Matrix* src, decx::_GPU_Matrix* kernel,
    decx::_GPU_Matrix* dst, decx::hip_stream* S, de::DH* handle)
{
    _cu_F2_FP64_Kcaller _kernel_ptr = decx::dsp::_cu_F2_FP64_Kcallers[(this->_kernel_layout->width - 2) / 2];

    _kernel_ptr(this, (double2*)src->Mat.ptr, (double*)kernel->Mat.ptr, (double2*)dst->Mat.ptr, dst->get_layout().pitch, S);
}
