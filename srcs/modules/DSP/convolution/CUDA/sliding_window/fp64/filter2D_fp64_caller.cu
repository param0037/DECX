#include "hip/hip_runtime.h"
/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../cuda_filter2D_planner.cuh"
#include "../filter2D_kernel.cuh"


decx::ResourceHandle decx::dsp::_cuda_filter2D_fp64;


template <> template <uint32_t _ext_w> void decx::dsp::cuda_Filter2D_planner<double>::
_cu_Filter2D_fp64_caller(const decx::dsp::cuda_Filter2D_planner<double>* _fake_this, 
                           const double2* src,
                           const double* kernel, 
                           double2* dst, 
                           const uint32_t pitchdst_v1, 
                           decx::hip_stream* S)
{
    if (_fake_this->_conv_border_method != de::extend_label::_EXTEND_NONE_) 
    {
        checkCudaErrors(hipMemcpy2DAsync((double*)_fake_this->_ext_src._ptr.ptr + (_fake_this->_kernel_layout->width >> 1),
            _fake_this->_ext_src._dims.x * sizeof(double),
            src,
            _fake_this->_src_layout->pitch * sizeof(double),
            _fake_this->_src_layout->width * sizeof(double),
            _fake_this->_src_layout->height,
            hipMemcpyDeviceToDevice,
            S->get_raw_stream_ref()));
        
        decx::dsp::GPUK::cu_filter2D_BC_fp64<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (double2*)_fake_this->_ext_src._ptr.ptr,
            (double*)kernel,
            (double2*)dst,
            _fake_this->_ext_src._dims.x / 2,
            pitchdst_v1 / 2,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
    else {
        decx::dsp::GPUK::cu_filter2D_NB_fp64<_ext_w> << <_fake_this->_grid, _fake_this->_block, 
                                                    0, S->get_raw_stream_ref() >> > (
            (double2*)src,
            (double*)kernel,
            (double2*)dst,
            _fake_this->_src_layout->pitch / 2,
            pitchdst_v1 / 2,
            make_uint3(_fake_this->_kernel_layout->width, 
                       _fake_this->_kernel_layout->height, 
                       _fake_this->_kernel_layout->pitch),
            _fake_this->_dst_dims);
    }
}



namespace decx
{
namespace dsp {
    decx::dsp::_cu_F2_FP64_Kcaller _cu_F2_FP64_Kcallers[32] = 
    {
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<2>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<4>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<6>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<8>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<10>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<12>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<14>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<16>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<18>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<20>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<22>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<24>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<26>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<28>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<30>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<32>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<34>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<36>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<38>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<40>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<42>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<44>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<46>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<48>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<50>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<52>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<54>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<56>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<58>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<60>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<62>,
        &decx::dsp::cuda_Filter2D_planner<double>::_cu_Filter2D_fp64_caller<64>,
    };
}
}



template <> void
decx::dsp::cuda_Filter2D_planner<double>::run(decx::_GPU_Matrix* src, decx::_GPU_Matrix* kernel,
    decx::_GPU_Matrix* dst, decx::hip_stream* S, de::DH* handle)
{
    _cu_F2_FP64_Kcaller _kernel_ptr = decx::dsp::_cu_F2_FP64_Kcallers[(this->_kernel_layout->width - 2) / 2];

    _kernel_ptr(this, (double2*)src->Mat.ptr, (double*)kernel->Mat.ptr, (double2*)dst->Mat.ptr, dst->get_layout().pitch, S);
}
