#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "../cuda_Filter2D_planner.cuh"
#include "../filter2D_kernel.cuh"


template <> void
decx::dsp::cuda_Filter2D_planner<uint8_t>::run(decx::_GPU_Matrix* src, decx::_GPU_Matrix* kernel,
    decx::_GPU_Matrix* dst, decx::hip_stream* S, de::DH* handle)
{
    if (this->_conv_border_method != de::extend_label::_EXTEND_NONE_) 
    {
        checkCudaErrors(hipMemcpy2DAsync((uint8_t*)this->_ext_src._ptr.ptr + (this->_kernel_layout->width >> 1),
            _ext_src._dims.x * sizeof(uint8_t),
            src->Mat.ptr,
            this->_src_layout->pitch * sizeof(uint8_t),
            this->_src_layout->width * sizeof(uint8_t),
            this->_src_layout->height,
            hipMemcpyDeviceToDevice,
            S->get_raw_stream_ref()));
        
        decx::dsp::GPUK::cu_filter2D_BC_u8_fp32<32> << <this->_grid, this->_block, 0, S->get_raw_stream_ref() >> > (
            (double*)_ext_src._ptr.ptr,
            (float*)kernel->Mat.ptr,
            (float4*)dst->Mat.ptr,
            this->_ext_src._dims.x / 8,
            dst->get_layout().pitch / 8,
            make_uint3(kernel->Width(), kernel->Height(), kernel->get_layout().pitch),
            this->_dst_dims);
    }
    else
    {
        decx::dsp::GPUK::cu_filter2D_NB_u8_fp32<32> << <this->_grid, this->_block, 0, S->get_raw_stream_ref() >> > (
            (double*)src->Mat.ptr,
            (float*)kernel->Mat.ptr,
            (float4*)dst->Mat.ptr,
            this->_src_layout->pitch / 8,
            dst->get_layout().pitch / 8,
            make_uint3(kernel->Width(), kernel->Height(), kernel->get_layout().pitch),
            this->_dst_dims);
    }
}