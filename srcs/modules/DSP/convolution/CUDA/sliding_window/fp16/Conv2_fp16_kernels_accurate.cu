#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "Conv2_fp16_kernels_accurate.cuh"


__global__
void decx::conv::GPUK::cu_hConv2_r8_within_accu(const float4* __restrict               src,
                                                const __half* kernel,
                                                float4* __restrict               dst,
                                                const uint              pitch_src,
                                                const uint              pitch_dst,
                                                const uint              total_ker_len,
                                                const uint              Wker,
                                                const uint2              kernel_shift,
                                                const uint2             dst_dims)
{
#if __ABOVE_SM_53
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    __shared__ __half src_frag[32][144 + sharedmem_offset * 2];

    half2_8 reg_0, reg_1, reg_2;
    float fval_ker;

    uint glo_dex = idx * pitch_src + idy;
    *((float4*)&reg_0) = src[glo_dex];
    glo_dex += 16 * pitch_src;
    *((float4*)&reg_1) = src[glo_dex];

    hstore_to_shmem_L

        if (threadIdx.y < 2) {
            glo_dex = idx * pitch_src + idy + 16;
            *((float4*)&reg_0) = src[glo_dex];
            glo_dex += 16 * pitch_src;
            *((float4*)&reg_1) = src[glo_dex];

            hstore_to_shmem_R
        }

    __syncthreads();

    int dx, dy;
    *((uint4*)&reg_1) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    *((uint4*)&reg_2) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    for (int i = 0; i < total_ker_len; ++i)
    {
        // ���ﲢ����ÿ�ζ��ӹ����ڴ���ȫ�����ݣ�ƽ�ƾͿ�����
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            ((__half*)&reg_0)[0] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy];
            ((__half*)&reg_0)[1] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 1];
            ((__half*)&reg_0)[2] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 2];
            ((__half*)&reg_0)[3] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 3];
            ((__half*)&reg_0)[4] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 4];
            ((__half*)&reg_0)[5] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 5];
            ((__half*)&reg_0)[6] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 6];
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }
        else {
            reg_shift_fp16(&reg_0);
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }

        fval_ker = __half2float(kernel[i]);

        *((float*)&reg_1.x) = fmaf(__half2float(reg_0.x.x), fval_ker, *((float*)&reg_1.x));
        *((float*)&reg_1.y) = fmaf(__half2float(reg_0.x.y), fval_ker, *((float*)&reg_1.y));
        *((float*)&reg_1.z) = fmaf(__half2float(reg_0.y.x), fval_ker, *((float*)&reg_1.z));
        *((float*)&reg_1.w) = fmaf(__half2float(reg_0.y.y), fval_ker, *((float*)&reg_1.w));
        *((float*)&reg_2.x) = fmaf(__half2float(reg_0.z.x), fval_ker, *((float*)&reg_2.x));
        *((float*)&reg_2.y) = fmaf(__half2float(reg_0.z.y), fval_ker, *((float*)&reg_2.y));
        *((float*)&reg_2.z) = fmaf(__half2float(reg_0.w.x), fval_ker, *((float*)&reg_2.z));
        *((float*)&reg_2.w) = fmaf(__half2float(reg_0.w.y), fval_ker, *((float*)&reg_2.w));
    }

    glo_dex = idx * pitch_dst + idy;

    reg_0.x = __floats2half2_rn(*((float*)&reg_1.x), *((float*)&reg_1.y));
    reg_0.y = __floats2half2_rn(*((float*)&reg_1.z), *((float*)&reg_1.w));
    reg_0.z = __floats2half2_rn(*((float*)&reg_2.x), *((float*)&reg_2.y));
    reg_0.w = __floats2half2_rn(*((float*)&reg_2.z), *((float*)&reg_2.w));

    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = *((float4*)&reg_0);
    }
#endif
}





__global__
void decx::conv::GPUK::cu_hConv2_r16_within_accu(const float4* __restrict                src, 
                                                 const __half* kernel,
                                                 float4* __restrict                dst,
                                                 const uint             pitch_src, 
                                                 const uint             pitch_dst,
                                                 const uint             total_ker_len, 
                                                 const uint             Wker,
                                                 const uint2             kernel_shift,
                                                 const uint2            dst_dims)
{
#if __ABOVE_SM_53
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ __half src_frag[48][160 + sharedmem_offset * 2];

    half2_8 reg_0, reg_1, reg_2;
    float fval_ker;

    uint glo_dex = idx * pitch_src + idy;               *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(0)

    glo_dex += 16 * pitch_src;                          *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(16)

    glo_dex += 16 * pitch_src;                          *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(32)

    if (threadIdx.y < 4) {
        glo_dex = idx * pitch_src + idy + 16;           *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(0)

        glo_dex += 16 * pitch_src;                      *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(16)

        glo_dex += 16 * pitch_src;                      *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(32)
    }

    __syncthreads();

    int dx, dy;
    *((uint4*)&reg_1) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    *((uint4*)&reg_2) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    for (int i = 0; i < total_ker_len; ++i)
    {
        // ���ﲢ����ÿ�ζ��ӹ����ڴ���ȫ�����ݣ�ƽ�ƾͿ�����
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            ((__half*)&reg_0)[0] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy];
            ((__half*)&reg_0)[1] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 1];
            ((__half*)&reg_0)[2] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 2];
            ((__half*)&reg_0)[3] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 3];
            ((__half*)&reg_0)[4] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 4];
            ((__half*)&reg_0)[5] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 5];
            ((__half*)&reg_0)[6] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 6];
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }
        else {
            reg_shift_fp16(&reg_0);
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }

        fval_ker = __half2float(kernel[i]);

        *((float*)&reg_1.x) = fmaf(__half2float(reg_0.x.x), fval_ker, *((float*)&reg_1.x));
        *((float*)&reg_1.y) = fmaf(__half2float(reg_0.x.y), fval_ker, *((float*)&reg_1.y));
        *((float*)&reg_1.z) = fmaf(__half2float(reg_0.y.x), fval_ker, *((float*)&reg_1.z));
        *((float*)&reg_1.w) = fmaf(__half2float(reg_0.y.y), fval_ker, *((float*)&reg_1.w));
        *((float*)&reg_2.x) = fmaf(__half2float(reg_0.z.x), fval_ker, *((float*)&reg_2.x));
        *((float*)&reg_2.y) = fmaf(__half2float(reg_0.z.y), fval_ker, *((float*)&reg_2.y));
        *((float*)&reg_2.z) = fmaf(__half2float(reg_0.w.x), fval_ker, *((float*)&reg_2.z));
        *((float*)&reg_2.w) = fmaf(__half2float(reg_0.w.y), fval_ker, *((float*)&reg_2.w));
    }

    glo_dex = idx * pitch_dst + idy;

    reg_0.x = __floats2half2_rn(*((float*)&reg_1.x), *((float*)&reg_1.y));
    reg_0.y = __floats2half2_rn(*((float*)&reg_1.z), *((float*)&reg_1.w));
    reg_0.z = __floats2half2_rn(*((float*)&reg_2.x), *((float*)&reg_2.y));
    reg_0.w = __floats2half2_rn(*((float*)&reg_2.z), *((float*)&reg_2.w));

    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = *((float4*)&reg_0);
    }
#endif
}







__global__
void decx::conv::GPUK::cu_hConv2_r816_within_accu(const float4* __restrict          src,
                                                  const __half* kernel,
                                                  float4* __restrict                dst,
                                                  const uint             pitch_src, 
                                                  const uint             pitch_dst,
                                                  const uint             total_ker_len, 
                                                  const uint             Wker,
                                                  const uint2             kernel_shift,
                                                  const uint2            dst_dims)
{
#if __ABOVE_SM_53
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ __half src_frag[32][160 + sharedmem_offset * 2];

    half2_8 reg_0, reg_1, reg_2;
    float fval_ker;

    uint glo_dex = idx * pitch_src + idy;               *((float4*)&reg_0) = src[glo_dex];
    glo_dex += 16 * pitch_src;                          *((float4*)&reg_1) = src[glo_dex];

    hstore_to_shmem_L
    
    if (threadIdx.y < 4) {
        glo_dex = idx * pitch_src + idy + 16;           *((float4*)&reg_0) = src[glo_dex];
        glo_dex += 16 * pitch_src;                      *((float4*)&reg_1) = src[glo_dex];

        hstore_to_shmem_R
    }

    __syncthreads();

    int dx, dy;
    *((uint4*)&reg_1) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    *((uint4*)&reg_2) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);

    for (int i = 0; i < total_ker_len; ++i)
    {
        // ���ﲢ����ÿ�ζ��ӹ����ڴ���ȫ�����ݣ�ƽ�ƾͿ�����
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            ((__half*)&reg_0)[0] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy];
            ((__half*)&reg_0)[1] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 1];
            ((__half*)&reg_0)[2] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 2];
            ((__half*)&reg_0)[3] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 3];
            ((__half*)&reg_0)[4] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 4];
            ((__half*)&reg_0)[5] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 5];
            ((__half*)&reg_0)[6] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 6];
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }
        else {
            reg_shift_fp16(&reg_0);
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }

        fval_ker = __half2float(kernel[i]);

        *((float*)&reg_1.x) = fmaf(__half2float(reg_0.x.x), fval_ker, *((float*)&reg_1.x));
        *((float*)&reg_1.y) = fmaf(__half2float(reg_0.x.y), fval_ker, *((float*)&reg_1.y));
        *((float*)&reg_1.z) = fmaf(__half2float(reg_0.y.x), fval_ker, *((float*)&reg_1.z));
        *((float*)&reg_1.w) = fmaf(__half2float(reg_0.y.y), fval_ker, *((float*)&reg_1.w));
        *((float*)&reg_2.x) = fmaf(__half2float(reg_0.z.x), fval_ker, *((float*)&reg_2.x));
        *((float*)&reg_2.y) = fmaf(__half2float(reg_0.z.y), fval_ker, *((float*)&reg_2.y));
        *((float*)&reg_2.z) = fmaf(__half2float(reg_0.w.x), fval_ker, *((float*)&reg_2.z));
        *((float*)&reg_2.w) = fmaf(__half2float(reg_0.w.y), fval_ker, *((float*)&reg_2.w));
    }

    glo_dex = idx * pitch_dst + idy;

    reg_0.x = __floats2half2_rn(*((float*)&reg_1.x), *((float*)&reg_1.y));
    reg_0.y = __floats2half2_rn(*((float*)&reg_1.z), *((float*)&reg_1.w));
    reg_0.z = __floats2half2_rn(*((float*)&reg_2.x), *((float*)&reg_2.y));
    reg_0.w = __floats2half2_rn(*((float*)&reg_2.z), *((float*)&reg_2.w));

    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = *((float4*)&reg_0);
    }
#endif
}




__global__
/**
* The radius of convolutional kernel = 8��ÿ���̴߳���1x4������(one float4)��һ����16x16���̣߳�
* ��һ������Ҫ�Ĺ����ڴ�СΪ(16 * 4 + 8 * 2)*(16 + 8 * 2) ��shmem[32][80]
* So the alignments should be x64 in width, and x16 in height for Ddst
* The dims of Dsrc should be plus 8 * 2 = 16 on all directions(if float4 is consider horizentally, then +4 at width)
*
* ������64 x 16(floats), �⻷��8 x 8(floats)
* constant area: 64 x 16(floats), apron area: 8 x 8(floats)
* �����ά����8����
* */
void decx::conv::GPUK::cu_hConv2_r168_within_accu(const float4* __restrict          src,
                                                  const __half* kernel,
                                                  float4* __restrict                dst,
                                                  const uint              pitch_src,
                                                  const uint              pitch_dst,
                                                  const uint              total_ker_len,
                                                  const uint              Wker,
                                                  const uint2              kernel_shift,
                                                  const uint2             dst_dims)
{
#if __ABOVE_SM_53
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    __shared__ __half src_frag[48][144 + sharedmem_offset * 2];

    half2_8 reg_0, reg_1, reg_2;
    float fval_ker;

    uint glo_dex = idx * pitch_src + idy;               *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(0)

    glo_dex += 16 * pitch_src;                          *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(16)

    glo_dex += 16 * pitch_src;                          *((float4*)&reg_0) = src[glo_dex];
    hstore_to_shmem_L3(32)

    if (threadIdx.y < 2) {
        glo_dex = idx * pitch_src + idy + 16;           *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(0)

        glo_dex += 16 * pitch_src;                      *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(16)

        glo_dex += 16 * pitch_src;                      *((float4*)&reg_0) = src[glo_dex];
        hstore_to_shmem_R3(32)
    }

    __syncthreads();

    int dx, dy;
    *((uint4*)&reg_1) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);
    *((uint4*)&reg_2) = make_uint4(init_valueUint, init_valueUint, init_valueUint, init_valueUint);

    for (int i = 0; i < total_ker_len; ++i)
    {
        // ���ﲢ����ÿ�ζ��ӹ����ڴ���ȫ�����ݣ�ƽ�ƾͿ�����
        dx = kernel_shift.x + i / Wker;        dy = kernel_shift.y + (i % Wker);
        if (dy == kernel_shift.y) {
            ((__half*)&reg_0)[0] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy];
            ((__half*)&reg_0)[1] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 1];
            ((__half*)&reg_0)[2] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 2];
            ((__half*)&reg_0)[3] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 3];
            ((__half*)&reg_0)[4] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 4];
            ((__half*)&reg_0)[5] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 5];
            ((__half*)&reg_0)[6] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 6];
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }
        else {
            reg_shift_fp16(&reg_0);
            ((__half*)&reg_0)[7] = src_frag[threadIdx.x + dx][8 * (threadIdx.y) + dy + 7];
        }

        fval_ker = __half2float(kernel[i]);

        *((float*)&reg_1.x) = fmaf(__half2float(reg_0.x.x), fval_ker, *((float*)&reg_1.x));
        *((float*)&reg_1.y) = fmaf(__half2float(reg_0.x.y), fval_ker, *((float*)&reg_1.y));
        *((float*)&reg_1.z) = fmaf(__half2float(reg_0.y.x), fval_ker, *((float*)&reg_1.z));
        *((float*)&reg_1.w) = fmaf(__half2float(reg_0.y.y), fval_ker, *((float*)&reg_1.w));
        *((float*)&reg_2.x) = fmaf(__half2float(reg_0.z.x), fval_ker, *((float*)&reg_2.x));
        *((float*)&reg_2.y) = fmaf(__half2float(reg_0.z.y), fval_ker, *((float*)&reg_2.y));
        *((float*)&reg_2.z) = fmaf(__half2float(reg_0.w.x), fval_ker, *((float*)&reg_2.z));
        *((float*)&reg_2.w) = fmaf(__half2float(reg_0.w.y), fval_ker, *((float*)&reg_2.w));
    }

    glo_dex = idx * pitch_dst + idy;

    reg_0.x = __floats2half2_rn(*((float*)&reg_1.x), *((float*)&reg_1.y));
    reg_0.y = __floats2half2_rn(*((float*)&reg_1.z), *((float*)&reg_1.w));
    reg_0.z = __floats2half2_rn(*((float*)&reg_2.x), *((float*)&reg_2.y));
    reg_0.w = __floats2half2_rn(*((float*)&reg_2.z), *((float*)&reg_2.w));
    if (idx < dst_dims.y && idy < dst_dims.x) {
        dst[glo_dex] = *((float4*)&reg_0);
    }
#endif
}