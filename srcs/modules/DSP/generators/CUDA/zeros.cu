/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "zeros.cuh"
#include "../../../core/cudaStream_management/cudaEvent_queue.h"
#include "../../../core/cudaStream_management/cudaStream_queue.h"


_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Vector& src)
{
    de::DH handle;
    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Vec.block, _src->total_bytes, 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}




_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Matrix& src)
{
    de::DH handle;
    decx::_GPU_Matrix* _src = dynamic_cast<decx::_GPU_Matrix*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Mat.block, _src->get_total_bytes(), 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}


_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Tensor& src)
{
    de::DH handle;
    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL; 
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Tens.block, _src->total_bytes, 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}