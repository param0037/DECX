/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "zeros.cuh"
#include "../../../core/cudaStream_management/cudaEvent_queue.h"
#include "../../../core/cudaStream_management/cudaStream_queue.h"


_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Vector& src)
{
    de::DH handle;
    decx::_GPU_Vector* _src = dynamic_cast<decx::_GPU_Vector*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Vec.block, _src->total_bytes, 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}




_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Matrix& src)
{
    de::DH handle;
    decx::_GPU_Matrix* _src = dynamic_cast<decx::_GPU_Matrix*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Mat.block, _src->get_total_bytes(), 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}


_DECX_API_ de::DH de::gen::cuda::Zeros(de::GPU_Tensor& src)
{
    de::DH handle;
    decx::_GPU_Tensor* _src = dynamic_cast<decx::_GPU_Tensor*>(&src);

    if (!_src->is_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CLASS_NOT_INIT, CLASS_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM, CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL; 
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT, CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    decx::alloc::Memset_D(_src->Tens.block, _src->total_bytes, 0, S->get_raw_stream_ptr());

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}