#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "GPU_Vector.h"



void decx::_GPU_Vector::_attribute_assign(const int _type, size_t length)
{
    this->type = _type;
    this->_single_element_size = decx::core::_size_mapping(_type);

    uint _alignment = 0;
    switch (this->_single_element_size)
    {
    case _SIZE_INT32_:
        _alignment = _VECTOR_ALIGN_4B_;     break;
    case _SIZE_FLOAT64_:
        _alignment = _VECTOR_ALIGN_8B_;     break;
    case _SIZE_FLOAT16_:
        _alignment = _VECTOR_ALIGN_2B_;     break;
    case _SIZE_UINT8_:
        _alignment = _VECTOR_ALIGN_1B_;     break;
    default:
        break;
    }
    this->length = length;
    this->_init = true;
    this->_length = decx::utils::ceil<size_t>(length, (size_t)_alignment) * (size_t)_alignment;
    this->total_bytes = this->_length * this->_single_element_size;
}



int decx::_GPU_Vector::Type()
{
    return this->type;
}


void decx::_GPU_Vector::alloc_data_space()
{
    if (decx::alloc::_device_malloc<void>(&this->Vec, this->total_bytes)) {
        SetConsoleColor(4);
        printf("Vector on GPU malloc failed! Please check if there is enough space in your device.");
        ResetConsoleColor;
        return;
    }
}


void decx::_GPU_Vector::construct(const int _type, size_t length)
{
    this->_attribute_assign(_type, length);

    this->alloc_data_space();
}


void decx::_GPU_Vector::re_construct(const int _type, size_t length)
{
    if (this->type != _type || this->length != _length) {
        const size_t pre_size = this->total_bytes;

        this->_attribute_assign(_type, length);

        if (this->total_bytes > pre_size) {
            this->alloc_data_space();
        }
    }
}


decx::_GPU_Vector::_GPU_Vector(const int _type, size_t length)
{
    this->construct(_type, length);
}



decx::_GPU_Vector::_GPU_Vector()
{
    this->_attribute_assign(decx::_DATA_TYPES_FLAGS_::_VOID_, 0);
    this->_init = false;
}



void decx::_GPU_Vector::load_from_host(de::Vector& src)
{
    decx::_Vector* _src = dynamic_cast<decx::_Vector*>(&src);
    checkCudaErrors(hipMemcpy(this->Vec.ptr, _src->Vec.ptr, this->length * this->_single_element_size, hipMemcpyHostToDevice));
}



void decx::_GPU_Vector::load_to_host(de::Vector& dst)
{
    decx::_Vector* _dst = dynamic_cast<decx::_Vector*>(&dst);
    checkCudaErrors(hipMemcpy(_dst->Vec.ptr, this->Vec.ptr, this->length * this->_single_element_size, hipMemcpyDeviceToHost));
}




void decx::_GPU_Vector::release()
{
    decx::alloc::_device_dealloc(&this->Vec);
}



de::GPU_Vector& decx::_GPU_Vector::SoftCopy(de::GPU_Vector& src)
{
    const decx::_GPU_Vector& ref_src = dynamic_cast<decx::_GPU_Vector&>(src);

    this->_attribute_assign(ref_src.type, ref_src.length);
    decx::alloc::_device_malloc_same_place(&this->Vec);

    return *this;
}



namespace de
{
    _DECX_API_ de::GPU_Vector& CreateGPUVectorRef();


    _DECX_API_ de::GPU_Vector* CreateGPUVectorPtr();


    _DECX_API_ de::GPU_Vector& CreateGPUVectorRef(const int _type, const size_t length);


    _DECX_API_ de::GPU_Vector* CreateGPUVectorPtr(const int _type, const size_t length);
}



de::GPU_Vector& de::CreateGPUVectorRef() {
    return *(new decx::_GPU_Vector());
}



de::GPU_Vector* de::CreateGPUVectorPtr() {
    return new decx::_GPU_Vector();
}




de::GPU_Vector& de::CreateGPUVectorRef(const int _type, const size_t length) {
    return *(new decx::_GPU_Vector(_type, length));
}



de::GPU_Vector* de::CreateGPUVectorPtr(const int _type, const size_t length) {
    return new decx::_GPU_Vector(_type, length);
}
