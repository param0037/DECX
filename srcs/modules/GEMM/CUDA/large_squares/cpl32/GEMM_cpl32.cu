#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "GEMM_cpl32.cuh"

//
//// last storage (16, 16)
//// 计算 / 访存 比 is the crucial, reduce memory assess by vectorization
//__global__
///**
//* config -> <<<dim3(h / 128, w / 128, 1), int(16 * 16), 0, S>>>
//* __same should be 16-times and dstDims should be both 128-times
//* @param pitch_A : considered float4, is the true width on device memory (>= ~.width)
//* @param pitch_B : considered float4, is the true width on device memory (>= ~.width)
//* @param pitch_dst : considered float4
//* @param __iter : __linear(in float) / 16
//*/
//void cu_GEMM_cpl32_spec(float4 *                  A,
//                        float4 *                  B,
//                        float4 *                  dst,
//                        const uint                pitch_A,       // in float4
//                        const uint                pitch_B,       // in float4
//                        const uint                __iter)
//{
//    uint x_glo;
//    uint y_glo;
//    
//    __shared__ double4 shmemA[32][128 / 8 + 1];
//    __shared__ float4 shmemB[16][128 / 8 + 1];
//    
//    decx::utils::_cuda_vec128 sum[8];
//    
//    decx::utils::_cuda_vec128 tmp_A[4];
//    decx::utils::_cuda_vec128 tmp_B;
//
//    size_t glo_dex_A = 0, glo_dex_B = 0;
//    
//#pragma unroll 4
//    for (int _lane_id = 0; _lane_id < 4; ++_lane_id) {
//        Init_Sum_Union;
//
//        glo_dex_A = ((threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x) * pitch_A + (threadIdx.x % 4) * 2;
//        glo_dex_B = (threadIdx.x / 16) * pitch_B + ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;
//
//        for (uint i = 0; i < __iter; ++i)
//        {
//            tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
//            tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
//            tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
//            tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1
//
//            x_glo = 4 * (threadIdx.x % 4);            y_glo = (threadIdx.x % 16) / 4;
//
//            *((double*)&(shmemA[x_glo][(threadIdx.x / 16)]) + y_glo) = tmp_A[0]._vd.x;
//            *((double*)&(shmemA[x_glo + 1][(threadIdx.x / 16)]) + y_glo) = tmp_A[0]._vd.y;
//            *((double*)&(shmemA[x_glo + 2][(threadIdx.x / 16)]) + y_glo) = tmp_A[1]._vd.x;
//            *((double*)&(shmemA[x_glo + 3][(threadIdx.x / 16)]) + y_glo) = tmp_A[1]._vd.y;
//
//            *((double*)&(shmemA[x_glo + 16][(threadIdx.x / 16)]) + y_glo) = tmp_A[2]._vd.x;
//            *((double*)&(shmemA[x_glo + 17][(threadIdx.x / 16)]) + y_glo) = tmp_A[2]._vd.y;
//            *((double*)&(shmemA[x_glo + 18][(threadIdx.x / 16)]) + y_glo) = tmp_A[3]._vd.x;
//            *((double*)&(shmemA[x_glo + 19][(threadIdx.x / 16)]) + y_glo) = tmp_A[3]._vd.y;
//
//            x_glo = threadIdx.x / 16;            y_glo = threadIdx.x % 16;
//
//            tmp_B._vf = B[glo_dex_B];
//            shmemB[x_glo][y_glo] = tmp_B._vf;            //load globalB to shmemB
//
//            __syncthreads();
//
//            glo_dex_A += 8;
//            glo_dex_B += 16 * pitch_B;
//
//#pragma unroll 16
//            for (uint __line = 0; __line < 16; ++__line)
//            {
//                tmp_A[0]._vf = *((float4*)&shmemA[__line][x_glo].x);
//                tmp_A[1]._vf = *((float4*)&shmemA[__line][x_glo].z);
//                tmp_A[2]._vf = *((float4*)&shmemA[__line + 16][x_glo].x);
//                tmp_A[3]._vf = *((float4*)&shmemA[__line + 16][x_glo].z);
//
//                tmp_B._vf = shmemB[__line][y_glo];
//
//                cpl32fma_8x8;
//            }
//            __syncthreads();
//        }
//        x_glo = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
//        y_glo = (threadIdx.x % 16 + blockIdx.y * 16);
//        glo_dex_A = x_glo * pitch_B + y_glo * 4 + _lane_id;
//        cpl32_store(glo_dex_A);
//    }
//}
//


__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_spec(float4*                   A,
                        float4*                   B,
                        float4*                   dst,
                        const uint                pitch_A,       // in float4
                        const uint                pitch_B,       // in float4
                        const uint                __iter)
{
    uint x_glo;
    uint y_glo;
    
    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];
    
    decx::utils::_cuda_vec128 sum[8];
    
    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;
    
#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id) {
        Init_Sum_Union;

        glo_dex_A = ((threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x) * pitch_A + (threadIdx.x % 4) * 2;
        glo_dex_B = (threadIdx.x / 16) * pitch_B + ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
            tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
            tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1

            x_glo = 4 * (threadIdx.x % 4);            y_glo = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_glo][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_glo + 1][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_glo + 2][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_glo + 3][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[1]._vd.y;
               
            *((double*)&(shmemA[x_glo + 16][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_glo + 17][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_glo + 18][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_glo + 19][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[3]._vd.y;

            x_glo = threadIdx.x / 16;            y_glo = threadIdx.x % 16;

            tmp_B._vf = B[glo_dex_B];
            shmemB[x_glo][y_glo] = tmp_B._vf;            //load globalB to shmemB

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_glo * 2];
                tmp_A[1]._vf = shmemA[__line][x_glo * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_glo * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_glo * 2 + 1];

                tmp_B._vf = shmemB[__line][y_glo];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_glo = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_glo = (threadIdx.x % 16 + blockIdx.y * 16);
        glo_dex_A = x_glo * pitch_B + y_glo * 4 + _lane_id;
        cpl32_store(glo_dex_A);
    }
}



__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_ABC_spec(float4 *                A,
                            float4 *                B,
                            float4 *                C,
                            float4 *                dst,
                            const uint              pitch_A,
                            const uint              pitch_B,
                            const uint              __iter)
{
    uint x_glo;
    uint y_glo;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;

#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id) 
    {
        x_glo = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_glo = (threadIdx.x % 16 + blockIdx.y * 16);
        glo_dex_A = x_glo * pitch_B + y_glo * 4 + _lane_id;
        cpl32_loadC(glo_dex_A);

        glo_dex_A = ((threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x) * pitch_A + (threadIdx.x % 4) * 2;
        glo_dex_B = (threadIdx.x / 16) * pitch_B + ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
            tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
            tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1

            x_glo = 4 * (threadIdx.x % 4);            y_glo = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_glo][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_glo + 1][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_glo + 2][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_glo + 3][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_glo + 16][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_glo + 17][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_glo + 18][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_glo + 19][(threadIdx.x / 16) * 2]) + y_glo) = tmp_A[3]._vd.y;

            x_glo = threadIdx.x / 16;            y_glo = threadIdx.x % 16;

            tmp_B._vf = B[glo_dex_B];
            shmemB[x_glo][y_glo] = tmp_B._vf;            //load globalB to shmemB

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_glo * 2];
                tmp_A[1]._vf = shmemA[__line][x_glo * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_glo * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_glo * 2 + 1];

                tmp_B._vf = shmemB[__line][y_glo];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_glo = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_glo = (threadIdx.x % 16 + blockIdx.y * 16);
        glo_dex_A = x_glo * pitch_B + y_glo * 4 + _lane_id;
        cpl32_store(glo_dex_A);
    }
}




__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_anyWH_specL(float4*                   A,
                              float4*                   B,
                              float4*                   dst,
                              const uint                pitch_A,
                              const uint                pitch_B,
                              const uint                Hdst,
                              const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;
    
#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id)
    {
        Init_Sum_Union;

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            if (x_gloA < Hdst) {
                tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
                tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            }
            if (x_gloA + 4 < Hdst) {
                tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
                tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1
            }

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            if (y_gloB < pitch_B)   tmp_B._vf = B[glo_dex_B];
            shmemB[x_loc][y_loc] = tmp_B._vf;            //load globalB to shmemB

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)          cpl32_store_one_line(0, glo_dex_A);
            if (x_gloA + 1 < Hdst)      cpl32_store_one_line(1, glo_dex_A);
            if (x_gloA + 2 < Hdst)      cpl32_store_one_line(2, glo_dex_A);
            if (x_gloA + 3 < Hdst)      cpl32_store_one_line(3, glo_dex_A);
            if (x_gloA + 4 < Hdst)      cpl32_store_one_line(4, glo_dex_A);
            if (x_gloA + 5 < Hdst)      cpl32_store_one_line(5, glo_dex_A);
            if (x_gloA + 6 < Hdst)      cpl32_store_one_line(6, glo_dex_A);
            if (x_gloA + 7 < Hdst)      cpl32_store_one_line(7, glo_dex_A);
        }
    }
}




__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_ABC_anyWH_specL(float4*                   A,
                                   float4*                   B,
                                   float4*                   C,
                                   float4*                   dst,
                                   const uint                pitch_A,
                                   const uint                pitch_B,
                                   const uint                Hdst,
                                   const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;

#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id)
    {
        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)          { sum[0]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 1 < Hdst)      { sum[1]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 2 < Hdst)      { sum[2]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 3 < Hdst)      { sum[3]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 4 < Hdst)      { sum[4]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 5 < Hdst)      { sum[5]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 6 < Hdst)      { sum[6]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 7 < Hdst)      { sum[7]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
        }

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            if (x_gloA < Hdst) {
                tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
                tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            }
            if (x_gloA + 4 < Hdst) {
                tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
                tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1
            }

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            if (y_gloB < pitch_B)   tmp_B._vf = B[glo_dex_B];
            shmemB[x_loc][y_loc] = tmp_B._vf;            //load globalB to shmemB

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)          cpl32_store_one_line(0, glo_dex_A);
            if (x_gloA + 1 < Hdst)      cpl32_store_one_line(1, glo_dex_A);
            if (x_gloA + 2 < Hdst)      cpl32_store_one_line(2, glo_dex_A);
            if (x_gloA + 3 < Hdst)      cpl32_store_one_line(3, glo_dex_A);
            if (x_gloA + 4 < Hdst)      cpl32_store_one_line(4, glo_dex_A);
            if (x_gloA + 5 < Hdst)      cpl32_store_one_line(5, glo_dex_A);
            if (x_gloA + 6 < Hdst)      cpl32_store_one_line(6, glo_dex_A);
            if (x_gloA + 7 < Hdst)      cpl32_store_one_line(7, glo_dex_A);
        }
    }
}



__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_anyWH_anyL(float4*                   A,
                              float4*                   B,
                              float4*                   dst,
                              const uint                pitch_A,        // in float4
                              const uint                pitch_B,        // in float4
                              const uint                Hdst,
                              const uint                HB,
                              const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;
    
    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;
    
#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id) 
    {
        Init_Sum_Union;

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            if (y_gloA < pitch_A) {
                if (x_gloA < Hdst) {
                    tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
                    tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
                }
                if (x_gloA + 4 < Hdst) {
                    tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
                    tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1
                }
            }

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;
                                                                        
            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            if (x_gloB < HB) {
                if (y_gloB < pitch_B)   tmp_B._vf = B[glo_dex_B];
            }

            shmemB[x_loc][y_loc] = tmp_B._vf;

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)          cpl32_store_one_line(0, glo_dex_A);
            if (x_gloA + 1 < Hdst)      cpl32_store_one_line(1, glo_dex_A);
            if (x_gloA + 2 < Hdst)      cpl32_store_one_line(2, glo_dex_A);
            if (x_gloA + 3 < Hdst)      cpl32_store_one_line(3, glo_dex_A);
            if (x_gloA + 4 < Hdst)      cpl32_store_one_line(4, glo_dex_A);
            if (x_gloA + 5 < Hdst)      cpl32_store_one_line(5, glo_dex_A);
            if (x_gloA + 6 < Hdst)      cpl32_store_one_line(6, glo_dex_A);
            if (x_gloA + 7 < Hdst)      cpl32_store_one_line(7, glo_dex_A);
        }
    }
}



__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_ABC_anyWH_anyL(float4*                   A,
                                 float4*                   B,
                                 float4*                   C,
                                 float4*                   dst,
                                 const uint                pitch_A,
                                 const uint                pitch_B,
                                 const uint                Hdst,
                                 const uint                HB,
                                 const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;

#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id)
    {
        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)      { sum[0]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 1 < Hdst)  { sum[1]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 2 < Hdst)  { sum[2]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 3 < Hdst)  { sum[3]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 4 < Hdst)  { sum[4]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 5 < Hdst)  { sum[5]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 6 < Hdst)  { sum[6]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
            if (x_gloA + 7 < Hdst)  { sum[7]._vf = C[glo_dex_A];       glo_dex_A += pitch_B; }
        }

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            if (y_gloA < pitch_A) {
                if (x_gloA < Hdst) {
                    tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
                    tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
                }
                if (x_gloA + 4 < Hdst) {
                    tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
                    tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1
                }
            }

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            if (x_gloB < HB) {
                if (y_gloB < pitch_B)   tmp_B._vf = B[glo_dex_B];
            }

            shmemB[x_loc][y_loc] = tmp_B._vf;

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;

        if (y_gloA < pitch_B) {
            if (x_gloA < Hdst)          cpl32_store_one_line(0, glo_dex_A);
            if (x_gloA + 1 < Hdst)      cpl32_store_one_line(1, glo_dex_A);
            if (x_gloA + 2 < Hdst)      cpl32_store_one_line(2, glo_dex_A);
            if (x_gloA + 3 < Hdst)      cpl32_store_one_line(3, glo_dex_A);
            if (x_gloA + 4 < Hdst)      cpl32_store_one_line(4, glo_dex_A);
            if (x_gloA + 5 < Hdst)      cpl32_store_one_line(5, glo_dex_A);
            if (x_gloA + 6 < Hdst)      cpl32_store_one_line(6, glo_dex_A);
            if (x_gloA + 7 < Hdst)      cpl32_store_one_line(7, glo_dex_A);
        }
    }
}



__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_specWH_anyL(float4*                   A,
                              float4*                   B,
                              float4*                   dst,
                              const uint                pitch_A,
                              const uint                pitch_B,
                              const uint                HB,
                              const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;

#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id)
    {
        Init_Sum_Union;

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
            tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
            tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            tmp_B._vf = B[glo_dex_B];
            shmemB[x_loc][y_loc] = tmp_B._vf;            //load globalB to shmemB

            shmemB[x_loc][y_loc] = tmp_B._vf;

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;
        cpl32_store(glo_dex_A);
    }
}



__global__
void decx::gemm::GPUK::cu_GEMM_cpl32_ABC_specWH_anyL(float4*                   A,
                                  float4*                   B,
                                  float4*                   C,
                                  float4*                   dst,
                                  const uint                pitch_A,
                                  const uint                pitch_B,
                                  const uint                HB,
                                  const uint                __iter)
{
    uint x_gloA, y_gloA, x_gloB, y_gloB;
    uint x_loc, y_loc;

    __shared__ float4 shmemA[32][128 / 4 + 1];
    __shared__ float4 shmemB[16][128 / 8 + 1];

    decx::utils::_cuda_vec128 sum[8];

    decx::utils::_cuda_vec128 tmp_A[4];
    decx::utils::_cuda_vec128 tmp_B;

    size_t glo_dex_A = 0, glo_dex_B = 0;

#pragma unroll 4
    for (int _lane_id = 0; _lane_id < 4; ++_lane_id)
    {
        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;
        cpl32_loadC(glo_dex_A);

        x_gloA = (threadIdx.x / 16) * 8 + ((threadIdx.x / 4) % 4) + 128 * blockIdx.x;
        y_gloA = (threadIdx.x % 4) * 2;
        x_gloB = (threadIdx.x / 16);
        y_gloB = ((threadIdx.x % 16) + blockIdx.y * 16) * 4 + _lane_id;

        glo_dex_A = x_gloA * pitch_A + y_gloA;
        glo_dex_B = x_gloB * pitch_B + y_gloB;

        for (uint i = 0; i < __iter; ++i)
        {
            tmp_A[0]._vf = make_float4(0, 0, 0, 0);         tmp_A[1]._vf = make_float4(0, 0, 0, 0);
            tmp_A[2]._vf = make_float4(0, 0, 0, 0);         tmp_A[3]._vf = make_float4(0, 0, 0, 0);
            tmp_B._vf = make_float4(0, 0, 0, 0);

            tmp_A[0]._vf = A[glo_dex_A];                        // lane 0
            tmp_A[1]._vf = A[glo_dex_A + 1];                    // lane 1
            tmp_A[2]._vf = A[glo_dex_A + pitch_A * 4];          // lane 0
            tmp_A[3]._vf = A[glo_dex_A + pitch_A * 4 + 1];      // lane 1

            x_loc = 4 * (threadIdx.x % 4);            y_loc = (threadIdx.x % 16) / 4;

            *((double*)&(shmemA[x_loc][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.x;
            *((double*)&(shmemA[x_loc + 1][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[0]._vd.y;
            *((double*)&(shmemA[x_loc + 2][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.x;
            *((double*)&(shmemA[x_loc + 3][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[1]._vd.y;

            *((double*)&(shmemA[x_loc + 16][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.x;
            *((double*)&(shmemA[x_loc + 17][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[2]._vd.y;
            *((double*)&(shmemA[x_loc + 18][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.x;
            *((double*)&(shmemA[x_loc + 19][(threadIdx.x / 16) * 2]) + y_loc) = tmp_A[3]._vd.y;

            x_loc = threadIdx.x / 16;            y_loc = threadIdx.x % 16;

            tmp_B._vf = B[glo_dex_B];
            shmemB[x_loc][y_loc] = tmp_B._vf;            //load globalB to shmemB

            shmemB[x_loc][y_loc] = tmp_B._vf;

            __syncthreads();

            glo_dex_A += 8;
            glo_dex_B += 16 * pitch_B;
            y_gloA += 8;
            x_gloB += 16;

#pragma unroll 16
            for (uint __line = 0; __line < 16; ++__line)
            {
                tmp_A[0]._vf = shmemA[__line][x_loc * 2];
                tmp_A[1]._vf = shmemA[__line][x_loc * 2 + 1];
                tmp_A[2]._vf = shmemA[__line + 16][x_loc * 2];
                tmp_A[3]._vf = shmemA[__line + 16][x_loc * 2 + 1];

                tmp_B._vf = shmemB[__line][y_loc];

                cpl32fma_8x8;
            }
            __syncthreads();
        }

        x_gloA = (threadIdx.x / 16 + blockIdx.x * 16) * 8;
        y_gloA = (threadIdx.x % 16 + blockIdx.y * 16) * 4 + _lane_id;
        glo_dex_A = x_gloA * pitch_B + y_gloA;
        cpl32_store(glo_dex_A);
    }
}
