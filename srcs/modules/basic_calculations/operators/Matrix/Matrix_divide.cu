/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/



#include "../Div_kernel.cuh"
#include "../../../core/basic.h"
#include "Matrix_operators.h"
#include "../../../core/cudaStream_management/cudaEvent_queue.h"
#include "../../../core/cudaStream_management/cudaStream_queue.h"




de::DH de::cuda::Div(de::GPU_Matrix& A, de::GPU_Matrix& B, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _A = dynamic_cast<decx::_GPU_Matrix&>(A);
    decx::_GPU_Matrix& _B = dynamic_cast<decx::_GPU_Matrix&>(B);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::CUDA_Not_init<true>(&handle);
        return handle;
    }

    if (_A.Width() != _B.Width() || _A.Height() != _B.Height()) {
        decx::err::Mat_Dim_Not_Matching<true>(&handle);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::CUDA_Event_access_fail(&handle);
        return handle;
    }

    const uint64_t len = (uint64_t)_A.Pitch() * (uint64_t)_A.Height();
    switch (_A.Type())
    {
    case decx::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kdiv_m((de::Half*)_A.Mat.ptr, (de::Half*)_B.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kdiv_m((float*)_A.Mat.ptr, (float*)_B.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kdiv_m((int*)_A.Mat.ptr, (int*)_B.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kdiv_m((double*)_A.Mat.ptr, (double*)_B.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}




de::DH de::cuda::Div(de::GPU_Matrix& src, void* __x, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _src = dynamic_cast<decx::_GPU_Matrix&>(src);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::CUDA_Not_init(&handle);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::CUDA_Event_access_fail(&handle);
        return handle;
    }

    const uint64_t len = (uint64_t)_src.Pitch() * (uint64_t)_src.Height();
    switch (_src.Type())
    {
    case decx::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kdiv_c((de::Half*)_src.Mat.ptr, *(de::Half*)__x, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kdiv_c((float*)_src.Mat.ptr, *(float*)__x, (float*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kdiv_c((int*)_src.Mat.ptr, *(int*)__x, (int*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kdiv_c((double*)_src.Mat.ptr, *(double*)__x, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}




de::DH de::cuda::Div(void* __x, de::GPU_Matrix& src, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _src = dynamic_cast<decx::_GPU_Matrix&>(src);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::CUDA_Not_init(&handle);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::CUDA_Event_access_fail(&handle);
        return handle;
    }

    const uint64_t len = (uint64_t)_src.Pitch() * (uint64_t)_src.Height();
    switch (_src.Type())
    {
    case decx::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kdiv_cinv(*(de::Half*)__x, (de::Half*)_src.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kdiv_cinv(*(float*)__x, (float*)_src.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kdiv_cinv(*(int*)__x, (int*)_src.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kdiv_cinv(*(double*)__x, (double*)_src.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}