/**
*   ----------------------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ----------------------------------------------------------------------------------
* 
* This is a part of the open source project named "DECX", a high-performance scientific
* computational library. This project follows the MIT License. For more information 
* please visit https://github.com/param0037/DECX.
* 
* Copyright (c) 2021 Wayne Anderson
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy of this 
* software and associated documentation files (the "Software"), to deal in the Software 
* without restriction, including without limitation the rights to use, copy, modify, 
* merge, publish, distribute, sublicense, and/or sell copies of the Software, and to 
* permit persons to whom the Software is furnished to do so, subject to the following 
* conditions:
* 
* The above copyright notice and this permission notice shall be included in all copies 
* or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, 
* INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR 
* PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE 
* FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR 
* OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
* DEALINGS IN THE SOFTWARE.
*/


#include "../Fms_kernel.cuh"
#include "../../../core/basic.h"
#include "Matrix_operators.h"
#include "../../../core/cudaStream_management/cudaEvent_queue.h"
#include "../../../core/cudaStream_management/cudaStream_queue.h"



de::DH de::cuda::Fms(de::GPU_Matrix& A, de::GPU_Matrix& B, de::GPU_Matrix& C, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _A = dynamic_cast<decx::_GPU_Matrix&>(A);
    decx::_GPU_Matrix& _B = dynamic_cast<decx::_GPU_Matrix&>(B);
    decx::_GPU_Matrix& _C = dynamic_cast<decx::_GPU_Matrix&>(C);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init,
            CUDA_NOT_INIT);
        return handle;
    }

    if (_A.Width() != _B.Width() || _A.Height() != _B.Height()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init,
            CUDA_NOT_INIT);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM,
            CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT,
            CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    const uint64_t len = (uint64_t)_A.Pitch() * (uint64_t)_A.Height();
    switch (_A.Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kfms_m((de::Half*)_A.Mat.ptr, (de::Half*)_B.Mat.ptr, (de::Half*)_C.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kfms_m((float*)_A.Mat.ptr, (float*)_B.Mat.ptr, (float*)_C.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kfms_m((int*)_A.Mat.ptr, (int*)_B.Mat.ptr, (int*)_C.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kfms_m((double*)_A.Mat.ptr, (double*)_B.Mat.ptr, (double*)_C.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}



de::DH de::cuda::Fms(de::GPU_Matrix& A, void* __x, de::GPU_Matrix& B, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _A = dynamic_cast<decx::_GPU_Matrix&>(A);
    decx::_GPU_Matrix& _B = dynamic_cast<decx::_GPU_Matrix&>(B);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_not_init,
            CUDA_NOT_INIT);
        return handle;
    }

    if (_A.Width() != _B.Width() || _A.Height() != _B.Height()) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_DimsNotMatching,
            MAT_DIM_NOT_MATCH);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_STREAM,
            CUDA_STREAM_ACCESS_FAIL);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::handle_error_info_modify(&handle, decx::DECX_error_types::DECX_FAIL_CUDA_EVENT,
            CUDA_EVENT_ACCESS_FAIL);
        return handle;
    }

    const uint64_t len = (uint64_t)_A.Pitch() * (uint64_t)_A.Height();
    switch (_A.Type())
    {
    case de::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kfms_c((de::Half*)_A.Mat.ptr, *(de::Half*)__x, (de::Half*)_B.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kfms_c((float*)_A.Mat.ptr, *(float*)__x, (float*)_B.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kfms_c((int*)_A.Mat.ptr, *(int*)__x, (int*)_B.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case de::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kfms_c((double*)_A.Mat.ptr, *(double*)__x, (double*)_B.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}
