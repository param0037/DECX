/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "../Fms_kernel.cuh"
#include "../../../core/basic.h"
#include "Matrix_operators.h"
#include "../../../core/cudaStream_management/cudaEvent_queue.h"
#include "../../../core/cudaStream_management/cudaStream_queue.h"



de::DH de::cuda::Fms(de::GPU_Matrix& A, de::GPU_Matrix& B, de::GPU_Matrix& C, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _A = dynamic_cast<decx::_GPU_Matrix&>(A);
    decx::_GPU_Matrix& _B = dynamic_cast<decx::_GPU_Matrix&>(B);
    decx::_GPU_Matrix& _C = dynamic_cast<decx::_GPU_Matrix&>(C);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::CUDA_Not_init<true>(&handle);
        return handle;
    }

    if (_A.Width() != _B.Width() || _A.Height() != _B.Height()) {
        decx::err::Mat_Dim_Not_Matching<true>(&handle);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::CUDA_Event_access_fail(&handle);
        return handle;
    }

    const uint64_t len = (uint64_t)_A.Pitch() * (uint64_t)_A.Height();
    switch (_A.Type())
    {
    case decx::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kfms_m((de::Half*)_A.Mat.ptr, (de::Half*)_B.Mat.ptr, (de::Half*)_C.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kfms_m((float*)_A.Mat.ptr, (float*)_B.Mat.ptr, (float*)_C.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kfms_m((int*)_A.Mat.ptr, (int*)_B.Mat.ptr, (int*)_C.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kfms_m((double*)_A.Mat.ptr, (double*)_B.Mat.ptr, (double*)_C.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}



de::DH de::cuda::Fms(de::GPU_Matrix& A, void* __x, de::GPU_Matrix& B, de::GPU_Matrix& dst)
{
    decx::_GPU_Matrix& _A = dynamic_cast<decx::_GPU_Matrix&>(A);
    decx::_GPU_Matrix& _B = dynamic_cast<decx::_GPU_Matrix&>(B);
    decx::_GPU_Matrix& _dst = dynamic_cast<decx::_GPU_Matrix&>(dst);

    de::DH handle;
    if (!decx::cuda::_is_CUDA_init()) {
        decx::err::CUDA_Not_init<true>(&handle);
        return handle;
    }

    if (_A.Width() != _B.Width() || _A.Height() != _B.Height()) {
        decx::err::Mat_Dim_Not_Matching<true>(&handle);
        return handle;
    }

    decx::hip_stream* S = decx::cuda::get_cuda_stream_ptr(hipStreamNonBlocking);
    if (S == NULL) {
        decx::err::CUDA_Stream_access_fail(&handle);
        return handle;
    }

    decx::cuda_event* E = NULL;
    E = decx::cuda::get_cuda_event_ptr(hipEventBlockingSync);
    if (E == NULL) {
        decx::err::CUDA_Event_access_fail(&handle);
        return handle;
    }

    const uint64_t len = (uint64_t)_A.Pitch() * (uint64_t)_A.Height();
    switch (_A.Type())
    {
    case decx::_DATA_TYPES_FLAGS_::_FP16_:
        decx::calc::dev_Kfms_c((de::Half*)_A.Mat.ptr, *(de::Half*)__x, (de::Half*)_B.Mat.ptr, (de::Half*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP32_:
        decx::calc::dev_Kfms_c((float*)_A.Mat.ptr, *(float*)__x, (float*)_B.Mat.ptr, (float*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_INT32_:
        decx::calc::dev_Kfms_c((int*)_A.Mat.ptr, *(int*)__x, (int*)_B.Mat.ptr, (int*)_dst.Mat.ptr, len, S);
        break;

    case decx::_DATA_TYPES_FLAGS_::_FP64_:
        decx::calc::dev_Kfms_c((double*)_A.Mat.ptr, *(double*)__x, (double*)_B.Mat.ptr, (double*)_dst.Mat.ptr, len, S);
        break;
    default:
        break;
    }

    E->event_record(S);
    E->synchronize();

    S->detach();
    E->detach();

    decx::err::Success(&handle);
    return handle;
}
