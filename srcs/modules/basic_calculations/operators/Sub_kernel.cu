#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "Sub_kernel.cuh"


__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::sub_m_ivec4(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    int4 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((int4*)&A[tid]);
        tmpB = *((int4*)&B[tid]);

        tmpdst.x = tmpA.x - tmpB.x;
        tmpdst.y = tmpA.y - tmpB.y;
        tmpdst.z = tmpA.z - tmpB.z;
        tmpdst.w = tmpA.w - tmpB.w;

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_m_ivec4_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    int4 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = tmpA.x - tmpB.x;
        tmpdst.y = tmpA.y - tmpB.y;
        tmpdst.z = tmpA.z - tmpB.z;
        tmpdst.w = tmpA.w - tmpB.w;

        dst[dex] = *((float4*)&tmpdst);
    }
}


__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::sub_m_fvec4(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    float4 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = A[tid];
        tmpB = B[tid];

        tmpdst.x = __fsub_rn(tmpA.x, tmpB.x);
        tmpdst.y = __fsub_rn(tmpA.y, tmpB.y);
        tmpdst.z = __fsub_rn(tmpA.z, tmpB.z);
        tmpdst.w = __fsub_rn(tmpA.w, tmpB.w);

        dst[tid] = tmpdst;
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_m_fvec4_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    float4 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        tmpA = A[dex];
        tmpB = B[dex];

        tmpdst.x = __fsub_rn(tmpA.x, tmpB.x);
        tmpdst.y = __fsub_rn(tmpA.y, tmpB.y);
        tmpdst.z = __fsub_rn(tmpA.z, tmpB.z);
        tmpdst.w = __fsub_rn(tmpA.w, tmpB.w);

        dst[dex] = tmpdst;
    }
}



__global__
void decx::calc::GPUK::sub_m_hvec8(float4* A, float4* B, float4* dst, const size_t len)
{
#if __ABOVE_SM_53
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    half2_8 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((half2_8*)&A[tid]);
        tmpB = *((half2_8*)&B[tid]);

        tmpdst.x = __hsub2(tmpA.x, tmpB.x);
        tmpdst.y = __hsub2(tmpA.y, tmpB.y);
        tmpdst.z = __hsub2(tmpA.z, tmpB.z);
        tmpdst.w = __hsub2(tmpA.w, tmpB.w);

        dst[tid] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_m_hvec8_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
#if __ABOVE_SM_53
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    half2_8 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = __hsub2(tmpA.x, tmpB.x);
        tmpdst.y = __hsub2(tmpA.y, tmpB.y);
        tmpdst.z = __hsub2(tmpA.z, tmpB.z);
        tmpdst.w = __hsub2(tmpA.w, tmpB.w);

        dst[dex] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::sub_m_dvec2(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    double2 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((double2*)&A[tid]);
        tmpB = *((double2*)&B[tid]);

        tmpdst.x = __dsub_rn(tmpA.x, tmpB.x);
        tmpdst.y = __dsub_rn(tmpA.y, tmpB.y);

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_m_dvec2_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    double2 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = __dsub_rn(tmpA.x, tmpB.x);
        tmpdst.y = __dsub_rn(tmpA.y, tmpB.y);

        dst[dex] = *((float4*)&tmpdst);
    }
}




// ----------------------------- C --------------------------------------


__global__
void decx::calc::GPUK::sub_c_ivec4(float4* src, int __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    int4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((int4*)&src[tid]);

        tmpdst.x = tmpsrc.x - __x;
        tmpdst.y = tmpsrc.y - __x;
        tmpdst.z = tmpsrc.z - __x;
        tmpdst.w = tmpsrc.w - __x;

        dst[tid] = *((float4*)&tmpdst);
    }
}


__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_c_ivec4_2D(float4* src, int __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    int4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = tmpsrc.x - __x;
        tmpdst.y = tmpsrc.y - __x;
        tmpdst.z = tmpsrc.z - __x;
        tmpdst.w = tmpsrc.w - __x;

        dst[dex] = *((float4*)&tmpdst);
    }
}



__global__
void decx::calc::GPUK::sub_cinv_ivec4(int __x, float4* src, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    int4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((int4*)&src[tid]);

        tmpdst.x = __x - tmpsrc.x;
        tmpdst.y = __x - tmpsrc.y;
        tmpdst.z = __x - tmpsrc.z;
        tmpdst.w = __x - tmpsrc.w;

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_cinv_ivec4_2D(float4* src, int __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    int4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __x - tmpsrc.x;
        tmpdst.y = __x - tmpsrc.y;
        tmpdst.z = __x - tmpsrc.z;
        tmpdst.w = __x - tmpsrc.w;

        dst[dex] = *((float4*)&tmpdst);
    }
}



__global__
void decx::calc::GPUK::sub_c_fvec4(float4* src, float __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    float4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = src[tid];

        tmpdst.x = __fsub_rn(tmpsrc.x, __x);
        tmpdst.y = __fsub_rn(tmpsrc.y, __x);
        tmpdst.z = __fsub_rn(tmpsrc.z, __x);
        tmpdst.w = __fsub_rn(tmpsrc.w, __x);

        dst[tid] = tmpdst;
    }
}



__global__
void decx::calc::GPUK::sub_cinv_fvec4(float __x, float4* src, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    float4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = src[tid];

        tmpdst.x = __fsub_rn(__x, tmpsrc.x);
        tmpdst.y = __fsub_rn(__x, tmpsrc.y);
        tmpdst.z = __fsub_rn(__x, tmpsrc.z);
        tmpdst.w = __fsub_rn(__x, tmpsrc.w);

        dst[tid] = tmpdst;
    }
}


__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_c_fvec4_2D(float4* src, float __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    float4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        tmpsrc = src[dex];

        tmpdst.x = __fsub_rn(tmpsrc.x, __x);
        tmpdst.y = __fsub_rn(tmpsrc.y, __x);
        tmpdst.z = __fsub_rn(tmpsrc.z, __x);
        tmpdst.w = __fsub_rn(tmpsrc.w, __x);

        dst[dex] = tmpdst;
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_cinv_fvec4_2D(float4* src, float __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    float4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        tmpsrc = src[dex];

        tmpdst.x = __fsub_rn(__x, tmpsrc.x);
        tmpdst.y = __fsub_rn(__x, tmpsrc.y);
        tmpdst.z = __fsub_rn(__x, tmpsrc.z);
        tmpdst.w = __fsub_rn(__x, tmpsrc.w);

        dst[dex] = tmpdst;
    }
}



__global__
void decx::calc::GPUK::sub_c_hvec8(float4* src, half2 __x, float4* dst, const size_t len)
{
#if __ABOVE_SM_53
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    half2_8 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((half2_8*)&src[tid]);

        tmpdst.x = __hsub2(tmpsrc.x, __x);
        tmpdst.y = __hsub2(tmpsrc.y, __x);
        tmpdst.z = __hsub2(tmpsrc.z, __x);
        tmpdst.w = __hsub2(tmpsrc.w, __x);

        dst[tid] = *((float4*)&tmpdst);
    }
#endif
}



__global__
void decx::calc::GPUK::sub_cinv_hvec8(half2 __x, float4* src, float4* dst, const size_t len)
{
#if __ABOVE_SM_53
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    half2_8 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((half2_8*)&src[tid]);

        tmpdst.x = __hsub2(__x, tmpsrc.x);
        tmpdst.y = __hsub2(__x, tmpsrc.y);
        tmpdst.z = __hsub2(__x, tmpsrc.z);
        tmpdst.w = __hsub2(__x, tmpsrc.w);

        dst[tid] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_c_hvec8_2D(float4* src, half2 __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
#if __ABOVE_SM_53
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    half2_8 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __hsub2(tmpsrc.x, __x);
        tmpdst.y = __hsub2(tmpsrc.y, __x);
        tmpdst.z = __hsub2(tmpsrc.z, __x);
        tmpdst.w = __hsub2(tmpsrc.w, __x);

        dst[dex] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_cinv_hvec8_2D(float4* src, half2 __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
#if __ABOVE_SM_53
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    half2_8 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __hsub2(__x, tmpsrc.x);
        tmpdst.y = __hsub2(__x, tmpsrc.y);
        tmpdst.z = __hsub2(__x, tmpsrc.z);
        tmpdst.w = __hsub2(__x, tmpsrc.w);

        dst[dex] = *((float4*)&tmpdst);
    }
#endif
}



__global__
void decx::calc::GPUK::sub_c_dvec2(float4* src, double __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    double2 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((double2*)&src[tid]);

        tmpdst.x = __dsub_rn(tmpsrc.x, __x);
        tmpdst.y = __dsub_rn(tmpsrc.y, __x);

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
void decx::calc::GPUK::sub_cinv_dvec2(double __x, float4* src, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    double2 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((double2*)&src[tid]);

        tmpdst.x = __dsub_rn(__x, tmpsrc.x);
        tmpdst.y = __dsub_rn(__x, tmpsrc.y);

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_c_dvec2_2D(float4* src, double __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    double2 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __dsub_rn(tmpsrc.x, __x);
        tmpdst.y = __dsub_rn(tmpsrc.y, __x);

        dst[dex] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::sub_cinv_dvec2_2D(float4* src, double __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    double2 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __dsub_rn(__x, tmpsrc.x);
        tmpdst.y = __dsub_rn(__x, tmpsrc.y);

        dst[dex] = *((float4*)&tmpdst);
    }
}