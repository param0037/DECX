#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/backup_1
*/

#include "Mul_kernel.cuh"



__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::mul_m_ivec4(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    int4 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((int4*)&A[tid]);
        tmpB = *((int4*)&B[tid]);

        tmpdst.x = tmpA.x * tmpB.x;
        tmpdst.y = tmpA.y * tmpB.y;
        tmpdst.z = tmpA.z * tmpB.z;
        tmpdst.w = tmpA.w * tmpB.w;

        dst[tid] = *((float4*)&tmpdst);
    }
}


__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_m_ivec4_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    int4 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = tmpA.x * tmpB.x;
        tmpdst.y = tmpA.y * tmpB.y;
        tmpdst.z = tmpA.z * tmpB.z;
        tmpdst.w = tmpA.w * tmpB.w;

        dst[dex] = *((float4*)&tmpdst);
    }
}


__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::mul_m_fvec4(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    float4 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = A[tid];
        tmpB = B[tid];

        tmpdst.x = __fmul_rn(tmpA.x, tmpB.x);
        tmpdst.y = __fmul_rn(tmpA.y, tmpB.y);
        tmpdst.z = __fmul_rn(tmpA.z, tmpB.z);
        tmpdst.w = __fmul_rn(tmpA.w, tmpB.w);

        dst[tid] = tmpdst;
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_m_fvec4_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    float4 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        tmpA = A[dex];
        tmpB = B[dex];

        tmpdst.x = __fmul_rn(tmpA.x, tmpB.x);
        tmpdst.y = __fmul_rn(tmpA.y, tmpB.y);
        tmpdst.z = __fmul_rn(tmpA.z, tmpB.z);
        tmpdst.w = __fmul_rn(tmpA.w, tmpB.w);

        dst[dex] = tmpdst;
    }
}



__global__
void decx::calc::GPUK::mul_m_hvec8(float4* A, float4* B, float4* dst, const size_t len)
{
#if __ABOVE_SM_53
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    half2_8 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((half2_8*)&A[tid]);
        tmpB = *((half2_8*)&B[tid]);

        tmpdst.x = __hmul2(tmpA.x, tmpB.x);
        tmpdst.y = __hmul2(tmpA.y, tmpB.y);
        tmpdst.z = __hmul2(tmpA.z, tmpB.z);
        tmpdst.w = __hmul2(tmpA.w, tmpB.w);

        dst[tid] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_m_hvec8_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
#if __ABOVE_SM_53
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    half2_8 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = __hmul2(tmpA.x, tmpB.x);
        tmpdst.y = __hmul2(tmpA.y, tmpB.y);
        tmpdst.z = __hmul2(tmpA.z, tmpB.z);
        tmpdst.w = __hmul2(tmpA.w, tmpB.w);

        dst[dex] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param len : have considered vec4
*/
void decx::calc::GPUK::mul_m_dvec2(float4* A, float4* B, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    double2 tmpA, tmpB, tmpdst;

    if (tid < len) {
        tmpA = *((double2*)&A[tid]);
        tmpB = *((double2*)&B[tid]);

        tmpdst.x = __dmul_rn(tmpA.x, tmpB.x);
        tmpdst.y = __dmul_rn(tmpA.y, tmpB.y);

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_m_dvec2_2D(float4* A, float4* B, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    double2 tmpA, tmpB, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpA) = A[dex];
        *((float4*)&tmpB) = B[dex];

        tmpdst.x = __dmul_rn(tmpA.x, tmpB.x);
        tmpdst.y = __dmul_rn(tmpA.y, tmpB.y);

        dst[dex] = *((float4*)&tmpdst);
    }
}



// ----------------------------- C --------------------------------------


__global__
void decx::calc::GPUK::mul_c_ivec4(float4* src, int __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    int4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((int4*)&src[tid]);

        tmpdst.x = tmpsrc.x * __x;
        tmpdst.y = tmpsrc.y * __x;
        tmpdst.z = tmpsrc.z * __x;
        tmpdst.w = tmpsrc.w * __x;

        dst[tid] = *((float4*)&tmpdst);
    }
}


__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_c_ivec4_2D(float4* src, int __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    int4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = tmpsrc.x * __x;
        tmpdst.y = tmpsrc.y * __x;
        tmpdst.z = tmpsrc.z * __x;
        tmpdst.w = tmpsrc.w * __x;

        dst[dex] = *((float4*)&tmpdst);
    }
}


__global__
void decx::calc::GPUK::mul_c_fvec4(float4* src, float __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    float4 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = src[tid];

        tmpdst.x = __fmul_rn(tmpsrc.x, __x);
        tmpdst.y = __fmul_rn(tmpsrc.y, __x);
        tmpdst.z = __fmul_rn(tmpsrc.z, __x);
        tmpdst.w = __fmul_rn(tmpsrc.w, __x);

        dst[tid] = tmpdst;
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_c_fvec4_2D(float4* src, float __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    float4 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        tmpsrc = src[dex];

        tmpdst.x = __fmul_rn(tmpsrc.x, __x);
        tmpdst.y = __fmul_rn(tmpsrc.y, __x);
        tmpdst.z = __fmul_rn(tmpsrc.z, __x);
        tmpdst.w = __fmul_rn(tmpsrc.w, __x);

        dst[dex] = tmpdst;
    }
}


__global__
void decx::calc::GPUK::mul_c_hvec8(float4* src, half2 __x, float4* dst, const size_t len)
{
#if __ABOVE_SM_53
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    half2_8 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((half2_8*)&src[tid]);

        tmpdst.x = __hmul2(tmpsrc.x, __x);
        tmpdst.y = __hmul2(tmpsrc.y, __x);
        tmpdst.z = __hmul2(tmpsrc.z, __x);
        tmpdst.w = __hmul2(tmpsrc.w, __x);

        dst[tid] = *((float4*)&tmpdst);
    }
#endif
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_c_hvec8_2D(float4* src, half2 __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
#if __ABOVE_SM_53
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    half2_8 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __hmul2(tmpsrc.x, __x);
        tmpdst.y = __hmul2(tmpsrc.y, __x);
        tmpdst.z = __hmul2(tmpsrc.z, __x);
        tmpdst.w = __hmul2(tmpsrc.w, __x);

        dst[dex] = *((float4*)&tmpdst);
    }
#endif
}



__global__
void decx::calc::GPUK::mul_c_dvec2(float4* src, double __x, float4* dst, const size_t len)
{
    size_t tid = threadIdx.x + blockDim.x * blockIdx.x;
    double2 tmpsrc, tmpdst;

    if (tid < len) {
        tmpsrc = *((double2*)&src[tid]);

        tmpdst.x = __dmul_rn(tmpsrc.x, __x);
        tmpdst.y = __dmul_rn(tmpsrc.y, __x);

        dst[tid] = *((float4*)&tmpdst);
    }
}



__global__
/**
* int* x2, add together
* @param eq_pitch : have considered vec4
* @param bounds.x : The width, in float4
* @param bounds.y : The height, in float
*/
void decx::calc::GPUK::mul_c_dvec2_2D(float4* src, double __x, float4* dst, const size_t eq_pitch, const uint2 bounds)
{
    uint tidx = threadIdx.x + blockIdx.x * blockDim.x;
    uint tidy = threadIdx.y + blockIdx.y * blockDim.y;

    size_t dex = (size_t)tidx * eq_pitch + (size_t)tidy;
    double2 tmpsrc, tmpdst;

    if (tidx < bounds.y && tidy < bounds.x) {
        *((float4*)&tmpsrc) = src[dex];

        tmpdst.x = __dmul_rn(tmpsrc.x, __x);
        tmpdst.y = __dmul_rn(tmpsrc.y, __x);

        dst[dex] = *((float4*)&tmpdst);
    }
}