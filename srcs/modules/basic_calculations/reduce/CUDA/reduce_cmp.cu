#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "reduce_callers.cuh"
#include "../../../core/allocators.h"


template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 4;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_fp32<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp32<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<true, true> (decx::reduce::cuda_reduce1D_configs<float>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<float>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S);



template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 8;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const de::Half fill_val = _kp_configs->get_fill_val();

    const float4* read_ptr = NULL;
    __half* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (__half*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (__half*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_fp16<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, *((__half*)&fill_val));

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 8);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (__half*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp16<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, *((__half*)&fill_val));

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 8);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<true, true> (decx::reduce::cuda_reduce1D_configs<de::Half>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<de::Half>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);




template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v16 = _kp_configs->get_proc_len() / 16;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    uint8_t* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (uint8_t*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (uint8_t*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_u8<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v16, proc_len_v1, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v16 = decx::utils::ceil<uint64_t>(proc_len_v1, 16);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (uint8_t*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_u8<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v16, proc_len_v1, _kp_configs->get_fill_val());

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v16 = decx::utils::ceil<uint64_t>(proc_len_v1, 16);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);
        }
    }
}


template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<true, true>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);