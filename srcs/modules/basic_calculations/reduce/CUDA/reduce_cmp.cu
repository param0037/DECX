#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "reduce_callers.cuh"
#include "../../../core/allocators.h"


template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 4;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_fp32<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp32<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<true, true> (decx::reduce::cuda_reduce1D_configs<float>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<float>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp32_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S);



template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_fp64_caller_Async(decx::reduce::cuda_reduce1D_configs<double>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 2;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const double2* read_ptr = NULL;
    double* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (double2*)_kp_configs->get_dev_src().ptr;
        write_ptr = (double*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (double2*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (double*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_fp64<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 2);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (double2*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (double*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp64<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 2);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_cmp_fp64_caller_Async<true, true>(decx::reduce::cuda_reduce1D_configs<double>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp64_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<double>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp64_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<double>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp64_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<double>* _kp_configs, decx::hip_stream* S);




template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 8;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const de::Half fill_val = _kp_configs->get_fill_val();

    const float4* read_ptr = NULL;
    __half* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (__half*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (__half*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_fp16<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, *((__half*)&fill_val));

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 8);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (__half*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp16<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, *((__half*)&fill_val));

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 8);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<true, true> (decx::reduce::cuda_reduce1D_configs<de::Half>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<de::Half>*    _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_fp16_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);




template <bool _src_from_device, bool _is_max>
void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v16 = _kp_configs->get_proc_len() / 16;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    uint8_t* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (uint8_t*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (uint8_t*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_cmp1D_u8<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v16, proc_len_v1, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v16 = decx::utils::ceil<uint64_t>(proc_len_v1, 16);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (uint8_t*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_u8<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v16, proc_len_v1, _kp_configs->get_fill_val());

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v16 = decx::utils::ceil<uint64_t>(proc_len_v1, 16);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v16, _REDUCE1D_BLOCK_DIM_);
        }
    }
}


template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<true, true>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<false, true>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<true, false>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_cmp_u8_caller_Async<false, false>(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S);



template <bool _is_max>
void decx::reduce::reduce_cmp2D_h_fp32_Async(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S)
{
    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    uint32_t grid_x = decx::utils::ceil<uint32_t>(_configs->get_proc_dims_v().x, _REDUCE2D_BLOCK_DIM_X_);
    const uint32_t grid_y = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().y, _REDUCE2D_BLOCK_DIM_Y_);

    uint2 proc_dims_actual = _configs->get_actual_proc_dims();

    uint32_t Wdsrc_v4_varient = _configs->get_dtmp1()._dims.x / 4;
    uint32_t Wddst_v1_varient = decx::utils::ceil<uint32_t>(grid_x, 4) * 4;

    for (int i = 0; i < _configs->get_kernel_call_times() - 1; ++i)
    {
        read_ptr = (float4*)_configs->get_leading_ptr();
        write_ptr = (float*)_configs->get_lagging_ptr();

        _configs->reverse_MIF_states();

        decx::reduce::GPUK::cu_warp_reduce_cmp2D_h_fp32<_is_max> << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
            0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wdsrc_v4_varient, Wddst_v1_varient, proc_dims_actual);

        Wdsrc_v4_varient = decx::utils::ceil<uint32_t>(grid_x, 4);
        proc_dims_actual.x = grid_x;

        grid_x = decx::utils::ceil<uint32_t>(decx::utils::ceil<uint32_t>(proc_dims_actual.x, 4), _REDUCE2D_BLOCK_DIM_X_);
        Wddst_v1_varient = decx::utils::ceil<uint32_t>(grid_x, 4) * 4;
    }

    read_ptr = (float4*)_configs->get_leading_ptr();
    write_ptr = (float*)_configs->get_lagging_ptr();

    _configs->reverse_MIF_states();

    decx::reduce::GPUK::cu_warp_reduce_cmp2D_h_fp32_transp<_is_max> << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wdsrc_v4_varient, proc_dims_actual.y, proc_dims_actual);
}

template void decx::reduce::reduce_cmp2D_h_fp32_Async<true>(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S);
template void decx::reduce::reduce_cmp2D_h_fp32_Async<false>(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S);




template <bool _is_max>
void decx::reduce::reduce_cmp2D_v_fp32_Async(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S)
{
    const float4* read_ptr = NULL;
    float4* write_ptr = NULL;

    uint32_t grid_y = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().y, _REDUCE2D_BLOCK_DIM_Y_);
    const uint32_t grid_x = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().x, _REDUCE2D_BLOCK_DIM_X_ * 4);

    uint2 _proc_dims_v4 = _configs->get_proc_dims_v();

    const uint32_t Wsrc_v4 = _configs->get_proc_dims_v().x;
    const uint32_t Wdst_v4 = Wsrc_v4;

    while (true)
    {
        read_ptr = (float4*)_configs->get_leading_ptr();
        write_ptr = (float4*)_configs->get_lagging_ptr();

        _configs->reverse_MIF_states();

        decx::reduce::GPUK::cu_warp_reduce_cmp2D_v_fp32<_is_max> << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
            0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wsrc_v4, Wdst_v4, _proc_dims_v4);

        if (grid_y == 1) {
            break;
        }

        _proc_dims_v4.y = grid_y;
        grid_y = decx::utils::ceil<uint32_t>(_proc_dims_v4.y, _REDUCE2D_BLOCK_DIM_Y_);
    }
}

template void decx::reduce::reduce_cmp2D_v_fp32_Async<true>(decx::reduce::cuda_reduce2D_1way_configs<float>*, decx::hip_stream*); 
template void decx::reduce::reduce_cmp2D_v_fp32_Async<false>(decx::reduce::cuda_reduce2D_1way_configs<float>*, decx::hip_stream*);





template <bool _is_max, bool _src_from_device>
void decx::reduce::reduce_cmp2D_full_fp32_Async(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, const uint2 proc_dims, 
    const uint32_t _pitch_src_v4, decx::hip_stream* S)
{
    const uint2 proc_dims_v4 = make_uint2(decx::utils::ceil<uint32_t>(proc_dims.x, 4), proc_dims.y);
    const dim3 _flatten_K_grid = dim3(decx::utils::ceil<uint32_t>(proc_dims.x, _REDUCE2D_BLOCK_DIM_X_ * 4),
                                      decx::utils::ceil<uint32_t>(proc_dims.y, _REDUCE2D_BLOCK_DIM_Y_));

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_warp_reduce_cmp2D_1D_fp32<_is_max> << <_flatten_K_grid, dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, _pitch_src_v4, proc_dims, _kp_configs->get_fill_val());

    _kp_configs->inverse_mutex_MIF_states();

    // 1D-layout array processing
    uint64_t grid_len = _flatten_K_grid.x * _flatten_K_grid.y;
    uint64_t proc_len_v1, proc_len_v;

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_cmp1D_fp32<_is_max> << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1, _kp_configs->get_fill_val());
            
            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::reduce_cmp2D_full_fp32_Async<true, true>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*); 
template void decx::reduce::reduce_cmp2D_full_fp32_Async<true, false>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*);
template void decx::reduce::reduce_cmp2D_full_fp32_Async<false, true>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*);
template void decx::reduce::reduce_cmp2D_full_fp32_Async<false, false>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*);