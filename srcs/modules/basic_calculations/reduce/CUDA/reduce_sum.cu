#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/


#include "reduce_callers.cuh"
#include "../../../core/allocators.h"



template <bool _src_from_device>
void decx::reduce::cuda_reduce1D_sum_fp32_caller_Async(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 4;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_sum1D_fp32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_sum1D_fp32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}


template void decx::reduce::cuda_reduce1D_sum_fp32_caller_Async<true> (decx::reduce::cuda_reduce1D_configs<float>*, decx::hip_stream*);
template void decx::reduce::cuda_reduce1D_sum_fp32_caller_Async<false>(decx::reduce::cuda_reduce1D_configs<float>*, decx::hip_stream*);



template <bool _src_from_device>
void decx::reduce::cuda_reduce1D_sum_u8_i32_caller_Async(decx::reduce::cuda_reduce1D_configs<uint8_t>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 16;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const int4* read_ptr = NULL;
    int32_t* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (int4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (int32_t*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (int4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (int32_t*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_sum1D_u8_i32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1) 
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (int4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (int32_t*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_sum1D_int32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}


template void decx::reduce::cuda_reduce1D_sum_u8_i32_caller_Async<true> (decx::reduce::cuda_reduce1D_configs<uint8_t>*  _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_sum_u8_i32_caller_Async<false>(decx::reduce::cuda_reduce1D_configs<uint8_t>*  _kp_configs, decx::hip_stream* S);



template <bool _src_from_device>
void decx::reduce::cuda_reduce1D_sum_fp16_fp32_caller_Async(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S)
{
    uint64_t proc_len_v = _kp_configs->get_proc_len() / 8;
    uint64_t proc_len_v1 = _kp_configs->get_actual_len();
    uint64_t grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_block_reduce_sum1D_fp16_fp32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

    _kp_configs->inverse_mutex_MIF_states();

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_sum1D_fp32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::cuda_reduce1D_sum_fp16_fp32_caller_Async<true> (decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);
template void decx::reduce::cuda_reduce1D_sum_fp16_fp32_caller_Async<false>(decx::reduce::cuda_reduce1D_configs<de::Half>* _kp_configs, decx::hip_stream* S);




void decx::reduce::reduce_sum2D_h_fp32_Async(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S)
{
    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    uint32_t grid_x = decx::utils::ceil<uint32_t>(_configs->get_proc_dims_v().x, _REDUCE2D_BLOCK_DIM_X_);
    const uint32_t grid_y = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().y, _REDUCE2D_BLOCK_DIM_Y_);

    uint2 proc_dims_actual = _configs->get_actual_proc_dims();

    uint32_t Wdsrc_v4_varient = _configs->get_dtmp1()._dims.x / 4;
    uint32_t Wddst_v1_varient = decx::utils::ceil<uint32_t>(grid_x, 4) * 4;

    for (int i = 0; i < _configs->get_kernel_call_times() - 1; ++i)
    {
        read_ptr = (float4*)_configs->get_leading_ptr();
        write_ptr = (float*)_configs->get_lagging_ptr();

        _configs->reverse_MIF_states();

        decx::reduce::GPUK::cu_warp_reduce_sum2D_h_fp32 << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
            0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wdsrc_v4_varient, Wddst_v1_varient, proc_dims_actual);

        Wdsrc_v4_varient = decx::utils::ceil<uint32_t>(grid_x, 4);
        proc_dims_actual.x = grid_x;

        grid_x = decx::utils::ceil<uint32_t>(decx::utils::ceil<uint32_t>(proc_dims_actual.x, 4), _REDUCE2D_BLOCK_DIM_X_);
        Wddst_v1_varient = decx::utils::ceil<uint32_t>(grid_x, 4) * 4;
    }

    read_ptr = (float4*)_configs->get_leading_ptr();
    write_ptr = (float*)_configs->get_lagging_ptr();

    _configs->reverse_MIF_states();

    decx::reduce::GPUK::cu_warp_reduce_sum2D_h_fp32_transp << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wdsrc_v4_varient, proc_dims_actual.y, proc_dims_actual);
}



void decx::reduce::reduce_sum2D_v_fp32_Async(decx::reduce::cuda_reduce2D_1way_configs<float>* _configs, decx::hip_stream* S)
{
    const float4* read_ptr = NULL;
    float4* write_ptr = NULL;

    uint32_t grid_y = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().y, _REDUCE2D_BLOCK_DIM_Y_);
    const uint32_t grid_x = decx::utils::ceil<uint32_t>(_configs->get_actual_proc_dims().x, _REDUCE2D_BLOCK_DIM_X_ * 4);

    uint2 _proc_dims_v4 = _configs->get_proc_dims_v();

    const uint32_t Wsrc_v4 = _configs->get_proc_dims_v().x;
    const uint32_t Wdst_v4 = Wsrc_v4;

    while (true)
    {
        read_ptr = (float4*)_configs->get_leading_ptr();
        write_ptr = (float4*)_configs->get_lagging_ptr();

        _configs->reverse_MIF_states();

        decx::reduce::GPUK::cu_warp_reduce_sum2D_v_fp32 << <dim3(grid_x, grid_y), dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
            0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, Wsrc_v4, Wdst_v4, _proc_dims_v4);

        if (grid_y == 1) {
            break;
        }

        _proc_dims_v4.y = grid_y;
        grid_y = decx::utils::ceil<uint32_t>(_proc_dims_v4.y, _REDUCE2D_BLOCK_DIM_Y_);
    }
}



template <bool _src_from_device>
void decx::reduce::reduce_sum2D_full_fp32_Async(decx::reduce::cuda_reduce1D_configs<float>* _kp_configs, const uint2 proc_dims, 
    const uint32_t _pitch_src_v4, decx::hip_stream* S)
{
    const uint2 proc_dims_v4 = make_uint2(decx::utils::ceil<uint32_t>(proc_dims.x, 4), proc_dims.y);
    const dim3 _flatten_K_grid = dim3(decx::utils::ceil<uint32_t>(proc_dims.x, _REDUCE2D_BLOCK_DIM_X_ * 4),
                                      decx::utils::ceil<uint32_t>(proc_dims.y, _REDUCE2D_BLOCK_DIM_Y_));

    const float4* read_ptr = NULL;
    float* write_ptr = NULL;

    if (_src_from_device) {
        read_ptr = (float4*)_kp_configs->get_dev_src().ptr;
        write_ptr = (float*)_kp_configs->get_dev_tmp1().ptr;
    }
    else {
        read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
        write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;
    }

    decx::reduce::GPUK::cu_warp_reduce_sum2D_1D_fp32 << <_flatten_K_grid, dim3(_REDUCE2D_BLOCK_DIM_X_, _REDUCE2D_BLOCK_DIM_Y_),
        0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, _pitch_src_v4, proc_dims);

    _kp_configs->inverse_mutex_MIF_states();

    // 1D-layout array processing
    uint64_t grid_len = _flatten_K_grid.x * _flatten_K_grid.y;
    uint64_t proc_len_v1, proc_len_v;

    if (grid_len > 1)
    {
        proc_len_v1 = grid_len;
        proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
        grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);

        while (true)
        {
            read_ptr = (float4*)_kp_configs->get_leading_MIF().mem;
            write_ptr = (float*)_kp_configs->get_lagging_MIF().mem;

            decx::reduce::GPUK::cu_block_reduce_sum1D_fp32 << <grid_len, _REDUCE1D_BLOCK_DIM_,
                0, S->get_raw_stream_ref() >> > (read_ptr, write_ptr, proc_len_v, proc_len_v1);

            _kp_configs->inverse_mutex_MIF_states();

            if (grid_len == 1) {
                break;
            }

            proc_len_v1 = grid_len;
            proc_len_v = decx::utils::ceil<uint64_t>(proc_len_v1, 4);
            grid_len = decx::utils::ceil<uint64_t>(proc_len_v, _REDUCE1D_BLOCK_DIM_);
        }
    }
}

template void decx::reduce::reduce_sum2D_full_fp32_Async<true>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*); 
template void decx::reduce::reduce_sum2D_full_fp32_Async<false>(decx::reduce::cuda_reduce1D_configs<float>*, const uint2, const uint32_t, decx::hip_stream*);