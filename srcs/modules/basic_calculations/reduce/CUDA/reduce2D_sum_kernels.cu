#include "hip/hip_runtime.h"
/**
*   ---------------------------------------------------------------------
*   Author : Wayne Anderson
*   Date   : 2021.04.16
*   ---------------------------------------------------------------------
*   This is a part of the open source program named "DECX", copyright c Wayne,
*   2021.04.16, all right reserved.
*   More information please visit https://github.com/param0037/DECX
*/

#include "reduce_sum.cuh"



__global__ void 
decx::reduce::GPUK::cu_warp_reduce_sum2D_h_fp32(const float4 * __restrict   src, 
                                                float* __restrict           dst,
                                                const uint32_t              Wsrc_v4, 
                                                uint32_t                    Wdst_v1, 
                                                const uint2                 proc_dims)
{
    uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t tidy = threadIdx.y + blockDim.y * blockIdx.y;

    uint64_t LDG_dex = Wsrc_v4 * tidy + tidx;
    uint64_t STG_dex = Wdst_v1 * tidy + blockIdx.x;

    uint32_t proc_W_v4 = decx::utils::ceil<uint32_t>(proc_dims.x, 4);

    decx::utils::_cuda_vec128 _recv;
    _recv._vf = decx::utils::vec4_set1_fp32(0);

    float _thread_sum = 0, _warp_reduce_res = 0;

    if (tidx < proc_W_v4 && tidy < proc_dims.y) {
        _recv._vf = src[LDG_dex];
        if (tidx == proc_W_v4 - 1) {
            for (int i = 4 - (proc_W_v4 * 4 - proc_dims.x); i < 4; ++i) {
                _recv._arrf[i] = 0.f;
            }
        }
    }

    _thread_sum = decx::reduce::GPUK::float4_sum(_recv._vf);

    decx::reduce::GPUK::cu_warp_reduce_fp32<float(float, float), 32>(__fadd_rn, &_thread_sum, &_warp_reduce_res);

    if (threadIdx.x == 0 && tidy < proc_dims.y) {
        dst[STG_dex] = _warp_reduce_res;
    }
}



__global__ void 
decx::reduce::GPUK::cu_warp_reduce_sum2D_h_fp32_transp(const float4 * __restrict   src, 
                                                       float* __restrict           dst,
                                                       const uint32_t              Wsrc_v4, 
                                                       uint32_t                    Wdst_v1, 
                                                       const uint2                 proc_dims)
{
    uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t tidy = threadIdx.y + blockDim.y * blockIdx.y;

    uint64_t LDG_dex = Wsrc_v4 * tidy + tidx;
    uint64_t STG_dex = Wdst_v1 * blockIdx.x + tidy;

    uint32_t proc_W_v4 = decx::utils::ceil<uint32_t>(proc_dims.x, 4);

    decx::utils::_cuda_vec128 _recv;
    _recv._vf = decx::utils::vec4_set1_fp32(0);

    float _thread_sum = 0, _warp_reduce_res = 0;

    if (tidx < proc_W_v4 && tidy < proc_dims.y) {
        _recv._vf = src[LDG_dex];
        if (tidx == proc_W_v4 - 1) {
            for (int i = 4 - (proc_W_v4 * 4 - proc_dims.x); i < 4; ++i) {
                _recv._arrf[i] = 0.f;
            }
        }
    }

    _thread_sum = decx::reduce::GPUK::float4_sum(_recv._vf);

    decx::reduce::GPUK::cu_warp_reduce_fp32<float(float, float), 32>(__fadd_rn, &_thread_sum, &_warp_reduce_res);

    if (threadIdx.x == 0 && tidy < proc_dims.y) {
        dst[STG_dex] = _warp_reduce_res;
    }
}



__global__ void 
decx::reduce::GPUK::cu_warp_reduce_sum2D_v_fp32(const float4 * __restrict   src, 
                                                float4* __restrict          dst,
                                                const uint32_t              Wsrc_v4, 
                                                uint32_t                    Wdst_v4, 
                                                const uint2                 proc_dims_v4)
{
    uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t tidy = threadIdx.y + blockDim.y * blockIdx.y;

    uint64_t LDG_dex = Wsrc_v4 * tidy + tidx;
    uint64_t STG_dex = blockIdx.y * Wdst_v4 + tidx;

    __shared__ float4 _workspace[8][32 + 1];

    decx::utils::_cuda_vec128 _recv;
    _recv._vf = decx::utils::vec4_set1_fp32(0);

    float2 tmp1, tmp2, tmp3, tmp4;
    
    /**
    * No need to fill in zeros to the remaining spaces, since the loading
    * process goes all the way down vertically. The process stops at exactly 
    * where the matrix ends.
    */
    if (tidx < proc_dims_v4.x && tidy < proc_dims_v4.y) {
        _recv._vf = src[LDG_dex];
    }

    _workspace[threadIdx.y][threadIdx.x] = _recv._vf;

    __syncthreads();

    tmp1 = ((float2*)_workspace[threadIdx.x % 8])[threadIdx.y * 4 + threadIdx.x / 8];
    tmp2 = ((float2*)_workspace[threadIdx.x % 8])[32 + threadIdx.y * 4 + threadIdx.x / 8];

    __syncwarp(0xffffffff);

    decx::reduce::GPUK::cu_warp_reduce_fp64<double(double, double), 32, 4>(decx::utils::cuda::__float2_add, ((double*)&tmp1), ((double*)&tmp3));
    decx::reduce::GPUK::cu_warp_reduce_fp64<double(double, double), 32, 4>(decx::utils::cuda::__float2_add, ((double*)&tmp2), ((double*)&tmp4));

    __syncthreads();

    if (threadIdx.x % 8 == 0) {
        ((float2*)_workspace[0])[threadIdx.y * 4 + threadIdx.x / 8] = tmp3;
        ((float2*)_workspace[0])[32 + threadIdx.y * 4 + threadIdx.x / 8] = tmp4;
    }

    __syncthreads();
    
    _recv._vf = _workspace[threadIdx.y][threadIdx.x];

    if (tidx < proc_dims_v4.x && threadIdx.y == 0) {
        dst[STG_dex] = _recv._vf;
    }
}